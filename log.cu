#include "hip/hip_runtime.h"
//      log.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Arquivo com funções relativas a impressão do log de saída do programa
//
//		30/03/2012

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "linkedlist.h"

void prepareLog();
void print_time();
void printString(char*,char*);
void printSet(int);
void print_matchs(int,int);
void print_tempo(float);
void closeLog();


FILE *logfile;
FILE *logfileDetalhado;

void prepareLog(){
  //Abre e prepara arquivo log.dat para receber mensagens de log
  
  logfile = fopen("log.dat","a");
  logfileDetalhado = fopen("logDetalhado.dat","a");
  
	if(ferror(logfile) != 0 || ferror(logfileDetalhado) !=0){
		printf("Erro! Impossível salvar log\n");
		exit(1);
	}
   
  fprintf(logfile,"\n\n-------------------------\n");
  fprintf(logfileDetalhado,"\n\n-------------------------\n");
 
  print_time();
}

void print_time(){
 
 struct tm *local;
 time_t t;
 
 t = time(NULL);
 local = localtime(&t);
 
 fprintf(logfile,"%s\n",asctime(local));
 fprintf(logfileDetalhado,"%s\n\n",asctime(local));

  return;
}

//Métodos específicos#######

void print_open_file(char *c){
	fprintf(logfile,"Arquivo %s aberto.",c);
	fprintf(logfileDetalhado,"Arquivo %s aberto.",c);
	return;
}
void printString(char *c,char *s){
	if(s != NULL)
		fprintf(logfile,"%s %s\n",c,s);
	else
		fprintf(logfile,"%s\n",c);
	
}

void printSet(int n){
	fprintf(logfile,"Bases por sequência: %d.\n",n);
	fprintf(logfileDetalhado,"Bases por sequência: %d.\n",n);
}

void print_seqs_carregadas(int n){
	fprintf(logfileDetalhado,"Sequências carregadas: %d\n",n);
}

void print_matchs(int sensos,int antisensos){
	fprintf(logfile,"Sequências senso encontradas: %d.\nSequências antisenso encontradas: %d.",sensos,antisensos);
	fprintf(logfileDetalhado,"Sequências senso encontradas: %d.\nSequências antisenso encontradas: %d.",sensos,antisensos);
}

void print_resultados(lista_ligada** resultados){
	int i;
	i = 0;
	
	fprintf(logfile,"Pares encontrados:\n\n");
	while(resultados[i]->pares != -1){
			fprintf(logfile,"	%s x%d => %.3f \%\n",resultados[i]->senso,resultados[i]->pares,resultados[i]->qnt_relativa*100);
			fprintf(logfileDetalhado,"	%s x%d => %.3f \%\n",resultados[i]->senso,resultados[i]->pares,resultados[i]->qnt_relativa*100);
			i++;
	}
	
	i=0;
	fprintf(logfile,"Sensos despareados:\n\n");
	while(resultados[i]->pares != -1){
			///fprintf(logfile,"	%s x%d => %.3f \%\n",resultados[i]->senso,resultados[i]->pares,resultados[i]->qnt_relativa*100);
			fprintf(logfileDetalhado,"	%s S:%d - AS:%d\n",resultados[i]->senso,resultados[i]->qsenso,resultados[i]->qasenso);
			i++;
	}
}

void print_tempo(float tempo){
	if(tempo > 0.5)
		fprintf(logfile,"Tempo decorrido: %fs\n",tempo/1000.0);
	else
		fprintf(logfile,"Tempo decorrido: %fms\n",tempo);
		
		fprintf(logfileDetalhado,"Tempo decorrido: %fms\n",tempo);
	
}

//##########################

void closeLog(){
	
  fprintf(logfile,"\n-------------------------\n");
  
   if(logfile != NULL)
      fclose(logfile);
   if(logfileDetalhado != NULL)
		fclose(logfileDetalhado);
	return;
}

