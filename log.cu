
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
FILE *logfile;
//FILE *logfileDetalhado;

void prepareLog();
void print_time();
void printToLog_Fim(int tiro,int raio,int fim,float x,float z);
void printToLog_Mem(float MemUsada,float MemPorIter);
void printToLog_Runtime(float t);
void printString(char*,char*);
void printSet(int,int);
void print_matchs(int,int);
void print_tempo(float);
void closeLog();

void prepareLog(){
  //Abre e prepara arquivo log.dat para receber mensagens de log
  
  logfile = fopen("log.dat","a");
  //logfileDetalhado = fopen("logDetalhado.dat","a");
  
	if(ferror(logfile) != 0){
		printf("Erro! Impossível salvar log\n");
		exit(1);
	}
   
  fprintf(logfile,"\n\n-------------------------\n");
 // fprintf(logfileDetalhado,"-------------------------\n");
  print_time();
}

void print_time()
{
 
 struct tm *local;
 time_t t;
 
 t = time(NULL);
 local = localtime(&t);
 
 fprintf(logfile,"%s\n",asctime(local));
 //fprintf(logfileDetalhado,"%s\n\n",asctime(local));

  return;
}

//Métodos específicos#######
void printString(char *c,char *s){
	if(s != NULL)
		fprintf(logfile,"%s %s\n",c,s);
	else
		fprintf(logfile,"%s\n",c);
	
}

void printSet(int m,int n){
	fprintf(logfile,"Sequências: %d.\nBases por sequência: %d.\n",m,n);
}

void print_seqs_carregadas(int n){
	fprintf(logfile,"Sequências carregadas: %d\n",n);
}

void print_matchs(int sensos,int antisensos){
	fprintf(logfile,"Sequências senso encontradas: %d.\nSequências antisenso encontradas: %d.",sensos,antisensos);
}

void print_tempo(float tempo){
	if(tempo > 0.5)
		fprintf(logfile,"Tempo decorrido: %fs\n",tempo/1000.0);
	else
		fprintf(logfile,"Tempo decorrido: %fms\n",tempo);
	
}

//##########################

void printToLog_Runtime(float t){
	fprintf(logfile,"\nTempo de execução: %f ms.\n",t);
	//fprintf(logfileDetalhado,"\nTempo de execução: %f ms.\n",t);
}

void closeLog(){
	
  fprintf(logfile,"\n-------------------------\n");
  
   if(logfile != NULL)
      fclose(logfile);
  // if(logfileDetalhado != NULL)
//		fclose(logfileDetalhado);
	return;
}

