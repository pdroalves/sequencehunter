
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
FILE *logfile;
//FILE *logfileDetalhado;

void prepareLog();
void print_time();
void printToLog_Fim(int tiro,int raio,int fim,float x,float z);
void printToLog_Mem(float MemUsada,float MemPorIter);
void printToLog_Runtime(float t);
void printString(char*,char*);
void printSet(int,int);
void print_matchs(int,int);
void closeLog();

void prepareLog(){
  //Abre e prepara arquivo log.dat para receber mensagens de log
  
  logfile = fopen("log.dat","a");
  //logfileDetalhado = fopen("logDetalhado.dat","a");
  
  //if(ferror(logfile) != 0 || ferror(logfileDetalhado) != 0){
//  	printf("Erro! Impossível salvar log\n");
 // 	exit(1);
 // }
   
  fprintf(logfile,"-------------------------\n");
 // fprintf(logfileDetalhado,"-------------------------\n");
  print_time();
}

void print_time()
{
 
 struct tm *local;
 time_t t;
 
 t = time(NULL);
 local = localtime(&t);
 
 fprintf(logfile,"%s\n\n",asctime(local));
 //fprintf(logfileDetalhado,"%s\n\n",asctime(local));

  return;
}

//Métodos específicos#######
void printString(char *c,char *s){
	fprintf(logfile,"%s %s\n",c,s);
}

void printSet(int m,int n){
	fprintf(logfile,"Sequências: %d.\nBases por sequência: %d.\n",m,n);
}

void print_seqs_carregadas(int n){
	fprintf(logfile,"Sequências carregadas: %d\n",n);
}

void print_matchs(int sensos,int antisensos){
	fprintf(logfile,"Sequências senso encontradas: %d.\nSequências antisenso encontradas: %d.",sensos,antisensos);
}
//##########################

void printToLog_Runtime(float t){
	fprintf(logfile,"\nTempo de execução: %f ms.\n",t);
	//fprintf(logfileDetalhado,"\nTempo de execução: %f ms.\n",t);
}

void closeLog(){
   if(logfile != NULL)
      fclose(logfile);
  // if(logfileDetalhado != NULL)
//		fclose(logfileDetalhado);
	return;
}

