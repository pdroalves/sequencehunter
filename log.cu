//      log.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Arquivo com funções relativas a impressão do log de saída do programa
//
//		30/03/2012


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

void prepareLog();
void print_time();
void printString(char*,char*);
void printSet(int);
void print_matchs(int,int);
void print_tempo(float);
void closeLog();


FILE *logfile;
FILE *logfileDetalhado;

void prepareLog(){
  //Abre e prepara arquivo log.dat para receber mensagens de log
  
  logfile = fopen("log.dat","a");
  logfileDetalhado = fopen("logDetalhado.dat","a");
  
	if(ferror(logfile) != 0 || ferror(logfileDetalhado) !=0){
		printf("Erro! Impossível salvar log\n");
		exit(1);
	}
   
  fprintf(logfile,"\n\n-------------------------\n");
  fprintf(logfileDetalhado,"\n\n-------------------------\n");
 
  print_time();
}

void print_time(){
 
 struct tm *local;
 time_t t;
 
 t = time(NULL);
 local = localtime(&t);
 
 fprintf(logfile,"%s\n",asctime(local));
 fprintf(logfileDetalhado,"%s\n\n",asctime(local));

  return;
}

//Métodos específicos#######
void printString(char *c,char *s){
	if(s != NULL)
		fprintf(logfile,"%s %s\n",c,s);
	else
		fprintf(logfile,"%s\n",c);
	
}

void printSet(int n){
	fprintf(logfile,"Bases por sequência: %d.\n",n);
	fprintf(logfileDetalhado,"Bases por sequência: %d.\n",n);
}

void print_seqs_carregadas(int n){
	fprintf(logfileDetalhado,"Sequências carregadas: %d\n",n);
}

void print_matchs(int sensos,int antisensos){
	fprintf(logfile,"Sequências senso encontradas: %d.\nSequências antisenso encontradas: %d.",sensos,antisensos);
	fprintf(logfileDetalhado,"Sequências senso encontradas: %d.\nSequências antisenso encontradas: %d.",sensos,antisensos);
}

void print_tempo(float tempo){
	if(tempo > 0.5)
		fprintf(logfile,"Tempo decorrido: %fs\n",tempo/1000.0);
	else
		fprintf(logfile,"Tempo decorrido: %fms\n",tempo);
		
		fprintf(logfileDetalhado,"Tempo decorrido: %fms\n",tempo);
	
}

//##########################

void closeLog(){
	
  fprintf(logfile,"\n-------------------------\n");
  
   if(logfile != NULL)
      fclose(logfile);
   if(logfileDetalhado != NULL)
		fclose(logfileDetalhado);
	return;
}

