#include "hip/hip_runtime.h"
	//      shunter-cmd.cu
	//      
	//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
	//      
	//		Sequence Hunter 
	//		Execução via linha de comando
	//
	//		27/03/2012

	#include <stdio.h>
	#include <stdlib.h>
	#include <hip/hip_runtime.h>
	#include <hip/hip_runtime_api.h>
	#include <glib.h>
	//#include "operacoes.h"
	//#include "linkedlist.h"
	#include "estruturas.h"
	#include "aux.h"
	#include "log.h"
	#include "load_data.h"
	#include "pilha.h"
	#include "processing_data.h"
	#define SEQ_BUSCA_TAM 1000

	//###############
	static GOptionEntry entries[] = 
	  {
		//O comando "rápido" suporta 1 caracter na chamada. Se for usado mais que isso, pode dar pau
		//Entrada de posicoes
		{ "disablecuda", 'd', 0, G_OPTION_ARG_NONE, &disable_cuda, "Impede o processamento através da arquitetura CUDA", NULL },
		{ "silent", 's', 0, G_OPTION_ARG_NONE, &silent, "Execução silenciosa", NULL },
		{ "verbose", 'v', 0, G_OPTION_ARG_NONE, &verbose, "Be verbose", NULL },
		{ NULL }
	  };


	//####################
	int main (int argc,char *argv[]) {
		
	  //###########################  
	  GError *error = NULL;
	  GOptionContext *context;

	  context = g_option_context_new (NULL);
	  g_option_context_add_main_entries (context, entries,NULL);
	  if (!g_option_context_parse (context, &argc, &argv, &error))
		{
		  g_print ("option parsing failed: %s\n", error->message);
		  exit (1);
		}  
	  //##########################
	  
	  char *c;
	  int c_size;
	  int err;
	  int b1_size;
	  int b2_size;
	  int bv_size;
	  pilha p_sensos;
	  pilha p_antisensos;
	  
	  
	  //Inicializa
	  prepareLog();
	p_sensos = criar_pilha();
	p_antisensos = criar_pilha();
	 
	  
	  c = (char*)malloc((SEQ_BUSCA_TAM+1)*sizeof(char));
	  if(c == NULL){
		  printf("Erro alocando memória.\n");
		  exit(1);
	  }
	 
	////////////////////////////////////////////////////////
	////////////////// Abre arquivos de bibliotecas/////////
	////////////////////////////////////////////////////////
	if(argc == 1){
		printf("Por favor, entre uma biblioteca válida.\n");
		exit(1);
	}
	  open_file(argv,argc);
	 //////////////////////////////////
	////////////////////////////////////////////////////////
	  
	  printf("Entre a sequência: ");
	  scanf("%s",c);
	  if(c == NULL){
		  printf("Erro na leitura\n");
		  exit(1);
	  }
	  
	 if(!check_seq(c,&b1_size,&b2_size,&bv_size)){
		 printf("Sequência de busca inválida\n");
		 exit(1);
	}  
	  printString("Sequência de busca: ",c);
	  
	 c_size = b1_size+b2_size+bv_size;
	  
	if(disable_cuda){
		printf("Forçando modo OpenMP.\n");
		printString(NULL,"Forçando modo OpenMP.");
		aux(0,c,b1_size,b2_size,c_size,&p_sensos,&p_antisensos); 
	}
	else aux(check_gpu_mode(),c,b1_size,b2_size,c_size,&p_sensos,&p_antisensos);
	processar(&p_sensos,&p_antisensos);
	  
	 close_file();
	 free(c);
	destroy(&p_sensos);
	destroy(&p_antisensos);
	return 0;
	}
