#include "hip/hip_runtime.h"
	//      shunter-cmd.cu
	//      
	//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
	//      
	//		Sequence Hunter 
	//		Execução via linha de comando
	//
	//		27/03/2012

	#include <stdio.h>
	#include <stdlib.h>
	#include "estruturas.h"
	#include <hip/hip_runtime.h>
	#include <hip/hip_runtime_api.h>
	//#include "operacoes.h"
	//#include "linkedlist.h"
	#include "aux.h"
	#include "log.h"
	#include <glib.h>
	#include "load_data.h"
	#include "pilha.h"
	#include "processing_data.h"

	#define SEQ_BUSCA_TAM 1000

	//###############
	//Parametros de entrada
	static gint tiros = 1;
	static gdouble inttiros_ = 100;
	static gboolean silent = FALSE;
	static gboolean verbose = FALSE;
	static gint placa = FALSE;
	static int CUDA;

	static GOptionEntry entries[] = 
	  {
		//O comando "rápido" suporta 1 caracter na chamada. Se for usado mais que isso, pode dar pau
		//Entrada de posicoes
		{ "tiros", 't', 0, G_OPTION_ARG_INT, &tiros, "Quantidade de Tiros - Default: 1", NULL },
		{ "escolherplaca", 'e', 0, G_OPTION_ARG_NONE, &placa, "Permite que o usuário escolha qual placa de vídeo deve ser usada", NULL },
		{ "intervalodetiros", 'i', 0, G_OPTION_ARG_DOUBLE, &inttiros_, "Distancia entre cada tiro - Default: 100", NULL },
		{ "verbose", 'v', 0, G_OPTION_ARG_NONE, &verbose, "Be verbose", NULL },
		{ "silent", 's', 0, G_OPTION_ARG_NONE, &silent, "Execução silenciosa", NULL },
		{ NULL }
	  };
	//#######################
	
	// General GPU Device CUDA Initialization
int gpuDeviceInit(int devID)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
    {
        fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
        exit(-1);
    }

    if (devID < 0)
       devID = 0;
        
    if (devID > deviceCount-1)
    {
        fprintf(stderr, "\n");
        fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
        fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
        fprintf(stderr, "\n");
        return -devID;
    }

    hipDeviceProp_t deviceProp;
   hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.major < 1)
    {
        fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
        exit(-1);                                                  
    }
    
    hipSetDevice(devID);
    printf("gpuDeviceInit() CUDA Device [%d]: \"%s\n", devID, deviceProp.name);

    return devID;
}

inline int _ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct {
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] = 
	{ { 0x10,  8 },
	  { 0x11,  8 },
	  { 0x12,  8 },
	  { 0x13,  8 },
	  { 0x20, 32 },
	  { 0x21, 48 },
	  {   -1, -1 } 
	};

	int index = 0;
	while (nGpuArchCoresPerSM[index].SM != -1) {
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor) ) {
			return nGpuArchCoresPerSM[index].Cores;
		}
		index++;
	}
	printf("MapSMtoCores undefined SMversion %d.%d!\n", major, minor);
	return -1;
}

// This function returns the best GPU (with maximum GFLOPS)
int gpuGetMaxGflopsDeviceId()
{
    int current_device     = 0, sm_per_multiproc  = 0;
    int max_compute_perf   = 0, max_perf_device   = 0;
    int device_count       = 0, best_SM_arch      = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount( &device_count );
    
    // Find the best major SM Architecture GPU device
    while (current_device < device_count)
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major > 0 && deviceProp.major < 9999)
        {
            best_SM_arch = MAX(best_SM_arch, deviceProp.major);
        }
        current_device++;
    }

    // Find the best CUDA capable GPU device
    current_device = 0;
    while( current_device < device_count )
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major == 9999 && deviceProp.minor == 9999)
        {
            sm_per_multiproc = 1;
        }
        else
        {
            sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
        }
        
        int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
        
    if( compute_perf  > max_compute_perf )
    {
            // If we find GPU with SM major > 2, search only these
            if ( best_SM_arch > 2 )
            {
                // If our device==dest_SM_arch, choose this, or else pass
                if (deviceProp.major == best_SM_arch)
                {
                    max_compute_perf  = compute_perf;
                    max_perf_device   = current_device;
                 }
            }
            else
            {
                max_compute_perf  = compute_perf;
                max_perf_device   = current_device;
             }
        }
        ++current_device;
    }
    return max_perf_device;
}


// Initialization code to find the best CUDA Device
int findCudaDevice()
{
    hipDeviceProp_t deviceProp;
    int devID = 0;
    
    // Escolhe o device com maior taxa de Gflops/s
    devID = gpuGetMaxGflopsDeviceId();
    hipSetDevice( devID );
    hipGetDeviceProperties(&deviceProp, devID);
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    
    return devID;
}
// end of CUDA Helper Functions

	//####################
	int main (int argc,char *argv[]) {
		
	  //###########################  
	  GError *error = NULL;
	  GOptionContext *context;

	  context = g_option_context_new (NULL);
	  g_option_context_add_main_entries (context, entries,NULL);
	  if (!g_option_context_parse (context, &argc, &argv, &error))
		{
		  g_print ("option parsing failed: %s\n", error->message);
		  exit (1);
		}  
	  //##########################
	  
	  char *c;
	  int c_size;
	  int err;
	  int b1_size;
	  int b2_size;
	  int bv_size;
		pilha p_sensos;
		pilha p_antisensos;
	  
	  gpuDeviceInit(findCudaDevice());
	  
	  //Inicializa
	  prepareLog();
	p_sensos = criar_pilha();
	p_antisensos = criar_pilha();
	 
	  
	  c = (char*)malloc((SEQ_BUSCA_TAM+1)*sizeof(char));
	  if(c == NULL){
		  printf("Erro alocando memória.\n");
		  exit(1);
	  }
	 
	////////////////////////////////////////////////////////
	////////////////// Abre arquivos de bibliotecas/////////
	////////////////////////////////////////////////////////
	  err = open_file(argv,argc);
	  if(err == 0){
		  printf("Arquivo inválido: $s.\n",argv[1]);
		  exit(1);
	  }
	////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////
	  
	  printf("Entre a sequência: ");
	  scanf("%s",c);
	  if(c == NULL){
		  printf("Erro na leitura\n");
		  exit(1);
	  }
	  
	 if(!check_seq(c,&b1_size,&b2_size,&bv_size)){
		 printf("Sequência de busca inválida\n");
		 exit(1);
	}  
	  printString("Sequência de busca: ",c);
	  
	 c_size = b1_size+b2_size+bv_size;
	  
	  aux(1,c,b1_size,b2_size,c_size,&p_sensos,&p_antisensos);
	  processar(&p_sensos,&p_antisensos);
	  
	 close_file();
	 free(c);
	destroy(&p_sensos);
	destroy(&p_antisensos);
	return 0;
	}
