#include "hip/hip_runtime.h"
//      shunter-cmd.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Sequence Hunter 
//		Execução via linha de comando
//
//		27/03/2012

#include <stdio.h>
#include <stdlib.h>
#include "estruturas.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include "operacoes.h"
//#include "linkedlist.h"
#include "aux.h"
#include "log.h"
#include <glib.h>
#include "load_data.h"

//###############
//Parametros de entrada
static gint tiros = 1;
static gdouble inttiros_ = 100;
static gboolean silent = FALSE;
static gboolean verbose = FALSE;
static gint placa = FALSE;
static int CUDA;

static GOptionEntry entries[] = 
  {
    //O comando "rápido" suporta 1 caracter na chamada. Se for usado mais que isso, pode dar pau
    //Entrada de posicoes
    { "tiros", 't', 0, G_OPTION_ARG_INT, &tiros, "Quantidade de Tiros - Default: 1", NULL },
    { "escolherplaca", 'e', 0, G_OPTION_ARG_NONE, &placa, "Permite que o usuário escolha qual placa de vídeo deve ser usada", NULL },
    { "intervalodetiros", 'i', 0, G_OPTION_ARG_DOUBLE, &inttiros_, "Distancia entre cada tiro - Default: 100", NULL },
    { "verbose", 'v', 0, G_OPTION_ARG_NONE, &verbose, "Be verbose", NULL },
    { "silent", 's', 0, G_OPTION_ARG_NONE, &silent, "Execução silenciosa", NULL },
    { NULL }
  };
//#######################

void getDevice(int deviceCount,hipDeviceProp_t deviceProp){

  int i;
  int device;
  hipError_t erro;

     printf("Dispositivos encontrados:\n");
	  for(i=0;i<deviceCount;i++)
	    {
	      erro = hipGetDeviceProperties(&deviceProp,i);
	      g_assert(erro == hipSuccess);
	      printf("%d) %s\n",i,deviceProp.name);
	    }

	  printf("\n Usar dispositivo:");
	  scanf("%d",&device);
		
	  while(device > deviceCount)
	    {
	      printf("Dispositivo invalido\n");
	      scanf("%d",&device);
	    }
	
    //Libera o dispositivo em uso
    erro = hipDeviceReset();
    g_assert(erro == hipSuccess);
	
    //Configura qual dispositivo deve ser usado
    erro = hipSetDevice(device);
    g_assert(erro == hipSuccess);
		
    //O dispositivo foi escolhido
    erro = hipGetDeviceProperties(&deviceProp,device);
    g_assert(erro == hipSuccess);
	
    if(verbose == TRUE) printf("Dispositivo configurado para uso: %s\n\n",deviceProp.name); 
 
}

//####################
int main (int argc,char *argv[]) {
	
  //###########################  
  GError *error = NULL;
  GOptionContext *context;

  context = g_option_context_new (NULL);
  g_option_context_add_main_entries (context, entries,NULL);
  if (!g_option_context_parse (context, &argc, &argv, &error))
    {
      g_print ("option parsing failed: %s\n", error->message);
      exit (1);
    }  
  //##########################
  
  char *c;
  int c_size;
  int i;
  int deviceCount;
  int device;
  int err;
  hipDeviceProp_t tmp;
  hipDeviceProp_t deviceProp;
  hipError_t erro;
  
  //Inicializa
  prepareLog();
  hipGetDeviceCount(&deviceCount);
  
    // Função que verifica se existe um dispositivo que suporte CUDA.
	if(deviceCount == 0){
	  printf("Nao existe um dispositivo instalado na maquina que suporte CUDA\n");
	  CUDA = 0;//Sem suporte a CUDA
	}else{
		CUDA = 1;//Suporte a CUDA
	}
		
	if(placa == false){ 	
	   	for(i = 1; i < deviceCount;i++) {
			erro = hipGetDeviceProperties(&tmp,i-1);
		   	g_assert(erro == hipSuccess);
		   	erro = hipGetDeviceProperties(&deviceProp,i);	     
	    	g_assert(erro == hipSuccess);
	     	if(tmp.multiProcessorCount > deviceProp.multiProcessorCount) device = i-1;
	   		else device = i;
		  	}
	  	}else
	    	getDevice(device,deviceProp);
	    	
	/*    hipGetDeviceProperties(&deviceProp,device);
	    if(!deviceProp.deviceOverlap){
	    	printf("Placa gráfica não suporta overlaps. Não haverá ganho no uso de steams\n");
	    }*/
  	

  
  printf("Qual o tamanho da sequência?\n");
  scanf("%d",&c_size);
  
  c = (char*)malloc(c_size*sizeof(char));
  
  printf("Entre a sequência: ");
  scanf("%s",c);
  
  printString("Sequência de busca: ",c);
  
  err = open_file("sequências.dat");
  if(err == 0){
	  printf("Erro de arquivo.\n");
	  exit(1);
  }
  
  aux(CUDA);
  
  close_file();
return 0;
}
