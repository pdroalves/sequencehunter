#include "hip/hip_runtime.h"
	//      shunter-cmd.cu
	//      
	//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
	//      
	//		Sequence Hunter 
	//		Execução via linha de comando
	//
	//		27/03/2012

	#include <stdio.h>
	#include <stdlib.h>
	#include "estruturas.h"
	#include <hip/hip_runtime.h>
	#include <hip/hip_runtime_api.h>
	//#include "operacoes.h"
	//#include "linkedlist.h"
	#include "aux.h"
	#include "log.h"
	#include <glib.h>
	#include "load_data.h"
	#include "pilha.h"
	#include "processing_data.h"

	#define SEQ_BUSCA_TAM 1000

	//###############
	//Parametros de entrada
	static gboolean disable_cuda = FALSE;
	static gboolean silent = FALSE;
	static gboolean verbose = FALSE;
	

	static GOptionEntry entries[] = 
	  {
		//O comando "rápido" suporta 1 caracter na chamada. Se for usado mais que isso, pode dar pau
		//Entrada de posicoes
		{ "disable_cuda",'d',0,G_OPTION_ARG_NONE,&disable_cuda,"Impede o uso de CUDA para processamento",NULL},
		{ "verbose", 'v', 0, G_OPTION_ARG_NONE, &verbose, "Be verbose", NULL },
		{ "silent", 's', 0, G_OPTION_ARG_NONE, &silent, "Execução silenciosa", NULL },
		{ NULL }
	  };
	//#######################
	
	// General GPU Device CUDA Initialization
int gpuDeviceInit(int devID)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
    {
        fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
        return 0;
    }

    if (devID < 0)
       devID = 0;
        

    hipDeviceProp_t deviceProp;
   hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.major < 2)
    {
        fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA. Revision < 2.0.\n");
        return 0;                                                  
    }
    
    hipSetDevice(devID);
    printf("gpuDeviceInit() CUDA Device [%d]: \"%s\n", devID, deviceProp.name);

    return 1;
}

inline int _ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct {
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] = 
	{ { 0x10,  8 },
	  { 0x11,  8 },
	  { 0x12,  8 },
	  { 0x13,  8 },
	  { 0x20, 32 },
	  { 0x21, 48 },
	  {   -1, -1 } 
	};

	int index = 0;
	while (nGpuArchCoresPerSM[index].SM != -1) {
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor) ) {
			return nGpuArchCoresPerSM[index].Cores;
		}
		index++;
	}
	printf("MapSMtoCores undefined SMversion %d.%d!\n", major, minor);
	return -1;
}

// This function returns the best GPU (with maximum GFLOPS)
int gpuGetMaxGflopsDeviceId()
{
    int current_device     = 0, sm_per_multiproc  = 0;
    int max_compute_perf   = 0, max_perf_device   = 0;
    int device_count       = 0, best_SM_arch      = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount( &device_count );
    
    // Find the best major SM Architecture GPU device
    while (current_device < device_count)
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major > 0 && deviceProp.major < 9999)
        {
            best_SM_arch = MAX(best_SM_arch, deviceProp.major);
        }
        current_device++;
    }

    // Find the best CUDA capable GPU device
    current_device = 0;
    while( current_device < device_count )
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major == 9999 && deviceProp.minor == 9999)
        {
            sm_per_multiproc = 1;
        }
        else
        {
            sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
        }
        
        int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
        
    if( compute_perf  > max_compute_perf )
    {
            // If we find GPU with SM major > 2, search only these
            if ( best_SM_arch > 2 )
            {
                // If our device==dest_SM_arch, choose this, or else pass
                if (deviceProp.major == best_SM_arch)
                {
                    max_compute_perf  = compute_perf;
                    max_perf_device   = current_device;
                 }
            }
            else
            {
                max_compute_perf  = compute_perf;
                max_perf_device   = current_device;
             }
        }
        ++current_device;
    }
    return max_perf_device;
}


// Initialization code to find the best CUDA Device
int findCudaDevice()
{
    hipDeviceProp_t deviceProp;
    int devID = 0;
    
    // Escolhe o device com maior taxa de Gflops/s
    devID = gpuGetMaxGflopsDeviceId();
    hipSetDevice( devID );
    hipGetDeviceProperties(&deviceProp, devID);
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    
    return devID;
}
// end of CUDA Helper Functions

	//####################
	int main (int argc,char *argv[]) {
		
	  //###########################  
	  GError *error = NULL;
	  GOptionContext *context;

	  context = g_option_context_new (NULL);
	  g_option_context_add_main_entries (context, entries,NULL);
	  if (!g_option_context_parse (context, &argc, &argv, &error))
		{
		  g_print ("option parsing failed: %s\n", error->message);
		  exit (1);
		}  
	  //##########################
	  
	  char *c;
	  int c_size;
	  int err;
	  int b1_size;
	  int b2_size;
	  int bv_size;
		pilha p_sensos;
		pilha p_antisensos;
	  
	  CUDA = gpuDeviceInit(findCudaDevice());
	  
	  //Inicializa
	  prepareLog();
	p_sensos = criar_pilha();
	p_antisensos = criar_pilha();
	 
	  
	  c = (char*)malloc((SEQ_BUSCA_TAM+1)*sizeof(char));
	  if(c == NULL){
		  printf("Erro alocando memória.\n");
		  exit(1);
	  }
	 
	////////////////////////////////////////////////////////
	////////////////// Abre arquivos de bibliotecas/////////
	////////////////////////////////////////////////////////
	if(argc == 1){
		printf("Por favor, entre uma biblioteca válida.\n");
		exit(1);
	}
	  open_file(argv,argc);
	 //////////////////////////////////
	////////////////////////////////////////////////////////
	  
	  printf("Entre a sequência: ");
	  scanf("%s",c);
	  if(c == NULL){
		  printf("Erro na leitura\n");
		  exit(1);
	  }
	  
	 if(!check_seq(c,&b1_size,&b2_size,&bv_size)){
		 printf("Sequência de busca inválida\n");
		 exit(1);
	}  
	  printString("Sequência de busca: ",c);
	  
	 c_size = b1_size+b2_size+bv_size;
	  
	  if(disable_cuda){
		  printf("Forçando OpenMP mode.\n");
		  printString(NULL,"Forçando OpenMP mode,");
		  aux(0,c,b1_size,b2_size,c_size,&p_sensos,&p_antisensos);
	  }else
		aux(CUDA,c,b1_size,b2_size,c_size,&p_sensos,&p_antisensos);
	  processar(&p_sensos,&p_antisensos);
	  
	 close_file();
	 free(c);
	destroy(&p_sensos);
	destroy(&p_antisensos);
	return 0;
	}
