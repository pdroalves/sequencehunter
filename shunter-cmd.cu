#include "hip/hip_runtime.h"
	//      shunter-cmd.cu
	//      
	//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
	//      
	//		Sequence Hunter 
	//		Execução via linha de comando
	//
	//		27/03/2012

	#include <stdio.h>
	#include <stdlib.h>
	#include <hip/hip_runtime.h>
	#include <hip/hip_runtime_api.h>
	#include <glib.h>
	//#include "operacoes.h"
	//#include "linkedlist.h"
	#include "estruturas.h"
	#include "aux.h"
	#include "log.h"
	#include "load_data.h"
	#include "pilha.h"
	#include "processing_data.h"
	#define SEQ_BUSCA_TAM 1000

	//###############
	//Parametros de entrada
	static gint tiros = 1;
	static gdouble inttiros_ = 100;
	static gboolean silent = FALSE;
	static gboolean verbose = FALSE;
	static gint placa = FALSE;

	static GOptionEntry entries[] = 
	  {
		//O comando "rápido" suporta 1 caracter na chamada. Se for usado mais que isso, pode dar pau
		//Entrada de posicoes
		{ "tiros", 't', 0, G_OPTION_ARG_INT, &tiros, "Quantidade de Tiros - Default: 1", NULL },
		{ "escolherplaca", 'e', 0, G_OPTION_ARG_NONE, &placa, "Permite que o usuário escolha qual placa de vídeo deve ser usada", NULL },
		{ "intervalodetiros", 'i', 0, G_OPTION_ARG_DOUBLE, &inttiros_, "Distancia entre cada tiro - Default: 100", NULL },
		{ "verbose", 'v', 0, G_OPTION_ARG_NONE, &verbose, "Be verbose", NULL },
		{ "silent", 's', 0, G_OPTION_ARG_NONE, &silent, "Execução silenciosa", NULL },
		{ NULL }
	  };


	//####################
	int main (int argc,char *argv[]) {
		
	  //###########################  
	  GError *error = NULL;
	  GOptionContext *context;

	  context = g_option_context_new (NULL);
	  g_option_context_add_main_entries (context, entries,NULL);
	  if (!g_option_context_parse (context, &argc, &argv, &error))
		{
		  g_print ("option parsing failed: %s\n", error->message);
		  exit (1);
		}  
	  //##########################
	  
	  char *c;
	  int c_size;
	  int err;
	  int b1_size;
	  int b2_size;
	  int bv_size;
	  pilha p_sensos;
	  pilha p_antisensos;
	  
	  
	  //Inicializa
	  prepareLog();
	p_sensos = criar_pilha();
	p_antisensos = criar_pilha();
	 
	  
	  c = (char*)malloc((SEQ_BUSCA_TAM+1)*sizeof(char));
	  if(c == NULL){
		  printf("Erro alocando memória.\n");
		  exit(1);
	  }
	 
	////////////////////////////////////////////////////////
	////////////////// Abre arquivos de bibliotecas/////////
	////////////////////////////////////////////////////////
	if(argc == 1){
		printf("Por favor, entre uma biblioteca válida.\n");
		exit(1);
	}
	  open_file(argv,argc);
	 //////////////////////////////////
	////////////////////////////////////////////////////////
	  
	  printf("Entre a sequência: ");
	  scanf("%s",c);
	  if(c == NULL){
		  printf("Erro na leitura\n");
		  exit(1);
	  }
	  
	 if(!check_seq(c,&b1_size,&b2_size,&bv_size)){
		 printf("Sequência de busca inválida\n");
		 exit(1);
	}  
	  printString("Sequência de busca: ",c);
	  
	 c_size = b1_size+b2_size+bv_size;
	  
	  //aux(check_gpu_mode(),c,b1_size,b2_size,c_size,&p_sensos,&p_antisensos);
	  aux(0,c,b1_size,b2_size,c_size,&p_sensos,&p_antisensos);
	  processar(&p_sensos,&p_antisensos);
	  
	 close_file();
	 free(c);
	destroy(&p_sensos);
	destroy(&p_antisensos);
	return 0;
	}
