#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern "C" int gpuDeviceInit(int devID);
extern "C" inline int _ConvertSMVer2Cores(int major, int minor);
extern "C" int gpuGetMaxGflopsDeviceId();
extern "C" int findCudaDevice();

 int MAX(int A,int B){
	return A >= B ? A:B;
 }

extern "C" int check_gpu_mode(){
	
	return gpuDeviceInit(findCudaDevice());
}
extern "C" int escolhe_GPU(){
	  int num_devices, device,max_multiprocessors,max_device;
	  hipDeviceProp_t properties;	

	  hipGetDeviceCount(&num_devices);
	  max_device = 0;
		
	  if (num_devices > 1) {
			max_multiprocessors = 0;
			  
			for (device = 0; device < num_devices; device++) {//Busca a melhor GPU comparando a quantidade de multi processadores           
			  hipGetDeviceProperties(&properties, device);
			  if (max_multiprocessors < properties.multiProcessorCount) {
				max_multiprocessors = properties.multiProcessorCount;
				max_device = device;
			  }
			}
			hipSetDevice(max_device);
		  }
			
		  return max_device;
		}

extern "C" void getDevice(int deviceCount,hipDeviceProp_t deviceProp,bool verbose){

	  int i;
	  int device;
	  hipError_t erro;

		 printf("Dispositivos encontrados:\n");
		  for(i=0;i<deviceCount;i++)
			{
			  erro = hipGetDeviceProperties(&deviceProp,i);
				if(erro != hipSuccess) exit(1);
			  printf("%d) %s\n",i,deviceProp.name);
			}

		  printf("\n Usar dispositivo:");
		  scanf("%d",&device);
			
		  while(device > deviceCount)
			{
			  printf("Dispositivo invalido\n");
			  scanf("%d",&device);
			}
		
		//Libera o dispositivo em uso
		erro = hipDeviceReset();
		if(erro != hipSuccess) exit(1);
		
		//Configura qual dispositivo deve ser usado
		erro = hipSetDevice(device);
		if(erro != hipSuccess) exit(1);
			
		//O dispositivo foi escolhido
		erro = hipGetDeviceProperties(&deviceProp,device);
		if(erro != hipSuccess) exit(1);
		
		if(verbose) printf("Dispositivo configurado para uso: %s\n\n",deviceProp.name); 
	 
	}
	
	
extern "C" void copyStrToDevice(char *src,char *dst,int size){
	hipMemcpy(src,dst,size,hipMemcpyHostToDevice);
	return;
}
	
	//#######################
	
	// General GPU Device CUDA Initialization
extern "C" int gpuDeviceInit(int devID)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
    {
        fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
        return 0;
    }

    if (devID < 0)
       devID = 0;
        

    hipDeviceProp_t deviceProp;
   hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.major < 2)
    {
        fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA. Revision < 2.0.\n");
        return 0;                                                  
    }
    
    hipSetDevice(devID);
    printf("gpuDeviceInit() CUDA Device [%d]: \"%s\n", devID, deviceProp.name);

    return 1;
}

extern "C" inline int _ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct {
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] = 
	{ { 0x10,  8 },
	  { 0x11,  8 },
	  { 0x12,  8 },
	  { 0x13,  8 },
	  { 0x20, 32 },
	  { 0x21, 48 },
	  {   -1, -1 } 
	};

	int index = 0;
	while (nGpuArchCoresPerSM[index].SM != -1) {
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor) ) {
			return nGpuArchCoresPerSM[index].Cores;
		}
		index++;
	}
	printf("MapSMtoCores undefined SMversion %d.%d!\n", major, minor);
	return -1;
}

// This function returns the best GPU (with maximum GFLOPS)
extern "C" int gpuGetMaxGflopsDeviceId()
{
    int current_device     = 0, sm_per_multiproc  = 0;
    int max_compute_perf   = 0, max_perf_device   = 0;
    int device_count       = 0, best_SM_arch      = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount( &device_count );
    
    // Find the best major SM Architecture GPU device
    while (current_device < device_count)
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major > 0 && deviceProp.major < 9999)
        {
            best_SM_arch = MAX(best_SM_arch, deviceProp.major);
        }
        current_device++;
    }

    // Find the best CUDA capable GPU device
    current_device = 0;
    while( current_device < device_count )
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major == 9999 && deviceProp.minor == 9999)
        {
            sm_per_multiproc = 1;
        }
        else
        {
            sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
        }
        
        int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
        
    if( compute_perf  > max_compute_perf )
    {
            // If we find GPU with SM major > 2, search only these
            if ( best_SM_arch > 2 )
            {
                // If our device==dest_SM_arch, choose this, or else pass
                if (deviceProp.major == best_SM_arch)
                {
                    max_compute_perf  = compute_perf;
                    max_perf_device   = current_device;
                 }
            }
            else
            {
                max_compute_perf  = compute_perf;
                max_perf_device   = current_device;
             }
        }
        ++current_device;
    }
    return max_perf_device;
}


// Initialization code to find the best CUDA Device
extern "C" int findCudaDevice()
{
    hipDeviceProp_t deviceProp;
    int devID = 0;
    
    // Escolhe o device com maior taxa de Gflops/s
    devID = gpuGetMaxGflopsDeviceId();
    hipSetDevice( devID );
    hipGetDeviceProperties(&deviceProp, devID);
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    
    return devID;
}
// end of CUDA Helper Functions

