#include "hip/hip_runtime.h"
#include <stdio.h>
#include "estruturas.h"
#include "log.h"

int open_file(char*);
void close_file();
void prepare_buffer(Buffer*);
void fill_buffer(Buffer*);
int check_file_end();

FILE *f;

int open_file(char *entrada){
	f = fopen(entrada,"r+");
	return f!=NULL;
}

void close_file(){
	fclose(f);
}

void get_setup(int *m,int *n){
	fscanf(f,"%d %d",m,n);
	fgets(NULL,0,f);
}

void prepare_buffer(Buffer *b,int c){
	b->capacidade = c;
	b->seq = (char**)malloc(c*sizeof(char*));
	printString("Buffer configurado para: ","2");
}

void fill_buffer(Buffer *b,int n){
	int i;

	for(i=0;i<b->capacidade && feof(f) == 0;i++){
		b->seq[i] = (char*)malloc((n+1)*sizeof(char));
		fscanf(f,"%s",b->seq[i]);
		strcat(b->seq[i],"\0");
	}	
	
}

int check_file_end(){
	return feof(f);
}

