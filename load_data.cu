#include "hip/hip_runtime.h"
#include <stdio.h>
#include "estruturas.h"
#include "log.h"

#define TAM_MAX 10000

int open_file(char**,int);
void close_file();
int check_seq(char *seq,int *bloco1,int *bloco2,int *blocoV);
void prepare_buffer(Buffer*,int c);
void fill_buffer(Buffer*,int);

FILE **f;
int files = 0;
/* converts integer into string */

char* itoa(unsigned long num) {
        char* retstr = (char*)calloc(12, sizeof(char));
        if (sprintf(retstr, "%ld", num) > 0) {
                return retstr;
        } else {
                return NULL;
        }
}

int check_seq(char *seq,int *bloco1,int *bloco2,int *blocoV){
	int i;
	int tam;
	int tmp;
	int inv;
	
	*bloco1 = *bloco2 = *blocoV = -1;
	i = 0;
	tam = strlen(seq);
	inv = tmp = -1;
	
	while(i < tam && *bloco1 == -1){
		if(seq[i] == 'N')
			*bloco1 = i;
		i++;	
	}
	while(i < tam && tmp == -1){
		if(seq[i] != 'N')
			tmp = i;
		i++;	
	}
	*bloco2 = tam - tmp;
	*blocoV = tam - *bloco1 - *bloco2;
	while(i < tam && tmp != -1){
		if(seq[i] == 'N')
			inv = i;
		i++;
	}			
	
	if(*bloco1 == -1 || *bloco2 == -1 || inv != -1)
		return 0;
	return 1;
}

int open_file(char **entrada,int qnt){

	f = (FILE**)malloc(qnt*sizeof(FILE*));
	while(files + 1 < qnt){
		f[files] = fopen(entrada[files+1],"r+");
		files++;
	}
	return f[files-1]!=NULL;
}

void close_file(){
	int i;
	for(i=0;i<files;i++)
		fclose(f[i]);
	return;
}

void get_setup(int *n){
	char *tmp;
	//Suponho que todas as sequências nas bibliotecas tem o mesmo tamanho
	tmp = (char*)malloc(TAM_MAX*sizeof(char));
	fscanf(f[0],"%s",tmp);
	rewind(f[0]);
	*n = (int)(strlen(tmp));
	free(tmp);
	return;
}

void prepare_buffer(Buffer *b,int c){
	b->capacidade = c;
	b->seq = (char**)malloc(c*sizeof(char*));
	b->load = 0;
	printString("Buffer configurado para: ",itoa(c));
}

void fill_buffer(Buffer *b,int n){
	int i = 0;
	int j = 0;
	
	for(j=0;j < files && i < b->capacidade;j++){
		for(i=0;i < b->capacidade && feof(f[j]) == 0;i++){
				b->seq[i] = (char*)malloc((n+1)*sizeof(char));
				fscanf(f[j],"%s",b->seq[i]);
				strcat(b->seq[i],"\0");
		}
	
		b->load = i;
		if(feof(f[j]) == 1) b->load--;

		if(feof(f[j]) == 1 && b->load ==0) b->load = -1;//Arquivo acabou
	}
	return;
}

