#include "hip/hip_runtime.h"
#include <stdio.h>
#include "estruturas.h"
#include "log.h"

int open_file(char*);
void close_file();
void prepare_buffer(Buffer*);
void fill_buffer(Buffer*,int);

FILE *f;

int open_file(char *entrada){
	f = fopen(entrada,"r+");
	return f!=NULL;
}

void close_file(){
	fclose(f);
}

void get_setup(int *n){
	fscanf(f,"%d",n);
	fgets(NULL,0,f);
}

void prepare_buffer(Buffer *b,int c){
	b->capacidade = c;
	b->seq = (char**)malloc(c*sizeof(char*));
	b->load = 0;
	printString("Buffer configurado para: ","2");
}

void fill_buffer(Buffer *b,int n){
	int i;
	
	for(i=0;i < b->capacidade && feof(f) == 0;i++){
		b->seq[i] = (char*)malloc((n+1)*sizeof(char));
		fscanf(f,"%s",b->seq[i]);
		strcat(b->seq[i],"\0");
		b->load++;
	}
	b->load--;

	if(feof(f) == 1 && b->load == 0) b->load = -1;//Arquivo acabou
	
	return;
}

