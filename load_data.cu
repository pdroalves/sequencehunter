#include "hip/hip_runtime.h"
#include <stdio.h>
#include "estruturas.h"
#include "log.h"

#define TAM_MAX 10000

int open_file(char*);
void close_file();
int check_seq(char *seq,int *bloco1,int *bloco2,int *blocoV);
void prepare_buffer(Buffer*,int c);
void fill_buffer(Buffer*,int);

FILE *f;
/* converts integer into string */

char* itoa(unsigned long num) {
        char* retstr = (char*)calloc(12, sizeof(char));
        if (sprintf(retstr, "%ld", num) > 0) {
                return retstr;
        } else {
                return NULL;
        }
}

int check_seq(char *seq,int *bloco1,int *bloco2,int *blocoV){
	int i;
	int tam;
	int tmp;
	int inv;
	
	*bloco1 = *bloco2 = *blocoV = -1;
	i = 0;
	tam = strlen(seq);
	inv = tmp = -1;
	
	while(i < tam && *bloco1 == -1){
		if(seq[i] == 'N')
			*bloco1 = i;
		i++;	
	}
	while(i < tam && tmp == -1){
		if(seq[i] != 'N')
			tmp = i;
		i++;	
	}
	*bloco2 = tam - tmp;
	*blocoV = tam - *bloco1 - *bloco2;
	while(i < tam && tmp != -1){
		if(seq[i] == 'N')
			inv = i;
		i++;
	}			
	
	if(*bloco1 == -1 || *bloco2 == -1 || inv != -1)
		return 0;
	return 1;
}

int open_file(char *entrada){
	f = fopen(entrada,"r+");
	return f!=NULL;
}

void close_file(){
	fclose(f);
}

void get_setup(int *n){
	char *tmp;
	
	tmp = (char*)malloc(TAM_MAX*sizeof(char));
	fscanf(f,"%s",tmp);
	*n = (int)(strlen(tmp));
	free(tmp);
	return;
}

void prepare_buffer(Buffer *b,int c){
	b->capacidade = c;
	b->seq = (char**)malloc(c*sizeof(char*));
	b->load = 0;
	printString("Buffer configurado para: ",itoa(c));
}

void fill_buffer(Buffer *b,int n){
	int i;
	
	for(i=0;i < b->capacidade && feof(f) == 0;i++){
			b->seq[i] = (char*)malloc((n+1)*sizeof(char));
			fscanf(f,"%s",b->seq[i]);
			strcat(b->seq[i],"\0");
	}
	
	b->load = i;
	if(feof(f) == 1) b->load--;

	if(feof(f) == 1 && b->load ==0) b->load = -1;//Arquivo acabou
	
	return;
}

