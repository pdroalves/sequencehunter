#include "hip/hip_runtime.h"
#include <stdio.h>
#include <glib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "estruturas.h"
#include "log.h"

#define TAM_MAX 10000

void get_setup(int*);	
void fill_buffer(struct buffer*,int);
int check_seq(char*,int*,int*,int*);
void prepare_buffer(struct buffer*,int);
void close_file();
int open_file(char **entrada,int);
int gpuDeviceInit(int devID);
inline int _ConvertSMVer2Cores(int major, int minor);
int gpuGetMaxGflopsDeviceId();
int findCudaDevice();
int check_gpu_mode();

FILE **f;
int files = 0;
/* converts integer into string */

int check_gpu_mode(){
	
	return gpuDeviceInit(findCudaDevice());
}


char* itoa(unsigned long num) {
        char* retstr = (char*)calloc(12, sizeof(char));
        if (sprintf(retstr, "%ld", num) > 0) {
                return retstr;
        } else {
                return NULL;
        }
}

int check_seq(char *seq,int *bloco1,int *bloco2,int *blocoV){
	int i;
	int tam;
	int tmp;
	int inv;
	
	*bloco1 = *bloco2 = *blocoV = -1;
	i = 0;
	tam = strlen(seq);
	inv = tmp = -1;
	
	while(i < tam && *bloco1 == -1){
		if(seq[i] == 'N')
			*bloco1 = i;
		i++;	
	}
	while(i < tam && tmp == -1){
		if(seq[i] != 'N')
			tmp = i;
		i++;	
	}
	*bloco2 = tam - tmp;
	*blocoV = tam - *bloco1 - *bloco2;
	while(i < tam && tmp != -1){
		if(seq[i] == 'N')
			inv = i;
		i++;
	}			
	
	if(*bloco1 == -1 || *bloco2 == -1 || inv != -1)
		return 0;
	return 1;
}

int open_file(char **entrada,int qnt){
	int checks[qnt];
	int i;
	int abertos = 0;
	int tmp = 0;
	f = (FILE**)malloc(qnt*sizeof(FILE*));
	while(files < qnt && abertos+1 < qnt){
		f[files] = fopen(entrada[abertos+1],"r+");
		checks[files] = f[files]!=NULL;
		if(checks[files] == 0){
			printf("Arquivo %s não pode ser aberto.\n",entrada[files+1]);
			abertos++;
		}else{
			printf("Arquivo %s aberto.\n",entrada[abertos+1]);
			print_open_file(entrada[abertos+1]);
			files++;
			abertos++;
		}
	}
	for(i=0;i<qnt;i++) tmp += checks[files];
	return tmp==qnt;
}

void close_file(){
	int i;
	for(i=0;i<files;i++)
		fclose(f[i]);
	return;
}

void get_setup(int *n){
	char *tmp;
	//Suponho que todas as sequências nas bibliotecas tem o mesmo tamanho
	tmp = (char*)malloc(TAM_MAX*sizeof(char));
	fscanf(f[0],"%s",tmp);
	rewind(f[0]);
	*n = (int)(strlen(tmp));
	free(tmp);
	return;
}

void prepare_buffer(Buffer *b,int c){
	b->capacidade = c;
	b->seq = (char**)malloc(c*sizeof(char*));
	b->load = 0;
	printString("Buffer configurado para: ",itoa(c));
	return;
}

void fill_buffer(Buffer *b,int n){
	int i = 0;
	int j = 0;
	for(j=0;j < files && i < b->capacidade;j++){		
		while(i < b->capacidade && feof(f[j]) == 0){
				b->seq[i] = (char*)malloc((n+1)*sizeof(char));
				fscanf(f[j],"%s",b->seq[i]);
				strcat(b->seq[i],"\0");
				i++;
		}
		b->load = i;	
		if(i < b->capacidade && i!=0){ 
			b->load--;
			i = b->load;
		}
		if(feof(f[files-1]) == 1 && b->load == 0) b->load = -1;//Não há mais arquivos
	}
	return;
}

	//#######################
	
	// General GPU Device CUDA Initialization
int gpuDeviceInit(int devID)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
    {
        fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
        return 0;
    }

    if (devID < 0)
       devID = 0;
        

    hipDeviceProp_t deviceProp;
   hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.major < 2)
    {
        fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA. Revision < 2.0.\n");
        return 0;                                                  
    }
    
    hipSetDevice(devID);
    printf("gpuDeviceInit() CUDA Device [%d]: \"%s\n", devID, deviceProp.name);

    return 1;
}

inline int _ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct {
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] = 
	{ { 0x10,  8 },
	  { 0x11,  8 },
	  { 0x12,  8 },
	  { 0x13,  8 },
	  { 0x20, 32 },
	  { 0x21, 48 },
	  {   -1, -1 } 
	};

	int index = 0;
	while (nGpuArchCoresPerSM[index].SM != -1) {
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor) ) {
			return nGpuArchCoresPerSM[index].Cores;
		}
		index++;
	}
	printf("MapSMtoCores undefined SMversion %d.%d!\n", major, minor);
	return -1;
}

// This function returns the best GPU (with maximum GFLOPS)
int gpuGetMaxGflopsDeviceId()
{
    int current_device     = 0, sm_per_multiproc  = 0;
    int max_compute_perf   = 0, max_perf_device   = 0;
    int device_count       = 0, best_SM_arch      = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount( &device_count );
    
    // Find the best major SM Architecture GPU device
    while (current_device < device_count)
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major > 0 && deviceProp.major < 9999)
        {
            best_SM_arch = MAX(best_SM_arch, deviceProp.major);
        }
        current_device++;
    }

    // Find the best CUDA capable GPU device
    current_device = 0;
    while( current_device < device_count )
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major == 9999 && deviceProp.minor == 9999)
        {
            sm_per_multiproc = 1;
        }
        else
        {
            sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
        }
        
        int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
        
    if( compute_perf  > max_compute_perf )
    {
            // If we find GPU with SM major > 2, search only these
            if ( best_SM_arch > 2 )
            {
                // If our device==dest_SM_arch, choose this, or else pass
                if (deviceProp.major == best_SM_arch)
                {
                    max_compute_perf  = compute_perf;
                    max_perf_device   = current_device;
                 }
            }
            else
            {
                max_compute_perf  = compute_perf;
                max_perf_device   = current_device;
             }
        }
        ++current_device;
    }
    return max_perf_device;
}


// Initialization code to find the best CUDA Device
int findCudaDevice()
{
    hipDeviceProp_t deviceProp;
    int devID = 0;
    
    // Escolhe o device com maior taxa de Gflops/s
    devID = gpuGetMaxGflopsDeviceId();
    hipSetDevice( devID );
    hipGetDeviceProperties(&deviceProp, devID);
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    
    return devID;
}
// end of CUDA Helper Functions
