#include "hip/hip_runtime.h"
#include <stdio.h>
#include "estruturas.h"
#include "log.h"

int open_file(char*);
void close_file();
void prepare_buffer(Buffer*,int c);
void fill_buffer(Buffer*,int);

FILE *f;
/* converts integer into string */

char* itoa(unsigned long num) {
        char* retstr = (char*)calloc(12, sizeof(char));
        if (sprintf(retstr, "%ld", num) > 0) {
                return retstr;
        } else {
                return NULL;
        }
}

int open_file(char *entrada){
	f = fopen(entrada,"r+");
	return f!=NULL;
}

void close_file(){
	fclose(f);
}

void get_setup(int *n){
	fscanf(f,"%d",n);
	fgets(NULL,0,f);
}

void prepare_buffer(Buffer *b,int c){
	b->capacidade = c;
	b->seq = (char**)malloc(c*sizeof(char*));
	b->load = 0;
	printString("Buffer configurado para: ",itoa(c));
}

void fill_buffer(Buffer *b,int n){
	int i;
	
	for(i=0;i < b->capacidade && feof(f) == 0;i++){
			b->seq[i] = (char*)malloc((n+1)*sizeof(char));
			fscanf(f,"%s",b->seq[i]);
			strcat(b->seq[i],"\0");
	}
	
	b->load = i;
	if(feof(f) == 1) b->load--;

	if(feof(f) == 1 && b->load ==0) b->load = -1;//Arquivo acabou
	
	return;
}

