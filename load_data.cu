#include "hip/hip_runtime.h"
#include <stdio.h>
#include "estruturas.h"
#include "log.h"

#define TAM_MAX 10000

int open_file(char**,int);
void close_file();
int check_seq(char *seq,int *bloco1,int *bloco2,int *blocoV);
void prepare_buffer(Buffer*,int c);
void fill_buffer(Buffer*,int);

FILE **f;
int files = 0;
/* converts integer into string */

char* itoa(unsigned long num) {
        char* retstr = (char*)calloc(12, sizeof(char));
        if (sprintf(retstr, "%ld", num) > 0) {
                return retstr;
        } else {
                return NULL;
        }
}

int check_seq(char *seq,int *bloco1,int *bloco2,int *blocoV){
	int i;
	int tam;
	int tmp;
	int inv;
	
	*bloco1 = *bloco2 = *blocoV = -1;
	i = 0;
	tam = strlen(seq);
	inv = tmp = -1;
	
	while(i < tam && *bloco1 == -1){
		if(seq[i] == 'N')
			*bloco1 = i;
		i++;	
	}
	while(i < tam && tmp == -1){
		if(seq[i] != 'N')
			tmp = i;
		i++;	
	}
	*bloco2 = tam - tmp;
	*blocoV = tam - *bloco1 - *bloco2;
	while(i < tam && tmp != -1){
		if(seq[i] == 'N')
			inv = i;
		i++;
	}			
	
	if(*bloco1 == -1 || *bloco2 == -1 || inv != -1)
		return 0;
	return 1;
}

int open_file(char **entrada,int qnt){
	int checks[qnt];
	int i;
	int abertos = 0;
	int tmp = 0;
	f = (FILE**)malloc(qnt*sizeof(FILE*));
	while(files < qnt && abertos+1 < qnt){
		f[files] = fopen(entrada[abertos+1],"r+");
		checks[files] = f[files]!=NULL;
		if(checks[files] == 0){
			printf("Arquivo %s não pode ser aberto.\n",entrada[files+1]);
			abertos++;
		}else{
			printf("Arquivo %s aberto.\n",entrada[abertos+1]);
			print_open_file(entrada[abertos+1]);
			files++;
			abertos++;
		}
	}
	for(i=0;i<qnt;i++) tmp += checks[files];
	return tmp==qnt;
}

void close_file(){
	int i;
	for(i=0;i<files;i++)
		fclose(f[i]);
	return;
}

void get_setup(int *n){
	char *tmp;
	//Suponho que todas as sequências nas bibliotecas tem o mesmo tamanho
	tmp = (char*)malloc(TAM_MAX*sizeof(char));
	fscanf(f[0],"%s",tmp);
	rewind(f[0]);
	*n = (int)(strlen(tmp));
	free(tmp);
	return;
}

void prepare_buffer(Buffer *b,int c){
	b->capacidade = c;
	b->seq = (char**)malloc(c*sizeof(char*));
	b->load = 0;
	printString("Buffer configurado para: ",itoa(c));
	return;
}

void fill_buffer(Buffer *b,int n){
	int i = 0;
	int j = 0;
	for(j=0;j < files && i < b->capacidade;j++){		
		while(i < b->capacidade && feof(f[j]) == 0){
				b->seq[i] = (char*)malloc((n+1)*sizeof(char));
				fscanf(f[j],"%s",b->seq[i]);
				strcat(b->seq[i],"\0");
				i++;
		}
		b->load = i;	
		if(i < b->capacidade && i!=0){ 
			b->load--;
			i = b->load;
		}
		if(feof(f[files-1]) == 1 && b->load ==0) b->load = -1;//Não há mais arquivos
	}
	return;
}

