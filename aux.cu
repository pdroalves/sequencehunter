//      aux.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Arquivo com funções auxiliares para a execução da busca e análise da biblioteca
//
//		27/03/2012

#include <hip/hip_runtime.h>
#include "estruturas.h"
#include "load_data.h"
#include "operacoes.h"
#include "busca.h"

void setup_for_cuda(char*,vgrafo*,vgrafo*,vgrafo*, vgrafo*);

void aux(int CUDA,char *c){
	
	int m;//Quantidade sequências
	int n;//Elementos por sequência
	int buffer_size = 2;
	int *matchs;
	int *d_matchs;
	int i;
	Buffer buffer;
	char **d_buffer;
	char **tmp;
	char *check;
	vgrafo *d_a;
	vgrafo *d_c;
	vgrafo *d_g;
	vgrafo *d_t;
	
	get_setup(&m,&n);
	
	check = (char*)malloc(n*sizeof(char));
	hipMalloc((void**)&d_a,sizeof(vgrafo));
    hipMalloc((void**)&d_c,sizeof(vgrafo));
    hipMalloc((void**)&d_g,sizeof(vgrafo));
    hipMalloc((void**)&d_t,sizeof(vgrafo));
	hipMalloc((void**)&d_matchs,buffer_size*sizeof(int));
	matchs = (int*)calloc(0,buffer_size*sizeof(int));
	hipMemcpy(d_matchs,matchs,buffer_size*sizeof(int),hipMemcpyHostToDevice);
	d_buffer = (char**)malloc(buffer_size*sizeof(char*));
	for(i=0;i<buffer_size;i++)
		hipMalloc((void**)&(d_buffer[i]),n*sizeof(char));	
		
	//Inicializa
	prepare_buffer(&buffer,buffer_size);
	//hipMemset(d_matchs,9,buffer_size*sizeof(int));
	
	setup_for_cuda(c,d_a,d_c,d_g,d_t);
	
	while( check_file_end()== 0){
		//Realiza loop enquanto existirem sequências para encher o buffer
		fill_buffer(&buffer,n);
		for(i=0;i<buffer_size;i++)
			hipMemcpy(d_buffer[i],buffer.seq[i],n*sizeof(char), hipMemcpyHostToDevice);
		k_busca<<<1,buffer_size>>>(d_matchs,d_buffer,n,d_a,d_c,d_g,d_t);
		
		hipMemcpy(check,d_buffer[0],n*sizeof(char),hipMemcpyDeviceToHost);
		hipMemcpy(matchs,d_matchs,buffer_size*sizeof(int),hipMemcpyDeviceToHost);
	}
	
	hipFree(d_a);
	hipFree(d_c);
	hipFree(d_g);
	hipFree(d_t);
	hipFree(d_matchs);
	free(matchs);
	
	return;

}
void setup_for_cuda(char *seq,vgrafo *d_a,vgrafo *d_c,vgrafo *d_g, vgrafo *d_t){
	//Recebe um vetor de caracteres com o padrão a ser procurado
	//Recebe ponteiros para os quatro vértices do grafo já na memória da GPU
	char *d_senso;
	char *d_antisenso;
	int size = strlen(seq);
	
	//Aloca memória na GPU
    hipMalloc((void**)&d_senso,size*sizeof(char));
    hipMalloc((void**)&d_antisenso,size*sizeof(char));
    
    //hipMemcpy(d_seq,seq,size*sizeof(char),hipMemcpyHostToDevice);
    //hipMemcpy(d_seq_antisenso,get_antisenso(seq),size*sizeof(char),hipMemcpyHostToDevice);
    
    //Configura grafos direto na memória da GPU
	//set_grafo<<<1,1>>>(d_seq,d_seq_antisenso,d_a,d_c,d_g,d_t);
	
	hipFree(d_senso);
	hipFree(d_antisenso);
	return;
}


