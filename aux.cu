#include "hip/hip_runtime.h"
//      aux.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Arquivo com funções auxiliares para a execução da busca e análise da biblioteca
//
//		27/03/2012

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <omp.h> 
#include "estruturas.h"
#include "load_data.h"
#include "operacoes.h"
#include "busca.h"
#include "cuda_stack.h"
#include "log.h"
#include "pilha.h"

#define buffer_size 10//Capacidade máxima do buffer
__constant__ char *d_buffer[buffer_size];
int buffer_flag = 1;//0 se o buffer já foi carregado, 1 se estiver sendo carregado.

void setup_for_cuda(char*,vgrafo*,vgrafo*,vgrafo*, vgrafo*);
void load_buffer(Buffer *b,char** s,int n);

void aux(int CUDA,char *c,const int bloco1,const int bloco2,const int blocos){
	
	int m;//Quantidade sequências
	int n;//Elementos por sequência
	int *matchs;
	int *d_matchs;
	int i;
	int blocoV;
	Buffer buffer;
	vgrafo *d_a;
	vgrafo *d_c;
	vgrafo *d_g;
	vgrafo *d_t;
	char **s;
	char **d_sensos;
	char **d_antisensos;
	char **d_sensos_hold;
	char **d_antisensos_hold;
	char *tmp;
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
	pilha p_sensos;
	pilha p_antisensos;
	int p_sensos_size;
	int p_antisensos_size;
	
	get_setup(&m,&n);
	blocoV = blocos - bloco1 - bloco2+1;
	hipMalloc((void**)&d_a,sizeof(vgrafo));
    hipMalloc((void**)&d_c,sizeof(vgrafo));
    hipMalloc((void**)&d_g,sizeof(vgrafo));
    hipMalloc((void**)&d_t,sizeof(vgrafo));
    
    hipMalloc((void**)&s,buffer_size*sizeof(char*));
    
	hipMalloc((void**)&d_sensos,buffer_size*sizeof(char*));
	hipMalloc((void**)&d_antisensos,buffer_size*sizeof(char*));
	tmp = (char*)malloc(blocoV*sizeof(char));
	d_sensos_hold = (char**)malloc(buffer_size*sizeof(char*));
	d_antisensos_hold = (char**)malloc(buffer_size*sizeof(char*));
	for(i=0;i<buffer_size;i++){
		hipMalloc((void**)&d_sensos_hold[i],blocoV*sizeof(char));//Aloco n+1 posicoes para as bases e +1 para processamento interno no arquivo cuda_stack.cu
		hipMalloc((void**)&d_antisensos_hold[i],blocoV*sizeof(char));//Aloco n+1 posicoes para as bases e +1 para processamento interno no arquivo cuda_stack.cu
	}
	hipMemcpy(d_sensos,d_sensos_hold,buffer_size*sizeof(char*),hipMemcpyHostToDevice);
	hipMemcpy(d_antisensos,d_antisensos_hold,buffer_size*sizeof(char*),hipMemcpyHostToDevice);
	
	
	hipMalloc((void**)&d_matchs,buffer_size*sizeof(int));
	matchs = (int*)calloc(0,buffer_size*sizeof(int));
	hipMemcpy(d_matchs,matchs,buffer_size*sizeof(int),hipMemcpyHostToDevice);
	
	p_sensos = criar_pilha();
	p_antisensos = criar_pilha();
		
	//Inicializa
	prepare_buffer(&buffer,buffer_size);
	setup_for_cuda(c,d_a,d_c,d_g,d_t);
	
	printString("Dados inicializados.\n",NULL);
	printSet(m,n);
	printString("Iniciando iterações:\n",NULL);
	
	#pragma omp parallel sections num_threads(2) shared(buffer)
	{
	#pragma omp section
	{	
	load_buffer(&buffer,s,n);
	printf("Buffer vazio.\n");
	}
	
	#pragma omp section
	{
	while( buffer.load != -1){
		//Realiza loop enquanto existirem sequências para encher o buffer
		while(buffer_flag != 0){
		}//Espera o buffer ser carregado
		if(buffer.load != -1){
		dim3 dimBlock(buffer_size>=buffer.load?buffer.load:buffer_size);
		dim3 dimGrid(1);
		k_busca<<<dimGrid,dimBlock>>>(bloco1,bloco2,blocos,s,d_a,d_c,d_g,d_t);
		
		k_recupera<<<dimGrid,dimBlock>>>(s,d_sensos,d_antisensos);
		for(i=0;i<buffer_size;i++){//Copia sequências senso e antisenso encontradas
			hipMemcpy(tmp,d_sensos_hold[i],sizeof(char),hipMemcpyDeviceToHost);
			if(tmp[0] != '\0'){//Antes de copiar, verifica se existe alguma sequência útil
				hipMemcpy(tmp,d_buffer[i],blocoV*sizeof(char),hipMemcpyDeviceToHost);
				//printf("%s\n",tmp);
				empilha(tmp,&p_sensos);
			}
			
			hipMemcpy(tmp,d_antisensos_hold[i],sizeof(char),hipMemcpyDeviceToHost);
			if(tmp[0] != '\0'){//Antes de copiar, verifica se existe alguma sequência útil
				hipMemcpy(tmp,d_buffer[i],blocoV*sizeof(char),hipMemcpyDeviceToHost);
				//printf("%s\n",tmp);
				empilha(tmp,&p_antisensos);
			}
		}
		buffer.load = 0;//Avisando sobre buffer vazio
	}
	}
	
	
	}
	
	}
	p_sensos_size = tamanho_da_pilha(&p_sensos);
	p_antisensos_size = tamanho_da_pilha(&p_antisensos);
	printf("Sensos: %d.\nAntisensos: %d.\n",p_sensos_size,p_antisensos_size);
	
	for(i=0;i<p_sensos_size;i++){
		printf("%s\n",desempilha(&p_sensos));
	}
	
	printString("Iterações terminadas.",NULL);
	print_matchs(p_sensos_size,p_antisensos_size);
	
	hipFree(d_a);
	hipFree(d_c);
	hipFree(d_g);
	hipFree(d_t);
	hipFree(d_matchs);
	free(matchs);
	hipDeviceReset();
	return;

}
void setup_for_cuda(char *seq,vgrafo *d_a,vgrafo *d_c,vgrafo *d_g, vgrafo *d_t){
	//Recebe um vetor de caracteres com o padrão a ser procurado
	//Recebe ponteiros para os quatro vértices do grafo já na memória da GPU
	char *d_senso;
	char *d_antisenso;
	int size = strlen(seq)+1;
	
	//Aloca memória na GPU
    hipMalloc((void**)&d_senso,size*sizeof(char));
    hipMalloc((void**)&d_antisenso,size*sizeof(char));
    
    hipMemcpy(d_senso,seq,size*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_antisenso,get_antisenso(seq),size*sizeof(char),hipMemcpyHostToDevice);
    
    //Configura grafos direto na memória da GPU
	set_grafo<<<1,1>>>(d_senso,d_antisenso,d_a,d_c,d_g,d_t);
	printString("Grafo de busca contigurado.",NULL);
	hipFree(d_senso);
	hipFree(d_antisenso);
	return;
}

void load_buffer(Buffer *b,char** s,int n){
	//Essa função fica em loop até que todas as sequências tenham sido lidas e processadas
	int i;
	
	fill_buffer(b,buffer_size);//Enche o buffer e guarda a quantidade de sequências carregadas.
	print_seqs_carregadas(b->load);
	
	for(i=0;i<buffer_size;i++)
			hipMalloc((void**)&d_buffer[i],(n+2)*sizeof(char));//Aloco n+1 posicoes para as bases e +1 para processamento interno no arquivo cuda_stack.cu
	
	while(b->load != -1){
		///////////////////////////////////
		buffer_flag = 1;//Sinal fechado////
		///////////////////////////////////
		fill_buffer(b,buffer_size);//Enche o buffer e guarda a quantidade de sequências carregadas.
		if(b->load != -1){
		print_seqs_carregadas(b->load);
		
		for(i=0;i<buffer_size;i++){
			hipMemcpy(d_buffer[i],b->seq[i],(n+1)*sizeof(char),hipMemcpyHostToDevice);
		}
		hipMemcpy(s,d_buffer,buffer_size*sizeof(char*),hipMemcpyHostToDevice);
		
		//////////////////////////////////
		buffer_flag = 0;//Sinal aberto////
		//////////////////////////////////
		
		while(b->load > 0){
		}
		}else{
			//////////////////////////////////
			buffer_flag = 0;//Sinal aberto////
			//////////////////////////////////
			printf("Não há mais sequências.\n");
			return;
		}
		
	}
	return;
}


