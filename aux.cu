//      aux.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Arquivo com funções auxiliares para a execução da busca e análise da biblioteca
//
//		27/03/2012

#include <hip/hip_runtime.h>
#include "estruturas.h"
#include "load_data.h"
#include "operacoes.h"
#include "busca.h"
#include <stdio.h>

#define buffer_size 1
__constant__ char *d_buffer[buffer_size];

void setup_for_cuda(char*,vgrafo*,vgrafo*,vgrafo*, vgrafo*);

void aux(int CUDA,char *c){
	
	int m;//Quantidade sequências
	int n;//Elementos por sequência
	int *matchs;
	int *d_matchs;
	int i;
	Buffer buffer;
	char **tmp;
	char *check;
	vgrafo *d_a;
	vgrafo *d_c;
	vgrafo *d_g;
	vgrafo *d_t;
	char **s;
	
	get_setup(&m,&n);
	
	check = (char*)malloc((n+1)*sizeof(char));
	hipMalloc((void**)&d_a,sizeof(vgrafo));
    hipMalloc((void**)&d_c,sizeof(vgrafo));
    hipMalloc((void**)&d_g,sizeof(vgrafo));
    hipMalloc((void**)&d_t,sizeof(vgrafo));
    
    hipMalloc((void**)&d_matchs,buffer_size*sizeof(int));
    hipMalloc((void**)&s,buffer_size*sizeof(char*));
	matchs = (int*)calloc(0,buffer_size*sizeof(int));
	hipMemcpy(d_matchs,matchs,buffer_size*sizeof(int),hipMemcpyHostToDevice);
		
	//Inicializa
	prepare_buffer(&buffer,buffer_size);
	setup_for_cuda(c,d_a,d_c,d_g,d_t);
	
	while( check_file_end()== 0){
		//Realiza loop enquanto existirem sequências para encher o buffer
		fill_buffer(&buffer,n);
		for(i=0;i<buffer_size;i++){
			hipMalloc((void**)&d_buffer[i],(n+1)*sizeof(char));
			hipMemcpy(d_buffer[i],buffer.seq[i],(n+1)*sizeof(char),hipMemcpyHostToDevice);
		}
		hipMemcpy(s,d_buffer,buffer_size*sizeof(char*),hipMemcpyHostToDevice);
		k_busca<<<1,buffer_size>>>(d_matchs,s,strlen(c),d_a,d_c,d_g,d_t);
		//k_teste<<<1,buffer_size>>>(d_matchs,d_buffer,n,d_a,d_c,d_g,d_t);
		//k_printf<<<1,1>>>(s);
		
		hipMemcpy(matchs,d_matchs,buffer_size*sizeof(int),hipMemcpyDeviceToHost);
		for(i=0;i<buffer_size;i++)
			printf("%s:-> %d\n",buffer.seq[i],matchs[i]);
	}
	
	hipFree(d_a);
	hipFree(d_c);
	hipFree(d_g);
	hipFree(d_t);
	hipFree(d_matchs);
	free(matchs);
	
	return;

}
void setup_for_cuda(char *seq,vgrafo *d_a,vgrafo *d_c,vgrafo *d_g, vgrafo *d_t){
	//Recebe um vetor de caracteres com o padrão a ser procurado
	//Recebe ponteiros para os quatro vértices do grafo já na memória da GPU
	char *d_senso;
	char *d_antisenso;
	int size = strlen(seq)+1;
	
	//Aloca memória na GPU
    hipMalloc((void**)&d_senso,size*sizeof(char));
    hipMalloc((void**)&d_antisenso,size*sizeof(char));
    
    hipMemcpy(d_senso,seq,size*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_antisenso,get_antisenso(seq),size*sizeof(char),hipMemcpyHostToDevice);
    
    //Configura grafos direto na memória da GPU
	set_grafo<<<1,1>>>(d_senso,d_antisenso,d_a,d_c,d_g,d_t);
	printf("Grafo contigurado\n");
	hipFree(d_senso);
	hipFree(d_antisenso);
	return;
}


