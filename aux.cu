//      aux.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Arquivo com funções auxiliares para a execução da busca e análise da biblioteca
//
//		27/03/2012

#include <hip/hip_runtime.h>
#include "estruturas.h"
#include "load_data.h"
#include "operacoes.h"
#include "busca.h"

void setup_for_cuda(char*,vgrafo*,vgrafo*,vgrafo*, vgrafo*);

void aux(int CUDA,char *c){
	
	int m;//Quantidade sequências
	int n;//Elementos por sequência
	int buffer_size = 2;
	Buffer b;
	char *tmp;
	vgrafo *d_a;
	vgrafo *d_c;
	vgrafo *d_g;
	vgrafo *d_t;
	
	hipMalloc((void**)&d_a,sizeof(vgrafo));
    hipMalloc((void**)&d_c,sizeof(vgrafo));
    hipMalloc((void**)&d_g,sizeof(vgrafo));
    hipMalloc((void**)&d_t,sizeof(vgrafo));
	
	//Inicializa
	get_setup(&m,&n);
	prepare_buffer(&b,buffer_size);
	
	setup_for_cuda(c,d_a,d_c,d_g,d_t);
	
	while( check_file_end()== 0){
		//Realiza loop enquanto existirem sequências para encher o buffer
		fill_buffer(&b,n);
		 tmp = get_antisenso(b.seq[0]);
	}
	
	hipFree(d_a);
	hipFree(d_c);
	hipFree(d_g);
	hipFree(d_t);
	
	return;

}
void setup_for_cuda(char *seq,vgrafo *d_a,vgrafo *d_c,vgrafo *d_g, vgrafo *d_t){
	//Recebe um vetor de caracteres com o padrão a ser procurado
	//Recebe ponteiros para os quatro vértices do grafo já na memória da GPU
	char *d_senso;
	char *d_antisenso;
	int size = strlen(seq);
	
	//Aloca memória na GPU
    hipMalloc((void**)&d_senso,size*sizeof(char));
    hipMalloc((void**)&d_antisenso,size*sizeof(char));
    
    //hipMemcpy(d_seq,seq,size*sizeof(char),hipMemcpyHostToDevice);
    //hipMemcpy(d_seq_antisenso,get_antisenso(seq),size*sizeof(char),hipMemcpyHostToDevice);
    
    //Configura grafos direto na memória da GPU
	//set_grafo<<<1,1>>>(d_seq,d_seq_antisenso,d_a,d_c,d_g,d_t);
	
	hipFree(d_senso);
	hipFree(d_antisenso);
	return;
}


