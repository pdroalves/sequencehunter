//      aux.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Arquivo com funções auxiliares para a execução da busca e análise da biblioteca
//
//		27/03/2012

#include <hip/hip_runtime.h>
#include "estruturas.h"
#include "load_data.h"
#include "operacoes.h"
#include "busca.h"

void setup_for_cuda(char*);

void aux(int CUDA){
	
	int m;//Quantidade sequências
	int n;//Elementos por sequência
	int buffer_size = 2;
	Buffer b;
	char *tmp;
	vgrafo *d_a;
	vgrafo *d_c;
	vgrafo *d_g;
	vgrafo *d_t;
	
	//Inicializa
	get_setup(&m,&n);
	prepare_buffer(&b,buffer_size);
	
	while( check_file_end()== 0){
		//Realiza loop enquanto existirem sequências para encher o buffer
		fill_buffer(&b,n);
		 tmp = get_antisenso(b.seq[0]);
	}

}
void setup_for_cuda(char *seq,vgrafo *d_a,vgrafo *d_c,vgrafo *d_g, vgrafo *d_t){
	
	char *d_seq;
	char *d_seq_antisenso;
	
	//Aloca memória na GPU
	hipMalloc((void**)&d_a,sizeof(vgrafo));
    hipMalloc((void**)&d_c,sizeof(vgrafo));
    hipMalloc((void**)&d_g,sizeof(vgrafo));
    hipMalloc((void**)&d_t,sizeof(vgrafo));
    hipMalloc((void**)&d_seq,conta_posicoes(seq)*sizeof(char));
    hipMalloc((void**)&d_seq_antisenso,conta_posicoes(seq)*sizeof(char));
    
    hipMemcpy(d_seq,seq,conta_posicoes(seq)*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_seq_antisenso,get_antisenso(seq),conta_posicoes(seq)*sizeof(char),hipMemcpyHostToDevice);
    
    //Configura grafos direto na memória da GPU
    build_grafo<<<1,1>>>(&d_a,&d_c,&d_g,&d_t);
	set_grafo<<<1,1>>>(d_seq,d_seq_antisenso,&d_a,&d_c,&d_g,&d_t);
		
	return;
}


