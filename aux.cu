//      aux.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Arquivo com funções auxiliares para a execução da busca e análise da biblioteca
//
//		27/03/2012

#include <hip/hip_runtime.h>
#include "estruturas.h"
#include "load_data.h"
#include "operacoes.h"
#include "busca.h"
#include <stdio.h>

#define buffer_size 1
__constant__ char *d_buffer[buffer_size];

void setup_for_cuda(char*,vgrafo*,vgrafo*,vgrafo*, vgrafo*);

void aux(int CUDA,char *c,const int bloco1,const int bloco2,const int blocos){
	
	int m;//Quantidade sequências
	int n;//Elementos por sequência
	int *matchs;
	int *d_matchs;
	int i;
	Buffer buffer;
	vgrafo *d_a;
	vgrafo *d_c;
	vgrafo *d_g;
	vgrafo *d_t;
	char **s;
	
	get_setup(&m,&n);
	
	hipMalloc((void**)&d_a,sizeof(vgrafo));
    hipMalloc((void**)&d_c,sizeof(vgrafo));
    hipMalloc((void**)&d_g,sizeof(vgrafo));
    hipMalloc((void**)&d_t,sizeof(vgrafo));
    
    hipMalloc((void**)&d_matchs,buffer_size*sizeof(int));
    hipMalloc((void**)&s,buffer_size*sizeof(char*));
	matchs = (int*)calloc(0,buffer_size*sizeof(int));
	hipMemcpy(d_matchs,matchs,buffer_size*sizeof(int),hipMemcpyHostToDevice);
		
	//Inicializa
	prepare_buffer(&buffer,buffer_size);
	setup_for_cuda(c,d_a,d_c,d_g,d_t);
	
	while( check_file_end_and_fill_buffer(&buffer,n)== 0){
		//Realiza loop enquanto existirem sequências para encher o buffer
		for(i=0;i<buffer_size;i++){
			hipMalloc((void**)&d_buffer[i],(n+2)*sizeof(char));//Aloco n+1 posicoes para as bases e +1 para processamento interno no arquivo cuda_stack.cu
			hipMemcpy(d_buffer[i],buffer.seq[i],(n+1)*sizeof(char),hipMemcpyHostToDevice);
		}
		hipMemcpy(s,d_buffer,buffer_size*sizeof(char*),hipMemcpyHostToDevice);
		k_busca<<<1,buffer_size>>>(bloco1,bloco2,blocos,s,d_a,d_c,d_g,d_t);

	}
	
	hipFree(d_a);
	hipFree(d_c);
	hipFree(d_g);
	hipFree(d_t);
	hipFree(d_matchs);
	free(matchs);
	
	return;

}
void setup_for_cuda(char *seq,vgrafo *d_a,vgrafo *d_c,vgrafo *d_g, vgrafo *d_t){
	//Recebe um vetor de caracteres com o padrão a ser procurado
	//Recebe ponteiros para os quatro vértices do grafo já na memória da GPU
	char *d_senso;
	char *d_antisenso;
	int size = strlen(seq)+1;
	
	//Aloca memória na GPU
    hipMalloc((void**)&d_senso,size*sizeof(char));
    hipMalloc((void**)&d_antisenso,size*sizeof(char));
    
    hipMemcpy(d_senso,seq,size*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_antisenso,get_antisenso(seq),size*sizeof(char),hipMemcpyHostToDevice);
    
    //Configura grafos direto na memória da GPU
	set_grafo<<<1,1>>>(d_senso,d_antisenso,d_a,d_c,d_g,d_t);
	printf("Grafo contigurado\n");
	hipFree(d_senso);
	hipFree(d_antisenso);
	return;
}


