#include "hip/hip_runtime.h"
//      aux.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Arquivo com funções auxiliares para a execução da busca e análise da biblioteca
//
//		27/03/2012

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <omp.h> 
#include "estruturas.h"
#include "load_data.h"
#include "operacoes.h"
#include "busca.h"
#include "log.h"
#include "pilha.h"

#define buffer_size 512 //Capacidade máxima do buffer
__constant__ char *d_buffer[buffer_size];
int buffer_flag;//0 se o buffer já foi carregado, 1 se estiver sendo carregado.


void setup_for_cuda(char*,vgrafo*,vgrafo*,vgrafo*, vgrafo*);
void load_buffer(Buffer *b,char** s,int n);
void cudaIteracoes(int bloco1,int bloco2,int blocoV,int n,vgrafo *d_a,vgrafo *d_c,vgrafo *d_g,vgrafo *d_t,pilha *p_senso,pilha *p_antisenso);
	
void aux(int CUDA,char *c,const int bloco1,const int bloco2,const int blocos,pilha *p_sensos,pilha *p_antisensos){
	
	int n;//Elementos por sequência
	vgrafo *d_a;
	vgrafo *d_c;
	vgrafo *d_g;
	vgrafo *d_t;
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float tempo;
	
	get_setup(&n);
	
	hipMalloc((void**)&d_a,sizeof(vgrafo));
    hipMalloc((void**)&d_c,sizeof(vgrafo));
    hipMalloc((void**)&d_g,sizeof(vgrafo));
    hipMalloc((void**)&d_t,sizeof(vgrafo));
		
	//Inicializa
	setup_for_cuda(c,d_a,d_c,d_g,d_t);
	
	printString("Dados inicializados.\n",NULL);
	printSet(n);
	printString("Iniciando iterações:\n",NULL);
	
    hipEventRecord(start,0);
	cudaIteracoes(bloco1,bloco2,blocos,n,d_a,d_c,d_g,d_t,p_sensos,p_antisensos);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&tempo,start,stop);
    
	printString("Iterações terminadas. Tempo: ",NULL);
	print_tempo(tempo);
	
	hipFree(d_a);
	hipFree(d_c);
	hipFree(d_g);
	hipFree(d_t);
	hipDeviceReset();
	return;

}
void setup_for_cuda(char *seq,vgrafo *d_a,vgrafo *d_c,vgrafo *d_g, vgrafo *d_t){
	//Recebe um vetor de caracteres com o padrão a ser procurado
	//Recebe ponteiros para os quatro vértices do grafo já na memória da GPU
	char *d_senso;
	char *d_antisenso;
	int size = strlen(seq)+1;
	
	//Aloca memória na GPU
    hipMalloc((void**)&d_senso,size*sizeof(char));
    hipMalloc((void**)&d_antisenso,size*sizeof(char));
    
    hipMemcpy(d_senso,seq,size*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_antisenso,get_antisenso(seq),size*sizeof(char),hipMemcpyHostToDevice);
    
    //Configura grafos direto na memória da GPU
	set_grafo<<<1,1>>>(d_senso,d_antisenso,d_a,d_c,d_g,d_t);
	printString("Grafo de busca contigurado.",NULL);
	hipFree(d_senso);
	hipFree(d_antisenso);
	return;
}

void load_buffer(Buffer *b,char** s,int n){
	int i;
	
	if(b->load == 0){//Se for >0 ainda existem elementos no buffer anterior e se for == -1 não há mais elementos a serem carregados
		fill_buffer(b,buffer_size);//Enche o buffer e guarda a quantidade de sequências carregadas.
		if(b->load != -1){
			print_seqs_carregadas(b->load);
			//printf("%s\n",b->seq[0]);
			for(i=0;i<buffer_size;i++)
				hipMemcpy(d_buffer[i],b->seq[i],(n+1)*sizeof(char),hipMemcpyHostToDevice);
			
			hipMemcpy(s,d_buffer,buffer_size*sizeof(char*),hipMemcpyHostToDevice);
		}
		//////////////////////////////////
		buffer_flag = 0;//Sinal aberto////
		//////////////////////////////////
			
	}
		
	
	return;
}

void cudaIteracoes(int bloco1,int bloco2,int blocos,int n,vgrafo *d_a,vgrafo *d_c,vgrafo *d_g,vgrafo *d_t,pilha *p_sensos,pilha *p_antisensos){
	
	Buffer buffer;
	char **s;
	char *tmp;
	int i;
	int blocoV = blocos - bloco1 - bloco2+1;

	//Inicializa buffer
	prepare_buffer(&buffer,buffer_size);

	//Aloca memória
	for(i=0;i<buffer_size;i++)
			hipMalloc((void**)&d_buffer[i],(n+2)*sizeof(char));//Aloco n+1 posicoes para as bases e +1 para processamento interno no arquivo cuda_stack.cu
    hipMalloc((void**)&s,buffer_size*sizeof(char*));
	tmp = (char*)malloc(blocoV*sizeof(char));
    
			
	#pragma omp parallel num_threads(2) shared(buffer) shared(buffer_flag) shared(p_sensos) shared(p_antisensos)
	{	
		#pragma omp master
		{
			while(buffer.load != -1){//Looping até o final do buffer
				//printf("%d.\n",buffer.load);
				///////////////////////////////////
				buffer_flag = 1;//Sinal fechado////
				///////////////////////////////////	
				load_buffer(&buffer,s,n);
				while(buffer.load > 0){
				}
			}
			///////////////////////////////////
			buffer_flag = 0;//Sinal Aberto////
			///////////////////////////////////	
			
		}
		
		#pragma omp single		
		{
			int num_threads;
			int num_blocks=1;
			const char *error;
			while( buffer_flag == 1){
			}//Aguarda para que o buffer seja enchido pela primeira vez
			
			while( buffer.load != -1){
				//Realiza loop enquanto existirem sequências para encher o buffer		
		
					num_threads = buffer_size>=buffer.load?buffer.load:buffer_size;
					
					dim3 dimBlock(num_threads);
					dim3 dimGrid(num_blocks);
					k_busca<<<dimGrid,dimBlock>>>(bloco1,bloco2,blocos,s,d_a,d_c,d_g,d_t);//Kernel de busca
					error = hipGetErrorString(hipGetLastError());
					if(strcmp(error,"no error") != 0)
						printf("%s\n",error);
					for(i=0;i<num_threads;i++){//Copia sequências senso e antisenso encontradas
						hipMemcpy(tmp,d_buffer[i],sizeof(char),hipMemcpyDeviceToHost);
						
						switch(tmp[0]){
							case 'S':
								hipMemcpy(tmp,d_buffer[i]+1,blocoV*sizeof(char),hipMemcpyDeviceToHost);
								//printf("S: %s\n",tmp);
								empilha(p_sensos,criar_elemento_pilha(tmp));
							break;
							case 'N':
								hipMemcpy(tmp,d_buffer[i]+1,blocoV*sizeof(char),hipMemcpyDeviceToHost);
								//printf("N: %s\n",tmp);
								empilha(p_antisensos,criar_elemento_pilha(get_antisenso(tmp)));
							break;
							default:
							break;
						}
					}
					buffer.load = 0;//Avisando sobre buffer vazio
		
					while(buffer_flag == 1 || buffer.load == 0){
					}//Espera o buffer ser carregado
				
			}
		}
	}
	return;
}


