	//      operacoes.c
	//      
	//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
	//      
	//		Contem funções e métodos comuns ao projeto Sequence Hunter
	//
	//		27/03/2012

	#include <stdio.h>
	#include <glib.h>

	char* get_antisenso(char *s){
		int i;
		char *antisenso;
		int a_size;
		
		a_size = strlen(s);
		antisenso = (char*)malloc(a_size*sizeof(char));
		
		for(i=0;i<a_size;i++){
			switch(s[a_size - i -1]){
				case 'A':
					strcat(antisenso, "T");
				break;
				case 'C':
					strcat(antisenso, "G");
				break;
				case 'T':
					strcat(antisenso, "A");
				break;
				case 'G':
					strcat(antisenso, "C");
				break;
			}	
		}
		return antisenso;
	}

	int escolhe_GPU(){
	  int num_devices, device,max_multiprocessors,max_device;
	  hipDeviceProp_t properties;	

	  hipGetDeviceCount(&num_devices);
	  max_device = 0;
		
	  if (num_devices > 1) {
			max_multiprocessors = 0;
			  
			for (device = 0; device < num_devices; device++) {//Busca a melhor GPU comparando a quantidade de multi processadores           
			  hipGetDeviceProperties(&properties, device);
			  if (max_multiprocessors < properties.multiProcessorCount) {
				max_multiprocessors = properties.multiProcessorCount;
				max_device = device;
			  }
			}
			hipSetDevice(max_device);
		  }
			
		  return max_device;
		}

	void getDevice(int deviceCount,hipDeviceProp_t deviceProp,gboolean verbose){

	  int i;
	  int device;
	  hipError_t erro;

		 printf("Dispositivos encontrados:\n");
		  for(i=0;i<deviceCount;i++)
			{
			  erro = hipGetDeviceProperties(&deviceProp,i);
			  g_assert(erro == hipSuccess);
			  printf("%d) %s\n",i,deviceProp.name);
			}

		  printf("\n Usar dispositivo:");
		  scanf("%d",&device);
			
		  while(device > deviceCount)
			{
			  printf("Dispositivo invalido\n");
			  scanf("%d",&device);
			}
		
		//Libera o dispositivo em uso
		erro = hipDeviceReset();
		g_assert(erro == hipSuccess);
		
		//Configura qual dispositivo deve ser usado
		erro = hipSetDevice(device);
		g_assert(erro == hipSuccess);
			
		//O dispositivo foi escolhido
		erro = hipGetDeviceProperties(&deviceProp,device);
		g_assert(erro == hipSuccess);
		
		if(verbose == TRUE) printf("Dispositivo configurado para uso: %s\n\n",deviceProp.name); 
	 
	}
