#include "hip/hip_runtime.h"
#include <stdio.h>
#include "estruturas.h"
#include "log.h"
#include "linkedlist.h"
#include "pilha.h"

lista_ligada* processar(pilha *p_sensos,pilha *p_antisensos){
	
	lista_ligada *l;
	int s_tipos = 0;
	int retorno;
	l = criar_lista();
	
	//Processa sensos
	while(pilha_vazia(p_sensos) == 1){
		retorno = busca_lista_s(l,desempilha(p_sensos));
		if(retorno == 1)
			s_tipos++;
	}
	
	
	while( pilha_vazia(p_antisensos)== 1){
		retorno = busca_lista_as(l,desempilha(p_antisensos));
		if(retorno == 1){
			printf("Não encontrei!\n");
		}
	}
	
	printf("Tipos de senso encontrados: %d.\n",s_tipos);
	printf("Procurando sensos sem antisensos.\n");
	printf("Sensos solitários: %d.\n",limpando_sensos(l));
	
	imprimir_sensos(l);
	
	return l;
}

float qnt_relativa(){
	//Retorna a porcentagem de exemplares de determinada sequência dentro do total
}

void gerar_relatorio(){
		//		Deve criar arquivo data-hora.txt com a sequência buscada, a quantidade de sequências lidas na biblioteca, tamanho do buffer,
		//	tempo de processamento, tipos e quantidade de sensos e antisensos encontrados (inclusive sua quantidade relativa) e quantidade de
		//	sensos e antisensos encontrados mas que foram descartados por não possuirem pares.
	
	
}
