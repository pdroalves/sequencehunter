#include "hip/hip_runtime.h"
#include <stdio.h>
#include "estruturas.h"
#include "log.h"
#include "linkedlist.h"
#include "pilha.h"

lista_ligada* processar(pilha *p_sensos,pilha *p_antisensos){
	
	lista_ligada *l;
	int s_tipos = 0;
	int retorno;
	l = criar_lista();
	
	//Processa sensos
	while(pilha_vazia(p_sensos) == 1){
		retorno = busca_lista_s(l,desempilha(p_sensos));
		if(retorno == 1)
			s_tipos++;
	}
	
	
	while( pilha_vazia(p_antisensos)== 1){
		retorno = busca_lista_as(l,desempilha(p_antisensos));
		if(retorno == 1){
			printf("Não encontrei!\n");
		}
	}
	
	printf("Tipos de senso encontrados: %d.\n",s_tipos);
	printf("Procurando sensos sem antisensos.\n");
	printf("Sensos solitários: %d.\n",limpando_sensos(l));
	
	imprimir_sensos(l);
	
	return l;
}
