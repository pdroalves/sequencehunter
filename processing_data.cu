#include "hip/hip_runtime.h"
#include <stdio.h>
#include "estruturas.h"
#include "log.h"
#include "linkedlist.h"
#include "pilha.h"
#define TAM_MAX 10000


int check_seq_valida(char *p);

int get_sequencias_validas(FILE **f,int files){
	int j = 0;
	int notify_tamanho_variavel = 0;
	int seqs_validas = 0;
	int m = 0;
	int n = 0;
	char *tmp;
	
	tmp = (char*)malloc(TAM_MAX*sizeof(char));
	for(j=0;j < files;j++){		
		while(feof(f[j]) == 0){
				fscanf(f[j],"%s",tmp);		
				n = strlen(tmp);
				if(check_seq_valida(tmp)){	
					seqs_validas++;
					//printf("%d\n",seqs_validas);
					if(m == 0) m = n;
					else if(n!=m && notify_tamanho_variavel == 0) notify_tamanho_variavel == 1;
				}
		}
	}
	for(j=0;j<files;j++) rewind(f[j]);
	return -seqs_validas;
}

	int check_seq_valida(char *p){
	int i;
	int n = strlen(p);
	
	for(i = 0; i < n;i++){
			switch(p[i]){
				case 'A':
				break;
				case 'a':
				break;
				case 'C':
				break;
				case 'c':
				break;
				case 'G':
				break;
				case 'g':
				break;
				case 'T':
				break;
				case 't':
				break;
				default:
					return 0;
			}
	}
	
	return 1;
}

lista_ligada* processar(pilha *p_sensos,pilha *p_antisensos){
	
	lista_ligada *l;
	lista_ligada **resultados;
	int s_tipos = 0;
	int retorno;
	l = criar_lista();
	
	//Processa sensos
	while(pilha_vazia(p_sensos) == 1){
		retorno = busca_lista_s(l,desempilha(p_sensos));
		if(retorno == 1)
			s_tipos++;
	}
	
	
	while( pilha_vazia(p_antisensos)== 1){
		retorno = busca_lista_as(l,desempilha(p_antisensos));
		if(retorno == 1){
			printf("Não encontrei!\n");
		}
	}
	
	printf("Tipos de senso encontrados: %d.\n",s_tipos);
	printf("Procurando sensos sem antisensos.\n");
	printf("Sensos solitários: %d.\n",limpando_sensos(l));
	
	printf("Processando.\n");
	qnt_relativa(l);
	resultados = ordena_pares(l);
	printf("Frequencias estimadas.\n");
	imprimir_sensos(resultados);
	
	return l;
}

void swap(lista_ligada** a, lista_ligada** b) {
  lista_ligada *tmp;
  tmp = *a;
  *a = *b;
  *b = tmp;
return;
}
 
int particione(lista_ligada **l, int left, int right) {
  int i;
  int j;
 
  i = left;
  for (j = left + 1; j <= right; ++j) {
    if (&l[j]->qnt_relativa < &l[left]->qnt_relativa){
      i++;
      swap(&l[i], &l[j]);
    }
  }
  swap(&l[left], &l[i]);
 
  return i;
}
 
void quicksort(lista_ligada **l, int left, int right) {
  int r;
 
  if (right > left) {
    r = particione(l, left, right);
    quicksort(l, left, r - 1);
    quicksort(l, r + 1, right);
  }
  
}

void gerar_relatorio(){
		//		Deve criar arquivo data-hora.txt com a sequência buscada, a quantidade de sequências lidas na biblioteca, tamanho do buffer,
		//	tempo de processamento, tipos e quantidade de sensos e antisensos encontrados (inclusive sua quantidade relativa) e quantidade de
		//	sensos e antisensos encontrados mas que foram descartados por não possuirem pares.
	
	
}
