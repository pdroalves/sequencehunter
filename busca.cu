#include "hip/hip_runtime.h"
//      busca.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Implementação do algoritmo de busca por grafos.
//
//		27/03/2012

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "estruturas.h"


__global__ void set_grafo(char*,char *,vgrafo*,vgrafo*,vgrafo*, vgrafo*);

__host__ __device__ void caminhar(vgrafo*, vgrafo*, int*,int*);
__device__ void build_grafo(vgrafo*,vgrafo*,vgrafo*, vgrafo*);
__host__ __device__ vgrafo* busca_vertice(char,vgrafo *,vgrafo *,vgrafo *, vgrafo *);

__global__ void k_busca(int *matchs,char **data,int size,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
	
	int posicao = blockIdx.x*blockDim.x + threadIdx.x;
	char *seq;
	int i=0;
	int found = 0;//1 se encontrar uma seq
	int s_match = 0;
	int as_match = 0;
	vgrafo *atual;
	vgrafo *prox;
	
	seq =  data[posicao];
	#pragma unroll 1
	while(seq[i+1] != '\0' && s_match < (size-1) && as_match < (size-1)){
		atual = busca_vertice(seq[i],a,c,g,t);
		prox = busca_vertice(seq[i+1],a,c,g,t);
		caminhar(atual,prox,&s_match,&as_match);
		i++;
	}
	
	//Marca o que foi encontrado
	//1 se for senso
	//2 se for antisenso
	//0 se não for nada
	matchs[posicao] = s_match / (size-1);
	matchs[posicao] = 2*(as_match / (size-1));
	
	return;
}

__host__ __device__ vgrafo* busca_vertice(char base,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
	//Funcao temporária. Ficará aqui até eu pensar em algo melhor
	switch(base){
		case 'A':
			return a;
		case 'C':
			return c;
		case 'G':
			return g;
		case 'T':
			return t;
	}
	
	return NULL;
}

__host__ __device__ void caminhar(vgrafo *atual, vgrafo *prox, int *s_match,int *as_match){ 
	//Recebe o vertice atual
	//Recebe o próximo vertice
	//Recebe um contador de bases acertadas para a sequencia senso s_match
	//Recebe um contador de bases acertadas para a sequencia antisenso as_match
	
	//OTIMIZAR! Tem muitos IFs
	
	
	if(prox->psenso == atual->psenso+1)//A sequencia confere com o senso
		s_match++;
	else//A sequencia nao confere com o senso
		s_match=0;
		
	if(prox->pasenso == atual->pasenso+1)//A sequencia confere com o antisenso
		as_match++;
	else//A sequencia nao confere com o antisenso
		as_match=0;
		
	return;	
}

__host__ __device__ void build_grafo(vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
		
	//Define cada vértice
	a->vertice = 'A';
	c->vertice = 'C';
	g->vertice = 'G';
	t->vertice = 'T';
	
	//Inicializa as marcações
	a->psenso = 0;
	a->pasenso = 0;
	c->psenso = 0;
	c->pasenso = 0;
	g->psenso = 0;
	g->pasenso = 0;
	t->psenso = 0;
	t->pasenso = 0;
	
	//Conecta os vértices
	a->a = a;
	a->c = c;
	a->g = g;
	a->t = t;
	
	c->a = a;
	c->c = c;
	c->g = g;
	c->t = t;
	
	g->a = a;
	g->c = c;
	g->g = g;
	g->t = t;
	
	t->a = a;
	t->c = c;
	t->g = g;
	t->t = t;
	
	return;
}

__global__ void set_grafo(char *senso,char *antisenso,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
	
	//Configura grafo
	int i;
	vgrafo *atual;
	
	build_grafo(a,c,g,t);
	
	i=0;
	//Configura sequência senso
	while(senso[i] != '\0'){
		i++;
		atual = busca_vertice(senso[i],a,c,g,t);
		atual->psenso=i;
	}
	
	i=0;
	//Configura sequência antisenso
	while(antisenso[i] != '\0'){
		i++;
		atual = busca_vertice(antisenso[i],a,c,g,t);
		atual->pasenso=i;
	}
		
	
	return;
}
