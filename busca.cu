#include "hip/hip_runtime.h"
//      busca.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Implementação do algoritmo de busca por grafos.
//
//		27/03/2012

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "estruturas.h"


#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)//Toma cuidado de não usar printf sem que a máquina suporte.
#define printf(f, ...) ((void)(f, __VA_ARGS__),0)
#endif

extern "C" void set_grafo_helper(char *senso,char *antisenso,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t);
extern "C" void k_busca_helper(int num_blocks,int num_threads,const int bloco1,const int bloco2,const int blocos,char **s,vgrafo *d_a,vgrafo *d_c,vgrafo *d_g,vgrafo *d_t);
extern "C" __global__ void set_grafo(char*,char *,vgrafo*,vgrafo*,vgrafo*, vgrafo*);
extern "C" void set_grafo_NONCuda(char*,char *,vgrafo*,vgrafo*,vgrafo*, vgrafo*);
extern "C" __host__ __device__ void caminhar(vgrafo*,vgrafo*,vgrafo*, int*,int*);
__device__ void build_grafo(vgrafo*,vgrafo*,vgrafo*, vgrafo*);
extern "C" __host__ __device__ vgrafo* busca_vertice(char,vgrafo *,vgrafo *,vgrafo *, vgrafo *);

////////////////////////////////////////////////////////////////////////////////////////
//////////////////					Kernel Principal 				////////////////////
////////////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void k_busca(const int bloco1,const int bloco2,const int blocos,char **data,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
 
  ////////
  ////////
  ////////
  ////////		Recebe o tamanho dos blocos 1 e 2
  ////////		Recebe o tamanho total da sequência
  ////////		Recebe o endereço com todo o buffer
  ////////		Recebe ponteiros para cada vertice do grafo
  ////////
  ////////
  ////////
  ////////
  											
  const int posicao = blockIdx.x*blockDim.x + threadIdx.x;
  char *seq = data[posicao];//Seto ponteiro para a sequência que será analisada
  //printf("%d: Peguei: %s\n",posicao,seq);
  int i;
  int s_match;
  int as_match;
  vgrafo *atual;
  vgrafo *anterior;
  vgrafo *ant_anterior;
  int x0=1;/////Essas variáveis guardam o intervalo onde podemos encontrar os elementos que queremos
  int xn;/////
  int size = bloco1 + bloco2;
  int blocoZ = blocos - size;//Total de bases que queremos encontrar
  char tipo = '\0';						
  s_match = as_match = 0;
  i=0;
  
  ////////////////////
  ////////////////////										
  //Iteração inicial//																			
  ////////////////////
  ////////////////////
  ant_anterior = busca_vertice(seq[i],a,c,g,t);
  caminhar(NULL,NULL,ant_anterior,&s_match,&as_match);
  i++;
  anterior = busca_vertice(seq[i],a,c,g,t);
  caminhar(NULL,ant_anterior,anterior,&s_match,&as_match);
  i++;
  
  																			
  ///////////////////////
  ///////////////////////					
  //Iterações seguintes//																			
  ///////////////////////
  ///////////////////////
  										
#pragma unroll 1
  while( seq[i] != '\0' && s_match < size && as_match < size) {
	  //printf("s_match: %d\n",s_match);
	  //printf("as_match: %d\n",as_match);
	  
	if(s_match == bloco1){
		//printf("Th: %d --> Bloco 1 encontrado na posicao %d, %s-> Sequência senso.\n",posicao,i,seq);
		tipo = 'S';//Senso
		x0 = i;//Marca primeiro elemento 
		xn = x0 + blocoZ;//Marca primeiro elemento do bloco 2
		i = xn;  //Salta o bloco variável
	}
	if(as_match == bloco2){
		//printf("Th: %d --> Bloco 2 encontrado na posicao %d, %s-> Sequência antisenso.\n",posicao,i,seq);
		tipo = 'N';//Não-Senso
		x0 = i;//Marca primeiro elemento 
		xn = x0 + blocoZ;//Marca primeiro elemento do bloco 2
		i = xn;  //Salta o bloco variável
	}
    atual = busca_vertice(seq[i],a,c,g,t);
    if(atual != NULL)
      caminhar(ant_anterior,anterior,atual,&s_match,&as_match);
    i++;
    ant_anterior = anterior;
    anterior = atual;
  }

  ///////////////////////////////											
  //Guarda o que foi encontrado//
  ///////////////////////////////
  
  //printf("s_match: %d - as_match: %d\n",s_match,as_match);
	
	
    if(s_match == size || as_match == size){
		//printf("%s -> s_match= %d e as_match=%d\n",seq,s_match,as_match);
		seq[0] = tipo;
		for(i=1;i<=blocoZ;i++){
		  seq[i] = seq[x0 + i-1];
		}
		seq[i] = '\0';
		return;
	}
	seq[0] = '\0';
	
	
  return;
}

extern "C" void k_busca_helper(int num_blocks,int num_threads,const int bloco1,const int bloco2,const int blocos,char **s,vgrafo *d_a,vgrafo *d_c,vgrafo *d_g,vgrafo *d_t){
	dim3 dimBlock(num_threads);
	dim3 dimGrid(num_blocks);
	k_busca<<<dimGrid,dimBlock>>>(bloco1,bloco2,blocos,s,d_a,d_c,d_g,d_t);//Kernel de busca
}

////////////////////////////////////////////////////////////////////////////////////////
//////////////////					Versão sem CUDA 				////////////////////
////////////////////////////////////////////////////////////////////////////////////////
extern "C" void busca(const int bloco1,const int bloco2,const int blocos,Buffer buffer,const int th_id,const int nthreads,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
 
  ////////
  ////////
  ////////
  ////////		Recebe o tamanho dos blocos 1 e 2
  ////////		Recebe o tamanho total da sequência
  ////////		Recebe o endereço com todo o buffer
  ////////		Recebe ponteiros para cada vertice do grafo
  ////////
  ////////
  ////////
  ////////
  int posicao;
  int tam = buffer.load;
  int razao = tam / nthreads;
  for(posicao=th_id*razao;posicao < th_id + razao;posicao++){
	  char *seq = buffer.seq[posicao];//Seto ponteiro para a sequência que será analisada
	  //printf("%d: Peguei: %s\n",posicao,seq);
	  int i;
	  int s_match;
	  int as_match;
	  vgrafo *atual;
	  vgrafo *anterior;
	  vgrafo *ant_anterior;
	  int x0=1;/////Essas variáveis guardam o intervalo onde podemos encontrar os elementos que queremos
	  int xn;/////
	  int size = bloco1 + bloco2;
	  int blocoZ = blocos - size;//Total de bases que queremos encontrar
	  char tipo = '\0';						
	  s_match = as_match = 0;
	  i=0;
	  
	  ////////////////////
	  ////////////////////										
	  //Iteração inicial//																			
	  ////////////////////
	  ////////////////////
	  ant_anterior = busca_vertice(seq[i],a,c,g,t);
	  caminhar(NULL,NULL,ant_anterior,&s_match,&as_match);
	  i++;
	  anterior = busca_vertice(seq[i],a,c,g,t);
	  caminhar(NULL,ant_anterior,anterior,&s_match,&as_match);
	  i++;
	  
																				
	  ///////////////////////
	  ///////////////////////					
	  //Iterações seguintes//																			
	  ///////////////////////
	  ///////////////////////
						
	  while( seq[i] != '\0' && s_match < size && as_match < size) {
		  //printf("s_match: %d\n",s_match);
		 //printf("as_match: %d\n",as_match);
		  
		if(s_match == bloco1){
			//printf("Th: %d --> Bloco 1 encontrado na posicao %d, %s-> Sequência senso.\n",posicao,i,seq);
			tipo = 'S';//Senso
			x0 = i;//Marca primeiro elemento 
			xn = x0 + blocoZ;//Marca primeiro elemento do bloco 2
			i = xn;  //Salta o bloco variável
		}
		if(as_match == bloco2){
			//printf("Th: %d --> Bloco 2 encontrado na posicao %d, %s-> Sequência antisenso.\n",posicao,i,seq);
			tipo = 'N';//Não-Senso
			x0 = i;//Marca primeiro elemento 
			xn = x0 + blocoZ;//Marca primeiro elemento do bloco 2
			i = xn;  //Salta o bloco variável
		}
		atual = busca_vertice(seq[i],a,c,g,t);
		if(atual != NULL)
		  caminhar(ant_anterior,anterior,atual,&s_match,&as_match);
		i++;
		ant_anterior = anterior;
		anterior = atual;
	  }

	  ///////////////////////////////											
	  //Guarda o que foi encontrado//
	  ///////////////////////////////
	  
	  //printf("s_match: %d - as_match: %d\n",s_match,as_match);

		if(s_match == size || as_match == size){
			//printf("%s -> s_match= %d e as_match=%d\n",seq,s_match,as_match);
			buffer.seq[posicao][0] = tipo;
			for(i=1;i<=blocoZ;i++){
			  buffer.seq[posicao][i] = seq[x0 + i-1];
			}
			buffer.seq[posicao][i] = '\0';
		}else
			buffer.seq[posicao][0] = '\0';
	}
	
  return;
}
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////

extern "C" __host__ __device__ vgrafo* busca_vertice(char base,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
  //Funcao temporária. Ficará aqui até eu pensar em algo melhor
  switch(base){
  case 'A':
    //	printf("Retornei A\n");
    return a;
  case 'C':
    //	printf("Retornei C\n");
    return c;
  case 'G':
    //	printf("Retornei G\n");
    return g;
  case 'T':
    //	printf("Retornei T\n");
    return t;
  }
											
  return NULL;
}

extern "C" __host__ __device__ void caminhar(vgrafo *ant_anterior,vgrafo* anterior,vgrafo *atual, int *s_match,int *as_match){ 
  //Recebe o vertice atual e o anterior
  //Recebe um contador de bases acertadas para a sequencia senso s_match
  //Recebe um contador de bases acertadas para a sequencia antisenso as_match
											
  //OTIMIZAR! Tem muitos IFs
											
  //printf("Analisando base %c. %d -> %d\n",atual->vertice,*s_match,atual->s_marcas[(*s_match)]);
  if(atual->s_marcas[(*s_match)] == 1)//Elemento e posição batem com o que queremos
    (*s_match)++;
  else{//Não bate
    if(anterior != NULL && ant_anterior != NULL)
      if(anterior->vertice != atual->vertice || anterior->vertice != ant_anterior->vertice)
		(*s_match)=0;
  }
												
  if(atual->as_marcas[(*as_match)] == 1)//Elemento e posição batem com o que queremos
    (*as_match)++;
  else{//Não bate
    if(anterior != NULL && ant_anterior != NULL)
      if(anterior->vertice != atual->vertice || anterior->vertice != ant_anterior->vertice)
		(*as_match)=0;
  }
  //printf("s_match: %d\n",*s_match);
  return;	
}

__host__ __device__ void build_grafo(int size,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
												
  int i;
											
  //Define cada vértice
  a->vertice = 'A';
  c->vertice = 'C';
  g->vertice = 'G';
  t->vertice = 'T';
											
  //Inicializa as marcações
  a->s_marcas = (int*)malloc(size*sizeof(int));
  c->s_marcas = (int*)malloc(size*sizeof(int));
  g->s_marcas = (int*)malloc(size*sizeof(int));
  t->s_marcas = (int*)malloc(size*sizeof(int));
													
  a->as_marcas = (int*)malloc(size*sizeof(int));
  c->as_marcas = (int*)malloc(size*sizeof(int));
  g->as_marcas = (int*)malloc(size*sizeof(int));
  t->as_marcas = (int*)malloc(size*sizeof(int));
											
  for(i=0;i<size;i++){
    a->s_marcas[i] = 0;
    c->s_marcas[i] = 0;
    g->s_marcas[i] = 0;
    t->s_marcas[i] = 0;
												
    a->as_marcas[i] = 0;
    c->as_marcas[i] = 0;
    g->as_marcas[i] = 0;
    t->as_marcas[i] = 0;
  }
											
  //Conecta os vértices
  a->a = a;
  a->c = c;
  a->g = g;
  a->t = t;
											
  c->a = a;
  c->c = c;
  c->g = g;
  c->t = t;
											
  g->a = a;
  g->c = c;
  g->g = g;
  g->t = t;
											
  t->a = a;
  t->c = c;
  t->g = g;
  t->t = t;
												
  return;
}

extern "C" __global__ void set_grafo(char *senso,char *antisenso,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
											
  //Configura grafo
  int i;
  int j;
  int size;
  vgrafo *atual;
											
  for(size=0;senso[size] != '\0';size++);//Pega tamanho das sequências
  size++;
  build_grafo(size,a,c,g,t);
											
  i=0;
  j=0;
  printf("Configurando senso. -> %s.\n",senso);
  //Configura sequência senso
  while(senso[i] != '\0'){
    atual = busca_vertice(senso[i],a,c,g,t);
    if(atual != NULL){
		atual->s_marcas[i-j]=1;
		printf("%c marcado na posicao %d.\n",atual->vertice,i-j);
	}else{
		//printf("Elemento variável encontrado.\n");
		j++;
	}
    i++;
  }
											
  i=0;
  j=0;
  printf("\nConfigurando antisenso. -> %s.\n",antisenso);
  //Configura sequência antisenso
  while(antisenso[i] != '\0'){
    atual = busca_vertice(antisenso[i],a,c,g,t);
      if(atual != NULL){
		atual->as_marcas[i-j]=1;
		printf("%c marcado na posicao %d.\n",atual->vertice,i-j);
	}else{
		//printf("Elemento variável encontrado.\n");
		j++;
	}
    i++;
  }
  /*
    for(i=0;i<size;i++){
    printf("%c: %d -> %d\n",'A',i,a->s_marcas[i]);
    printf("%c: %d -> %d\n",'C',i,c->s_marcas[i]);
    printf("%c: %d -> %d\n",'G',i,g->s_marcas[i]);
    printf("%c: %d -> %d\n",'T',i,t->s_marcas[i]);
    }*/
  return;
}


extern "C" void set_grafo_helper(char *senso,char *antisenso,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
set_grafo<<<1,1>>>(senso,antisenso,a,c,g,t);
}

extern "C" void set_grafo_NONCuda(char *senso,char *antisenso,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
											
  //Configura grafo
  int i;
  int j;
  int size;
  vgrafo *atual;
											
  size = strlen(senso);//Pega tamanho das sequências
  build_grafo(size,a,c,g,t);
											
  i=0;
  j=0;
  printf("Configurando senso. -> %s.\n",senso);
  //Configura sequência senso
  while(senso[i] != '\0'){
    atual = busca_vertice(senso[i],a,c,g,t);
    if(atual != NULL){
		atual->s_marcas[i-j]=1;
		printf("%c marcado na posicao %d.\n",atual->vertice,i-j);
	}else{
		//printf("Elemento variável encontrado.\n");
		j++;
	}
    i++;
  }
											
  i=0;
  j=0;
  printf("\nConfigurando antisenso. -> %s.\n",antisenso);
  //Configura sequência antisenso
  while(antisenso[i] != '\0'){
    atual = busca_vertice(antisenso[i],a,c,g,t);
      if(atual != NULL){
		atual->as_marcas[i-j]=1;
		printf("%c marcado na posicao %d.\n",atual->vertice,i-j);
	}else{
		//printf("Elemento variável encontrado.\n");
		j++;
	}
    i++;
  }
  /*
    for(i=0;i<size;i++){
    printf("%c: %d -> %d\n",'A',i,a->s_marcas[i]);
    printf("%c: %d -> %d\n",'C',i,c->s_marcas[i]);
    printf("%c: %d -> %d\n",'G',i,g->s_marcas[i]);
    printf("%c: %d -> %d\n",'T',i,t->s_marcas[i]);
    }*/
  return;
}
