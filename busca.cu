#include "hip/hip_runtime.h"
//      busca.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Implementação do algoritmo de busca por grafos.
//
//		27/03/2012

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "estruturas.h"


#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)//Toma cuidado de não usar printf sem que a máquina suporte.
#define printf(f, ...) ((void)(f, __VA_ARGS__),0)
#endif


__global__ void set_grafo(char*,char *,vgrafo*,vgrafo*,vgrafo*, vgrafo*);
__host__ __device__ void caminhar(vgrafo*,vgrafo*, int*,int*);
__device__ void build_grafo(vgrafo*,vgrafo*,vgrafo*, vgrafo*);
__host__ __device__ vgrafo* busca_vertice(char,vgrafo *,vgrafo *,vgrafo *, vgrafo *);

__global__ void k_busca(int *matchs,char **data,const int size,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
											
  //int posicao = blockIdx.x*blockDim.x + threadIdx.x;
  int posicao = threadIdx.x;
  const char *seq = data[posicao];//Seto ponteiro para a sequência que será analisada
  int i;
  //int found = 0;//1 se encontrar uma seq
  int s_match;
  int as_match;
  vgrafo *atual;
  vgrafo *anterior;
												
  s_match = as_match = 0;
  i=0;
											
												
  atual = busca_vertice(seq[0],a,c,g,t);
  if(atual != NULL)
    caminhar(NULL,atual,&s_match,&as_match);
  i++;
  anterior = atual;
											
#pragma unroll 1
  while( seq[i] != '\0' && s_match < (size) && as_match < (size)){
    atual = busca_vertice(seq[i],a,c,g,t);
    if(atual != NULL)
      caminhar(anterior,atual,&s_match,&as_match);
    i++;
    anterior = atual;
  }
											
  //Marca o que foi encontrado
  //1 se for senso
  //2 se for antisenso
  //0 se não for nada
  matchs[posicao] = s_match / (size);
  //matchs[posicao] = 2*(as_match / (size-1));
											
  return;
}

__host__ __device__ vgrafo* busca_vertice(char base,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
  //Funcao temporária. Ficará aqui até eu pensar em algo melhor
  switch(base){
  case 'A':
    //	printf("Retornei A\n");
    return a;
  case 'C':
    //	printf("Retornei C\n");
    return c;
  case 'G':
    //	printf("Retornei G\n");
    return g;
  case 'T':
    //	printf("Retornei T\n");
    return t;
  }
											
  return NULL;
}

__host__ __device__ void caminhar(vgrafo* anterior,vgrafo *atual, int *s_match,int *as_match){ 
  //Recebe o vertice atual e o anterior
  //Recebe um contador de bases acertadas para a sequencia senso s_match
  //Recebe um contador de bases acertadas para a sequencia antisenso as_match
											
  //OTIMIZAR! Tem muitos IFs
											
  //printf("Analisando base %c. %d -> %d\n",atual->vertice,*s_match,atual->s_marcas[(*s_match)]);
  if(atual->s_marcas[(*s_match)] == 1){//Elemento e posição batem com o que queremos
    (*s_match)++;
    //printf("Match: %c\n",atual->vertice);
  }
  else{//Não bate
    if(anterior != NULL)
      if(anterior->vertice != atual->vertice)
	(*s_match)=0;
  }
												
  if(atual->as_marcas[(*as_match)] == 1)//Elemento e posição batem com o que queremos
    (*as_match)++;
  else{//Não bate
    if(anterior != NULL)
      if(anterior->vertice != atual->vertice)
	(*as_match)=0;
  }
  //printf("s_match: %d\n",*s_match);
  return;	
}

__host__ __device__ void build_grafo(int size,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
												
  int i;
											
  //Define cada vértice
  a->vertice = 'A';
  c->vertice = 'C';
  g->vertice = 'G';
  t->vertice = 'T';
											
  //Inicializa as marcações
  a->s_marcas = (int*)malloc(size*sizeof(int));
  c->s_marcas = (int*)malloc(size*sizeof(int));
  g->s_marcas = (int*)malloc(size*sizeof(int));
  t->s_marcas = (int*)malloc(size*sizeof(int));
													
  a->as_marcas = (int*)malloc(size*sizeof(int));
  c->as_marcas = (int*)malloc(size*sizeof(int));
  g->as_marcas = (int*)malloc(size*sizeof(int));
  t->as_marcas = (int*)malloc(size*sizeof(int));
											
  for(i=0;i<size;i++){
    a->s_marcas[i] = 0;
    c->s_marcas[i] = 0;
    g->s_marcas[i] = 0;
    t->s_marcas[i] = 0;
												
    a->as_marcas[i] = 0;
    c->as_marcas[i] = 0;
    g->as_marcas[i] = 0;
    t->as_marcas[i] = 0;
  }
											
  //Conecta os vértices
  a->a = a;
  a->c = c;
  a->g = g;
  a->t = t;
											
  c->a = a;
  c->c = c;
  c->g = g;
  c->t = t;
											
  g->a = a;
  g->c = c;
  g->g = g;
  g->t = t;
											
  t->a = a;
  t->c = c;
  t->g = g;
  t->t = t;
												
  return;
}

__global__ void set_grafo(char *senso,char *antisenso,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
											
  //Configura grafo
  int i;
  int size;
  vgrafo *atual;
											
  for(size=0;senso[size] != '\0';size++);
											
  build_grafo(size,a,c,g,t);
											
  i=0;
  //Configura sequência senso
  while(senso[i] != '\0'){
    atual = busca_vertice(senso[i],a,c,g,t);
    atual->s_marcas[i]=1;
    printf("%c marcado na posicao %d.\n",atual->vertice,i);
    i++;
  }
											
  i=0;
  //Configura sequência antisenso
  while(antisenso[i] != '\0'){
    atual = busca_vertice(antisenso[i],a,c,g,t);
    atual->as_marcas[i]=1;
    i++;
  }
  /*
    for(i=0;i<size;i++){
    printf("%c: %d -> %d\n",'A',i,a->s_marcas[i]);
    printf("%c: %d -> %d\n",'C',i,c->s_marcas[i]);
    printf("%c: %d -> %d\n",'G',i,g->s_marcas[i]);
    printf("%c: %d -> %d\n",'T',i,t->s_marcas[i]);
    }*/
  return;
}

void send_buffer(Buffer *b,int n){
											
  return;
}
