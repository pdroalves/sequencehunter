#include "hip/hip_runtime.h"
//      busca.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Implementação do algoritmo de busca por grafos utilizando CUDA.
//
//		27/03/2012


#include <hip/hip_runtime.h>
#includa "estruturas.h"

__host__ __device__ void caminhar(vgrafo*, vgrafo*, int*,int*);

__global__ void k_busca(int *matchs,char **data,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
	
	int posicao = blockIdx.x*blockDim.x + threadIdx.x;
	char *seq = data[posicao];
	int i;
	int found = 0;//1 se encontrar uma seq
	int s_match = 0;
	int as_match = 0;
	vgrafo *atual;
	vgrafo *prox;
	
	#pragma unroll 1
	while(seq[i+1] != \0 && s_match < size-1 && as_match < size-1){
		atual = busca_vertice(seq[i],a,c,g,t);
		prox = busca_vertice(seq[i+1],a,c,g,t);
		caminhar(atual,prox,&s_match,&as_match);
	}
	
	//Marca o que foi encontrado
	//1 se for senso
	//2 se for antisenso
	//0 se não for nada
	matchs[posicao] = s_match / (size-1);
	matchs[posicao] = 2*(as_match / (size-1));
	
	return;
}

__host__ __device__ vgrafo* busca_vertice(char c,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
	//Funcao temporária. Ficará aqui até eu pensar em algo melhor
	
	switch(c){
		case "A":
			return a;
			break;
		case "C":
			return c;
			break;
		case "G":
			return g;
			break;
		case "T":
			return t;
			break;
	}
}

__host__ __device__ void caminhar(vgrafo *atual, vgrafo *prox, int *s_match,int *as_match){ 
	//Recebe o vertice atual
	//Recebe o próximo vertice
	//Recebe um contador de bases acertadas para a sequencia senso s_match
	//Recebe um contador de bases acertadas para a sequencia antisenso as_match
	
	//OTIMIZAR! Tem muitos IFs
	
	
	if(prox->psenso == atual->psenso+1)//A sequencia confere com o senso
		s_match++;
	else//A sequencia nao confere com o senso
		s_match=0;
		
	if(prox->pasenso == atual->pasenso+1)//A sequencia confere com o antisenso
		as_match++;
	else//A sequencia nao confere com o antisenso
		as_match=0;
		
	return;	
}
