#include "hip/hip_runtime.h"
//      busca.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Implementação do algoritmo de busca por grafos.
//
//		27/03/2012

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "estruturas.h"


#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)//Toma cuidado de não usar printf sem que a máquina suporte.
#define printf(f, ...) ((void)(f, __VA_ARGS__),0)
#endif


__global__ void set_grafo(char*,char *,vgrafo*,vgrafo*,vgrafo*, vgrafo*);
__host__ __device__ void caminhar(vgrafo*,vgrafo*, int*,int*);
__device__ void build_grafo(vgrafo*,vgrafo*,vgrafo*, vgrafo*);
__host__ __device__ vgrafo* busca_vertice(char,vgrafo *,vgrafo *,vgrafo *, vgrafo *);


////////////////////////////////////////////////////////////////////////////////////////
//////////////////					Kernel Principal 				////////////////////
////////////////////////////////////////////////////////////////////////////////////////
__global__ void k_busca(const int bloco1,const int bloco2,const int blocos,char **data,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
 
  ////////
  ////////
  ////////
  ////////		Recebe o tamanho dos blocos 1 e 2
  ////////		Recebe o tamanho total da sequência
  ////////		Recebe o endereço com todo o buffer
  ////////		Recebe ponteiros para cada vertice do grafo
  ////////
  ////////
  ////////
  ////////
  											
  const int posicao = blockIdx.x*blockDim.x + threadIdx.x;
  char *seq = data[posicao];//Seto ponteiro para a sequência que será analisada
  int i;
  int s_match;
  int as_match;
  vgrafo *atual;
  vgrafo *anterior;
  int x0;/////Essas variáveis guardam o intervalo onde podemos encontrar os elementos que queremos
  int xn;/////
  int size = bloco1 + bloco2;
  int blocoZ = blocos - size;//Total de bases que queremos encontrar
  char tipo;						
  s_match = as_match = 0;
  i=0;
  
  ////////////////////
  ////////////////////										
  //Iteração inicial//																			
  ////////////////////
  ////////////////////
  atual = busca_vertice(seq[0],a,c,g,t);
  if(atual != NULL)
    caminhar(NULL,atual,&s_match,&as_match);
  i++;
  anterior = atual;

  																			
  ///////////////////////
  ///////////////////////					
  //Iterações seguintes//																			
  ///////////////////////
  ///////////////////////
  										
#pragma unroll 1
  while( seq[i] != '\0' && s_match < size && as_match < size) {
	//  printf("s_match: %d\n",s_match);
	if(s_match == bloco1){
		//printf("Th: %d --> Bloco 1 encontrado na posicao %d, %c-> Sequência senso.\n",posicao,i,seq[i]);
		tipo = 'S';//Senso
		x0 = i;//Marca primeiro elemento 
		xn = x0 + blocoZ;//Marca primeiro elemento do bloco 2
		i = xn;  //Salta o bloco variável
	}
	if(as_match == bloco2){
	//	printf("Bloco 2 encontrado na posicao %d -> Sequência antisenso.\n",i);
		tipo = 'N';//Não-Senso
		x0 = i;//Marca primeiro elemento 
		xn = x0 + blocoZ;//Marca primeiro elemento do bloco 2
		i = xn;  //Salta o bloco variável
	}
    atual = busca_vertice(seq[i],a,c,g,t);
    if(atual != NULL)
      caminhar(anterior,atual,&s_match,&as_match);
    i++;
    anterior = atual;
  }

  ///////////////////////////////											
  //Guarda o que foi encontrado//
  ///////////////////////////////
  
  //printf("s_match: %d - as_match: %d\n",s_match,as_match);

	if(s_match == size){
	  seq[0] = tipo;
	  #pragma unroll 50
	  for(i=1;i<=blocoZ;i++){
		  seq[i] = seq[x0 + i-1];
		}
		seq[i] = '\0';
		//printf("%s\n",seq);
		//printf("%s\n\n",seq);
		return;
	}	
	
	if(as_match == size){
	  seq[0] = tipo;
	  #pragma unroll 50
	  for(i=1;i<=blocoZ;i++){
		  seq[i] = seq[x0 + i-1];
		}
		seq[i] = '\0';
		//printf("%s\n\n",seq);
		return;
	}
	
	seq[0] = '\0';
											
  return;
}
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////

__host__ __device__ vgrafo* busca_vertice(char base,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
  //Funcao temporária. Ficará aqui até eu pensar em algo melhor
  switch(base){
  case 'A':
    //	printf("Retornei A\n");
    return a;
  case 'C':
    //	printf("Retornei C\n");
    return c;
  case 'G':
    //	printf("Retornei G\n");
    return g;
  case 'T':
    //	printf("Retornei T\n");
    return t;
  }
											
  return NULL;
}

__host__ __device__ void caminhar(vgrafo* anterior,vgrafo *atual, int *s_match,int *as_match){ 
  //Recebe o vertice atual e o anterior
  //Recebe um contador de bases acertadas para a sequencia senso s_match
  //Recebe um contador de bases acertadas para a sequencia antisenso as_match
											
  //OTIMIZAR! Tem muitos IFs
											
  //printf("Analisando base %c. %d -> %d\n",atual->vertice,*s_match,atual->s_marcas[(*s_match)]);
  if(atual->s_marcas[(*s_match)] == 1){//Elemento e posição batem com o que queremos
    (*s_match)++;
    //printf("Match: %c\n",atual->vertice);
  }
  else{//Não bate
    if(anterior != NULL)
      if(anterior->vertice != atual->vertice)
	(*s_match)=0;
  }
												
  if(atual->as_marcas[(*as_match)] == 1)//Elemento e posição batem com o que queremos
    (*as_match)++;
  else{//Não bate
    if(anterior != NULL)
      if(anterior->vertice != atual->vertice)
	(*as_match)=0;
  }
  //printf("s_match: %d\n",*s_match);
  return;	
}

__host__ __device__ void build_grafo(int size,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
												
  int i;
											
  //Define cada vértice
  a->vertice = 'A';
  c->vertice = 'C';
  g->vertice = 'G';
  t->vertice = 'T';
											
  //Inicializa as marcações
  a->s_marcas = (int*)malloc(size*sizeof(int));
  c->s_marcas = (int*)malloc(size*sizeof(int));
  g->s_marcas = (int*)malloc(size*sizeof(int));
  t->s_marcas = (int*)malloc(size*sizeof(int));
													
  a->as_marcas = (int*)malloc(size*sizeof(int));
  c->as_marcas = (int*)malloc(size*sizeof(int));
  g->as_marcas = (int*)malloc(size*sizeof(int));
  t->as_marcas = (int*)malloc(size*sizeof(int));
											
  for(i=0;i<size;i++){
    a->s_marcas[i] = 0;
    c->s_marcas[i] = 0;
    g->s_marcas[i] = 0;
    t->s_marcas[i] = 0;
												
    a->as_marcas[i] = 0;
    c->as_marcas[i] = 0;
    g->as_marcas[i] = 0;
    t->as_marcas[i] = 0;
  }
											
  //Conecta os vértices
  a->a = a;
  a->c = c;
  a->g = g;
  a->t = t;
											
  c->a = a;
  c->c = c;
  c->g = g;
  c->t = t;
											
  g->a = a;
  g->c = c;
  g->g = g;
  g->t = t;
											
  t->a = a;
  t->c = c;
  t->g = g;
  t->t = t;
												
  return;
}

__global__ void set_grafo(char *senso,char *antisenso,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
											
  //Configura grafo
  int i;
  int j;
  int size;
  vgrafo *atual;
											
  for(size=0;senso[size] != '\0';size++);
											
  build_grafo(size,a,c,g,t);
											
  i=0;
  j=0;
  printf("Configurando senso.\n");
  //Configura sequência senso
  while(senso[i] != '\0'){
    atual = busca_vertice(senso[i],a,c,g,t);
    if(atual != NULL){
		atual->s_marcas[i-j]=1;
		//printf("%c marcado na posicao %d.\n",atual->vertice,i-j);
	}else{
		//printf("Elemento variável encontrado.\n");
		j++;
	}
    i++;
  }
											
  i=0;
  j=0;
  printf("\nConfigurando antisenso.\n");
  //Configura sequência antisenso
  while(antisenso[i] != '\0'){
    atual = busca_vertice(antisenso[i],a,c,g,t);
      if(atual != NULL){
		atual->as_marcas[i-j]=1;
		//printf("%c marcado na posicao %d.\n",atual->vertice,i-j);
	}else{
		//printf("Elemento variável encontrado.\n");
		j++;
	}
    i++;
  }
  /*
    for(i=0;i<size;i++){
    printf("%c: %d -> %d\n",'A',i,a->s_marcas[i]);
    printf("%c: %d -> %d\n",'C',i,c->s_marcas[i]);
    printf("%c: %d -> %d\n",'G',i,g->s_marcas[i]);
    printf("%c: %d -> %d\n",'T',i,t->s_marcas[i]);
    }*/
  return;
}
