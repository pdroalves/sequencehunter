#include "hip/hip_runtime.h"
//      busca.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Implementação do algoritmo de busca por grafos.
//
//		27/03/2012

#include <stdio.h>
#include <hip/hip_runtime.h>
extern "C" {
#include "estruturas.h"
}

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)//Toma cuidado de não usar printf sem que a máquina suporte.
#define printf(f, ...) ((void)(f, __VA_ARGS__),0)
#endif

extern "C" __host__ __device__ void caminhar(vgrafo*,vgrafo*,vgrafo*, int*,int*);
extern "C" __host__ __device__ vgrafo* busca_vertice(char,vgrafo *,vgrafo *,vgrafo *, vgrafo *);
__global__ void helloCUDA(float f)
{
  printf("Hello thread %d, block %d f=%f\n", threadIdx.x,blockIdx.x, f);
}
////////////////////////////////////////////////////////////////////////////////////////
//////////////////					Buscador		 				////////////////////
////////////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////////////
///////////////				Metodo de busca com CUDA				////////////////////
////////////////////////////////////////////////////////////////////////////////////////

__global__ void k_buscador(int bloco1,int bloco2, int blocos,char **data,int *resultados,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){

	
  ////////
  ////////
  ////////
  ////////		Recebe o tamanho dos blocos 1 e 2
  ////////		Recebe o tamanho total da sequência
  ////////		Recebe o endereço com todo o buffer
  ////////		Recebe ponteiros para cada vertice do grafo
  ////////
  ////////
  ////////
  ////////
  int size = bloco1 + bloco2;
  int blocoZ;//Total de bases que queremos encontrar
  int i;
  int s_match;
  int as_match;
  vgrafo *atual;
  vgrafo *anterior;
  vgrafo *ant_anterior;
  int x0;/////Essas variáveis guardam o intervalo onde podemos encontrar os elementos que queremos
  int x0S;
  int x0A;
  int totalmatchs;
  char *seq;
  int id;
  int tipo;
  
  blocoZ = blocos - size;
  x0 = 1;
  x0S = 1;
  x0A = 1;
  totalmatchs = blocos;
  id = threadIdx.x;
  seq = data[id];
  s_match = as_match = 0;
  tipo = 0;
  i=0;
  ////////////////////
  ////////////////////										
  //Iteração inicial//																			
  ////////////////////
  ////////////////////
  if(0 == bloco1) x0S = i;
  if(0 == bloco2) x0A = i;
  ant_anterior = busca_vertice(seq[i],a,c,g,t);
  if(ant_anterior != NULL){
    caminhar(NULL,NULL,ant_anterior,&s_match,&as_match);
    i++;
  }
		
  if(s_match == bloco1) x0S = i;
  if(as_match == bloco2) x0A = i;
  anterior = busca_vertice(seq[i],a,c,g,t);
  caminhar(NULL,ant_anterior,anterior,&s_match,&as_match);
  i++;
	  																	
  ///////////////////////
  ///////////////////////					
  //Iterações seguintes//																			
  ///////////////////////
  ///////////////////////
						
  while( seq[i] != '\0' && s_match < totalmatchs && as_match < totalmatchs) {
    //printf("%d - s_match: %d\n",i+1,s_match);
    //printf("%d - as_match: %d\n",i+1,as_match);
		  
    if(s_match == bloco1){
      //printf("Th: %d --> Bloco 1 encontrado na posicao %d, %s-> Sequência senso.\n",posicao,i,seq);
      x0S = i;
    }
    if(as_match == bloco2){
      //printf("Th: %d --> Bloco 2 encontrado na posicao %d, %s-> Sequência antisenso.\n",posicao,i,seq);
      x0A = i;
    }
    atual = busca_vertice(seq[i],a,c,g,t);
    if(atual != NULL)
      caminhar(ant_anterior,anterior,atual,&s_match,&as_match);
    i++;
    ant_anterior = anterior;
    anterior = atual;
  }
  ///////////////////////////////											
  //Guarda o que foi encontrado//
  ///////////////////////////////
	  
 // printf("s_match: %d - as_match: %d\n",s_match,as_match);
 // printf("totalmatchs: %d\n",totalmatchs);
  
  if(s_match == totalmatchs){
    x0 = x0S;
    tipo = 1;
  }
  if(as_match == totalmatchs){
    x0 = x0A;
    tipo = 2;
  }
	
  resultados[id] = tipo;

  //printf("%s -> s_match= %d e as_match=%d\n",seq,s_match,as_match);
  if(s_match == totalmatchs || as_match == totalmatchs){
		for(i=0;i<blocoZ;i++){
		  seq[i] = seq[x0 + i];
		}
		seq[i] = '\0';
	}
	
	return;
}
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////////////
///////////////				Metodo de busca sem CUDA				////////////////////
////////////////////////////////////////////////////////////////////////////////////////

extern "C" __host__ void buscador(const int bloco1,const int bloco2,const int blocos,Buffer *buffer,int *resultados,int id,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
 
  ////////
  ////////
  ////////
  ////////		Recebe o tamanho dos blocos 1 e 2
  ////////		Recebe o tamanho total da sequência
  ////////		Recebe o endereço com todo o buffer
  ////////		Recebe ponteiros para cada vertice do grafo
  ////////
  ////////
  ////////
  ////////
  int size = bloco1 + bloco2;
  int blocoZ = blocos - size;//Total de bases que queremos encontrar
  int i;
  int s_match;
  int as_match;
  vgrafo *atual;
  vgrafo *anterior;
  vgrafo *ant_anterior;
  int x0=1;/////Essas variáveis guardam o intervalo onde podemos encontrar os elementos que queremos
  int x0S=1;
  int x0A=1;
  int totalmatchs = blocos;
  s_match = as_match = 0;
  int tipo = 0;
  char *seq;
  seq = buffer->seq[id];
  i=0;
	  
  ////////////////////
  ////////////////////										
  //Iteração inicial//																			
  ////////////////////
  ////////////////////
  if(s_match == bloco1) x0S = i;
  if(as_match == bloco2) x0A = i;
  ant_anterior = busca_vertice(seq[i],a,c,g,t);
  if(ant_anterior != NULL){
    caminhar(NULL,NULL,ant_anterior,&s_match,&as_match);
    i++;
  }
		
  if(s_match == bloco1) x0S = i;
  if(as_match == bloco2) x0A = i;
  anterior = busca_vertice(seq[i],a,c,g,t);
  caminhar(NULL,ant_anterior,anterior,&s_match,&as_match);
  i++;
	  
																				
  ///////////////////////
  ///////////////////////					
  //Iterações seguintes//																			
  ///////////////////////
  ///////////////////////
						
  while( seq[i] != '\0' && s_match < totalmatchs && as_match < totalmatchs) {
    //printf("s_match: %d\n",s_match);
    //printf("as_match: %d\n",as_match);
		  
    if(s_match == bloco1){
      //printf("Th: %d --> Bloco 1 encontrado na posicao %d, %s-> Sequência senso.\n",posicao,i,seq);
      x0S = i;
    }
    if(as_match == bloco2){
      //printf("Th: %d --> Bloco 2 encontrado na posicao %d, %s-> Sequência antisenso.\n",posicao,i,seq);
      x0A = i;
    }
    atual = busca_vertice(seq[i],a,c,g,t);
    if(atual != NULL)
      caminhar(ant_anterior,anterior,atual,&s_match,&as_match);
    i++;
    ant_anterior = anterior;
    anterior = atual;
  }

  ///////////////////////////////											
  //Guarda o que foi encontrado//
  ///////////////////////////////
	  
  //printf("s_match: %d - as_match: %d\n",s_match,as_match);

  if(s_match == totalmatchs){
    x0 = x0S;
    tipo = 1;
  }
  if(as_match == totalmatchs){
    x0 = x0A;
    tipo = 2;
  }
	
  resultados[id] = tipo;

  if(s_match == totalmatchs || as_match == totalmatchs){
    //printf("%s -> s_match= %d e as_match=%d\n",seq,s_match,as_match);
    for(i=0;i<blocoZ;i++){
      seq[i] = seq[x0 + i];
    }
    seq[i] = '\0';
  }
	

	
return;
}
////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////   	Auxiliar     ///////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
extern "C" void checkCudaError(){
	char erro[100];
	strcpy(erro,hipGetErrorString(hipGetLastError()));
    if(strcmp(erro,"no error") != 0){
		printf("%s\n",erro);
		exit(1);
    }   
}

extern "C" void k_busca(int num_blocks,int num_threads,const int bloco1,const int bloco2,const int blocos,char **data,int *resultados,vgrafo *d_a,vgrafo *d_c,vgrafo *d_g,vgrafo *d_t){
	dim3 dimBlock(num_threads);
	dim3 dimGrid(num_blocks);
	
	k_buscador<<<dimGrid,dimBlock>>>(bloco1,bloco2,blocos,data,resultados,d_a,d_c,d_g,d_t);//Kernel de busca
	//k_buscador<<<dimGrid,dimBlock>>>();//Kernel de busca
	
	//helloCUDA<<<dimGrid,dimBlock>>>(5);
	checkCudaError();
	return;
}

extern "C" void busca(const int bloco1,const int bloco2,const int blocos,Buffer *buffer,int *resultados,vgrafo *h_a,vgrafo *h_c,vgrafo *h_g,vgrafo *h_t){
	int i;
	int size;
	
	size = buffer->load;
	
	for(i=0; i < size; i++)
		buscador(bloco1,bloco2,blocos,buffer,resultados,i,h_a,h_c,h_g,h_t);//Metodo de busca
		
	return;
}
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////


extern "C" __host__ __device__ vgrafo* busca_vertice(char base,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
  //Funcao temporária. Ficará aqui até eu pensar em algo melhor
  switch(base){
  case 'A':
    //	printf("Retornei A\n");
    return a;
  case 'C':
    //	printf("Retornei C\n");
    return c;
  case 'G':
    //	printf("Retornei G\n");
    return g;
  case 'T':
    //	printf("Retornei T\n");
    return t;
  }
											
  return NULL;
}

extern "C" __host__ __device__ void caminhar(vgrafo *ant_anterior,vgrafo* anterior,vgrafo *atual, int *s_match,int *as_match){ 
  //Recebe o vertice atual e o anterior
  //Recebe um contador de bases acertadas para a sequencia senso s_match
  //Recebe um contador de bases acertadas para a sequencia antisenso as_match
											
  //OTIMIZAR! Tem muitos IFs
											
  //printf("Analisando base %c. %d -> %d\n",atual->vertice,*s_match,atual->s_marcas[(*s_match)]);
  if(atual->s_marcas[(*s_match)] == 1)//Elemento e posição batem com o que queremos
    (*s_match)++;
  else{//Não bate
    if(anterior != NULL && ant_anterior != NULL)
      if(anterior->vertice != atual->vertice || anterior->vertice != ant_anterior->vertice)
	(*s_match)=0;
  }
												
  if(atual->as_marcas[(*as_match)] == 1)//Elemento e posição batem com o que queremos
    (*as_match)++;
  else{//Não bate
    if(anterior != NULL && ant_anterior != NULL)
      if(anterior->vertice != atual->vertice || anterior->vertice != ant_anterior->vertice)
	(*as_match)=0;
  }
  //printf("s_match: %d\n",*s_match);
  return;	
}

__host__ __device__ void build_grafo(int size,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
												
  int i;
											
  //Define cada vértice
  a->vertice = 'A';
  c->vertice = 'C';
  g->vertice = 'G';
  t->vertice = 'T';
											
  //Inicializa as marcações
  a->s_marcas = (int*)malloc(size*sizeof(int));
  c->s_marcas = (int*)malloc(size*sizeof(int));
  g->s_marcas = (int*)malloc(size*sizeof(int));
  t->s_marcas = (int*)malloc(size*sizeof(int));
													
  a->as_marcas = (int*)malloc(size*sizeof(int));
  c->as_marcas = (int*)malloc(size*sizeof(int));
  g->as_marcas = (int*)malloc(size*sizeof(int));
  t->as_marcas = (int*)malloc(size*sizeof(int));
											
  for(i=0;i<size;i++){
    a->s_marcas[i] = 0;
    c->s_marcas[i] = 0;
    g->s_marcas[i] = 0;
    t->s_marcas[i] = 0;
												
    a->as_marcas[i] = 0;
    c->as_marcas[i] = 0;
    g->as_marcas[i] = 0;
    t->as_marcas[i] = 0;
  }
											
  //Conecta os vértices
  a->a = a;
  a->c = c;
  a->g = g;
  a->t = t;
											
  c->a = a;
  c->c = c;
  c->g = g;
  c->t = t;
											
  g->a = a;
  g->c = c;
  g->g = g;
  g->t = t;
											
  t->a = a;
  t->c = c;
  t->g = g;
  t->t = t;
												
  return;
}

extern "C" __global__ void set_grafo(char *senso,char *antisenso,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
											
  //Configura grafo
  int i;
  int size;
  vgrafo *atual;
											
  for(size=0;senso[size] != '\0';size++);//Pega tamanho das sequências
  size++;
  build_grafo(size,a,c,g,t);
											
  i=0;
  printf("Configurando senso. -> %s.\n",senso);
  //Configura sequência senso
  while(senso[i] != '\0'){
    atual = busca_vertice(senso[i],a,c,g,t);
    if(atual != NULL){
      atual->s_marcas[i]=1;
      printf("%c marcado na posicao %d.\n",atual->vertice,i);
     }else{
      //printf("Elemento variável encontrado.\n");
      a->s_marcas[i]=1;
      c->s_marcas[i]=1;
      g->s_marcas[i]=1;
      t->s_marcas[i]=1;
    }
    i++;
  }
											
  i=0;
  printf("\nConfigurando antisenso. -> %s.\n",antisenso);
  //Configura sequência antisenso
  while(antisenso[i] != '\0'){
    atual = busca_vertice(antisenso[i],a,c,g,t);
    if(atual != NULL){
      atual->as_marcas[i]=1;
      printf("%c marcado na posicao %d.\n",atual->vertice,i);
      }else{
      //printf("Elemento variável encontrado.\n");
      a->as_marcas[i]=1;
      c->as_marcas[i]=1;
      g->as_marcas[i]=1;
      t->as_marcas[i]=1;
    }
    i++;
  }
  /*
    for(i=0;i<size;i++){
    printf("%c: %d -> %d\n",'A',i,a->s_marcas[i]);
    printf("%c: %d -> %d\n",'C',i,c->s_marcas[i]);
    printf("%c: %d -> %d\n",'G',i,g->s_marcas[i]);
    printf("%c: %d -> %d\n",'T',i,t->s_marcas[i]);
    }*/
  return;
}


extern "C" void set_grafo_helper(char *senso,char *antisenso,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
  set_grafo<<<1,1>>>(senso,antisenso,a,c,g,t);
}

extern "C" void set_grafo_NONCuda(char *senso,char *antisenso,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
											
  //Configura grafo
  int i;
  int size;
  vgrafo *atual;
											
  size = strlen(senso);//Pega tamanho das sequências
  build_grafo(size,a,c,g,t);
											
  i=0;
  printf("Configurando senso. -> %s.\n",senso);
  //Configura sequência senso
  while(senso[i] != '\0'){
    atual = busca_vertice(senso[i],a,c,g,t);
    if(atual != NULL){
      atual->s_marcas[i]=1;
      printf("%c marcado na posicao %d.\n",atual->vertice,i);
    }else{
      //printf("Elemento variável encontrado.\n");
      a->s_marcas[i]=1;
      c->s_marcas[i]=1;
      g->s_marcas[i]=1;
      t->s_marcas[i]=1;
    }
    i++;
  }
											
  i=0;
  printf("\nConfigurando antisenso. -> %s.\n",antisenso);
  //Configura sequência antisenso
  while(antisenso[i] != '\0'){
    atual = busca_vertice(antisenso[i],a,c,g,t);
    if(atual != NULL){
      atual->as_marcas[i]=1;
      printf("%c marcado na posicao %d.\n",atual->vertice,i);
    }else{
      //printf("Elemento variável encontrado.\n");
      a->as_marcas[i]=1;
      c->as_marcas[i]=1;
      g->as_marcas[i]=1;
      t->as_marcas[i]=1;
    }
    i++;
  }
  /*
    for(i=0;i<size;i++){
    printf("%c: %d -> %d\n",'A',i,a->s_marcas[i]);
    printf("%c: %d -> %d\n",'C',i,c->s_marcas[i]);
    printf("%c: %d -> %d\n",'G',i,g->s_marcas[i]);
    printf("%c: %d -> %d\n",'T',i,t->s_marcas[i]);
    }*/
  return;
}


extern "C" void destroy_grafo(vgrafo *a,vgrafo *c,vgrafo *g,vgrafo *t){
  free(a->s_marcas);
  free(a->as_marcas);
  //free(a);
  free(c->s_marcas);
  free(c->as_marcas);
  //free(c);
  free(g->s_marcas);
  free(g->as_marcas);
  //free(g);
  free(t->s_marcas);
  free(t->as_marcas);
  //free(t);
  return;
}

extern "C" void destroy_grafo_CUDA(int size,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
  hipFree(a);
  hipFree(c);
  hipFree(g);
  hipFree(t);
  return;
}
