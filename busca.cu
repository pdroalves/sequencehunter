#include "hip/hip_runtime.h"
//      busca.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Implementação do algoritmo de busca por grafos.
//
//		27/03/2012

#include <stdio.h>
#include <hip/hip_runtime.h>
extern "C" {
#include "estruturas.h"
}
#include "cuda_functions.h"


#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)//Toma cuidado de não usar printf sem que a máquina suporte.
#define printf(f, ...) ((void)(f, __VA_ARGS__),0)
#endif

extern "C" __host__ __device__ void caminhar(vgrafo*,vgrafo*,vgrafo*, int*,int*);
extern "C" __host__ __device__ vgrafo* busca_vertice(char,vgrafo *,vgrafo *,vgrafo *, vgrafo *);

////////////////////////////////////////////////////////////////////////////////////////
//////////////////					Buscador		 				////////////////////
////////////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////////////
///////////////				Metodo de busca com CUDA				////////////////////
////////////////////////////////////////////////////////////////////////////////////////

__global__ void k_buscador_analyse(int totalseqs,int n,char **data,int *resultados,int **matrix_senso,int **matrix_antisenso){

  
  ////////
  ////////
  ////////
  ////////		n: o tamanho da sequência de busca
  ////////		data: o endereço com todo o buffer
  ////////
  ////////
  ////////
  
  int i;
  int seqId;// id da sequencia analisada
  int baseId;// id da base analisada por cada thread
  int tipo;
  int linha[N_COL];// Cada thread cuida de uma linha
  int retorno;
  int fase;
  int p;
  __shared__ int retorno_sum;
  
  tipo = 0;
  seqId = blockIdx.x;
  baseId = threadIdx.x;
  retorno_sum = 0;
  fase = 0;

	if(seqId < totalseqs){
  
	   // Pega uma linha da matriz Ma
	   getLine(data[seqId][baseId],&linha,&p);  
  
	  while(fase + p < n && tipo == 0){
			   
			   // Subtrai a linha do thread da linha da matriz de busca senso
			   retorno = vec_diff(&linha,matrix_senso[baseId],p,fase);
			   retorno_sum += retorno;
			   
			   // Sincroniza todos os threads
			   __syncthreads();
			 
			   if(retorno_sum == 0){
				   // Eh senso
				   tipo = SENSO;
			   }else{
				   retorno_sum = 0;
				   
					// Subtrai a linha do thread da linha da matriz de busca antisenso
				   retorno = vec_diff(&linha,matrix_antisenso[baseId],p,fase);
				   retorno_sum += retorno;
				   
				   // Sincroniza todos os threads
				   __syncthreads();
				   
					if(retorno_sum == 0){
						// Eh antisenso
						tipo = ANTISENSO;
					}
			   }
			 
			fase++;   
		}
	}
	resultados[seqId] = tipo;
	   
	return;
}


////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////////////
///////////////				Metodo de busca sem CUDA				////////////////////
////////////////////////////////////////////////////////////////////////////////////////

extern "C" __host__ void buscador(const int bloco1,const int bloco2,const int blocos,Buffer *buffer,int *resultados,int id,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
 
  ////////
  ////////
  ////////
  ////////		Recebe o tamanho dos blocos 1 e 2
  ////////		Recebe o tamanho total da sequência
  ////////		Recebe o endereço com todo o buffer
  ////////		Recebe ponteiros para cada vertice do grafo
  ////////
  ////////
  ////////
  ////////
  int size = bloco1 + bloco2;
  int blocoZ = blocos - size;//Total de bases que queremos encontrar
  int i;
  int s_match;
  int as_match;
  vgrafo *atual;
  vgrafo *anterior;
  vgrafo *ant_anterior;
  int x0=1;/////Essas variáveis guardam o intervalo onde podemos encontrar os elementos que queremos
  int x0S=1;
  int x0A=1;
  int totalmatchs = blocos;
  s_match = as_match = 0;
  int tipo = 0;
  char *seq;
  seq = buffer->seq[id];
  i=0;
	  
  ////////////////////
  ////////////////////										
  //Iteração inicial//																			
  ////////////////////
  ////////////////////
  if(s_match == bloco1) x0S = i;
  if(as_match == bloco2) x0A = i;
  ant_anterior = busca_vertice(seq[i],a,c,g,t);
  if(ant_anterior != NULL){
    caminhar(NULL,NULL,ant_anterior,&s_match,&as_match);
    i++;
  }
		
  if(s_match == bloco1) x0S = i;
  if(as_match == bloco2) x0A = i;
  anterior = busca_vertice(seq[i],a,c,g,t);
  caminhar(NULL,ant_anterior,anterior,&s_match,&as_match);
  i++;
	  
																				
  ///////////////////////
  ///////////////////////					
  //Iterações seguintes//																			
  ///////////////////////
  ///////////////////////
						
  while( seq[i] != '\0' && s_match < totalmatchs && as_match < totalmatchs) {
    //printf("s_match: %d\n",s_match);
    //printf("as_match: %d\n",as_match);
		  
    if(s_match == bloco1){
      //printf("Th: %d --> Bloco 1 encontrado na posicao %d, %s-> Sequência senso.\n",posicao,i,seq);
      x0S = i;
    }
    if(as_match == bloco2){
      //printf("Th: %d --> Bloco 2 encontrado na posicao %d, %s-> Sequência antisenso.\n",posicao,i,seq);
      x0A = i;
    }
    atual = busca_vertice(seq[i],a,c,g,t);
    if(atual != NULL)
      caminhar(ant_anterior,anterior,atual,&s_match,&as_match);
    i++;
    ant_anterior = anterior;
    anterior = atual;
  }

  ///////////////////////////////											
  //Guarda o que foi encontrado//
  ///////////////////////////////
	  
  //printf("s_match: %d - as_match: %d\n",s_match,as_match);

  if(s_match == totalmatchs){
    x0 = x0S;
    tipo = 1;
  }
  if(as_match == totalmatchs){
    x0 = x0A;
    tipo = 2;
  }
	
  resultados[id] = tipo;

  if(s_match == totalmatchs || as_match == totalmatchs){
    //printf("%s -> s_match= %d e as_match=%d\n",seq,s_match,as_match);
    for(i=0;i<blocoZ;i++){
      seq[i] = seq[x0 + i];
    }
    seq[i] = '\0';
  }
	

	
return;
}
////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////   	Auxiliar     ///////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////

extern "C" void checkCudaError(){
	char erro[100];
	strcpy(erro,hipGetErrorString(hipGetLastError()));
    if(strcmp(erro,"no error") != 0){
		printf("%s\n",erro);
		exit(1);
    }   
}



extern "C" void cudaCopyCharArrays(char **src,char **dst,int n){
	int i;
	for(i=0;i<n;i++){
		hipMemcpy(dst[i],src[i],n*sizeof(char),hipMemcpyHostToDevice);
		checkCudaError();
	}
	return;
}

extern "C" char** cudaGetArrayOfArraysChar(int narrays,int arrays_size){
	char **array;
	int i;
	hipMalloc((void**)&array,narrays*sizeof(char*));
	checkCudaError();
	for(i=0;i<narrays;i++){ 
		hipMalloc((void**)&(array[i]),arrays_size*sizeof(char));
		checkCudaError();
	}
		
	return array;
}

extern "C" void k_busca(int num_threads,int num_blocks,const int loaded,const int bloco1,const int bloco2,const int blocos,char **data,int *resultados,char **founded,vgrafo *d_a,vgrafo *d_c,vgrafo *d_g,vgrafo *d_t,hipStream_t stream){
	dim3 dimBlock(num_threads);
	dim3 dimGrid(num_blocks);
	
	k_buscador<<<dimGrid,dimBlock,0,stream>>>(loaded,bloco1,bloco2,blocos,data,resultados,founded,d_a,d_c,d_g,d_t);//Kernel de busca
	checkCudaError();
	return;
}

extern "C" void busca(const int bloco1,const int bloco2,const int blocos,Buffer *buffer,int *resultados,vgrafo *h_a,vgrafo *h_c,vgrafo *h_g,vgrafo *h_t){
	int i;
	int size;
	
	size = buffer->load;
	
	for(i=0; i < size; i++)
		buscador(bloco1,bloco2,blocos,buffer,resultados,i,h_a,h_c,h_g,h_t);//Metodo de busca
		
	return;
}
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////


extern "C" __host__ __device__ vgrafo* busca_vertice(char base,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
  //Funcao temporária. Ficará aqui até eu pensar em algo melhor
  switch(base){
  case 'A':
    //	printf("Retornei A\n");
    return a;
  case 'C':
    //	printf("Retornei C\n");
    return c;
  case 'G':
    //	printf("Retornei G\n");
    return g;
  case 'T':
    //	printf("Retornei T\n");
    return t;
  }
											
  return NULL;
}

extern "C" __host__ __device__ void caminhar(vgrafo *ant_anterior,vgrafo* anterior,vgrafo *atual, int *s_match,int *as_match){ 
  //Recebe o vertice atual e o anterior
  //Recebe um contador de bases acertadas para a sequencia senso s_match
  //Recebe um contador de bases acertadas para a sequencia antisenso as_match
											
  //OTIMIZAR! Tem muitos IFs
											
  //printf("Analisando base %c. %d -> %d\n",atual->vertice,*s_match,atual->s_marcas[(*s_match)]);
  if(atual->s_marcas[(*s_match)] == 1)//Elemento e posição batem com o que queremos
    (*s_match)++;
  else{//Não bate
    if(anterior != NULL && ant_anterior != NULL)
      if(anterior->vertice != atual->vertice || anterior->vertice != ant_anterior->vertice)
	(*s_match)=0;
  }
												
  if(atual->as_marcas[(*as_match)] == 1)//Elemento e posição batem com o que queremos
    (*as_match)++;
  else{//Não bate
    if(anterior != NULL && ant_anterior != NULL)
      if(anterior->vertice != atual->vertice || anterior->vertice != ant_anterior->vertice)
	(*as_match)=0;
  }
  //printf("s_match: %d\n",*s_match);
  return;	
}

void build_grafo(int size,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
												
  int i;
											
  //Define cada vértice
  a->vertice = 'A';
  c->vertice = 'C';
  g->vertice = 'G';
  t->vertice = 'T';
											
  //Inicializa as marcações
  a->s_marcas = (int*)malloc(size*sizeof(int));
  c->s_marcas = (int*)malloc(size*sizeof(int));
  g->s_marcas = (int*)malloc(size*sizeof(int));
  t->s_marcas = (int*)malloc(size*sizeof(int));
													
  a->as_marcas = (int*)malloc(size*sizeof(int));
  c->as_marcas = (int*)malloc(size*sizeof(int));
  g->as_marcas = (int*)malloc(size*sizeof(int));
  t->as_marcas = (int*)malloc(size*sizeof(int));
											
  for(i=0;i<size;i++){
    a->s_marcas[i] = 0;
    c->s_marcas[i] = 0;
    g->s_marcas[i] = 0;
    t->s_marcas[i] = 0;
												
    a->as_marcas[i] = 0;
    c->as_marcas[i] = 0;
    g->as_marcas[i] = 0;
    t->as_marcas[i] = 0;
  }
											
  //Conecta os vértices
  a->a = a;
  a->c = c;
  a->g = g;
  a->t = t;
											
  c->a = a;
  c->c = c;
  c->g = g;
  c->t = t;
											
  g->a = a;
  g->c = c;
  g->g = g;
  g->t = t;
											
  t->a = a;
  t->c = c;
  t->g = g;
  t->t = t;
												
  return;
}

__device__ __host__ void getLine(char *c,int *linha,int *n){
	// Recebe um vetor de bases e retorna uma linha de binarios
	int i=0;
	while(c[i] != '\0'){
		switch(c[i]){
				case 'A':
					linha[i] = 1;	
				break;
				case 'C':
					linha[i] = 1;		
				break;
				case 'G':
					linha[i] = 1;	
				break;
				case 'T':
					linha[i] = 1;
				break;
				default:
					linha[i] = 1;
				break;
		}
		i++;
	}
	*n = i;
	return;
}

__device__ __host__ char* getBase(int *linha,int n){
	// Recebe uma linha de binarios e retorna uma base
		switch(n){
				case A:
					return 'A';	
				break;
				case C:
					return 'C';		
				break;
				case G:
					return 'G';	
				break;
				case T:
					return 'T';
				break;
				default:
					return 'N';
				break;
		}
	return;
}

__device__ __host__ int vec_diff(int *analise,int *busca,int n,int fase){
	// Subtrai os  elementos do vetor analise de busca. analise deve  ter n elementos.
	int i = 0;
	int j = fase;
	int results = 0;
	
	if(analise[N] == 1) return 0;
	
	while(i < n){
		results+=analise[i]-busca[j];
		i++;
		j++;
	}
	
	return results;
}

__device__ __host__ void getMatrix(int **matrix,char *str,int n){
	// Matrix já deve vir alocada
	int size_x;
	int size_y;
	int i;

	size_x = N_COL;
	size_y = n;

	// Preenche matriz
	for(i = 0; i < size_y;i++){
		getLine(str[i],matrix[i]);
	}	

	return matrix;
}



extern "C" void set_grafo_helper(char *senso,char *antisenso,int **d_matrix_senso,int **d_matrix_antisenso){
  set_grafo_CUDA<<<1,1>>>(senso,antisenso,d_matrix_senso,d_matrix_antisenso);
}

extern "C" __global__ void set_grafo_CUDA(char *senso,char *antisenso,int **matrix_senso,int **matrix_antisenso){
  // As matrizes já devem vir alocadas
  int i;
  int size;
											
  for(size=0;senso[size] != '\0';size++);//Pega tamanho das sequências
											
  i=0;
  printf("Configurando senso. -> %s.\n",senso);
  //Configura sequência senso
  getMatrix(matrix_senso,senso,size);
											
  i=0;
  printf("\nConfigurando antisenso. -> %s.\n",antisenso);
  //Configura sequência antisenso
  getMatrix(matrix_antisenso,antisenso,size);
  
  return;
}

extern "C" void set_grafo_NONCuda(char *senso,char *antisenso,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
											
  //Configura grafo
  int i;
  int size;
  vgrafo *atual;
											
  size = strlen(senso);//Pega tamanho das sequências
  build_grafo(size,a,c,g,t);
											
  i=0;
  printf("Configurando senso. -> %s.\n",senso);
  //Configura sequência senso
  while(senso[i] != '\0'){
    atual = busca_vertice(senso[i],a,c,g,t);
    if(atual != NULL){
      atual->s_marcas[i]=1;
      printf("%c marcado na posicao %d.\n",atual->vertice,i);
    }else{
      //printf("Elemento variável encontrado.\n");
      a->s_marcas[i]=1;
      c->s_marcas[i]=1;
      g->s_marcas[i]=1;
      t->s_marcas[i]=1;
    }
    i++;
  }
											
  i=0;
  printf("\nConfigurando antisenso. -> %s.\n",antisenso);
  //Configura sequência antisenso
  while(antisenso[i] != '\0'){
    atual = busca_vertice(antisenso[i],a,c,g,t);
    if(atual != NULL){
      atual->as_marcas[i]=1;
      printf("%c marcado na posicao %d.\n",atual->vertice,i);
    }else{
      //printf("Elemento variável encontrado.\n");
      a->as_marcas[i]=1;
      c->as_marcas[i]=1;
      g->as_marcas[i]=1;
      t->as_marcas[i]=1;
    }
    i++;
  }
  /*
    for(i=0;i<size;i++){
    printf("%c: %d -> %d\n",'A',i,a->s_marcas[i]);
    printf("%c: %d -> %d\n",'C',i,c->s_marcas[i]);
    printf("%c: %d -> %d\n",'G',i,g->s_marcas[i]);
    printf("%c: %d -> %d\n",'T',i,t->s_marcas[i]);
    }*/
  return;
}


extern "C" void destroy_grafo(vgrafo *a,vgrafo *c,vgrafo *g,vgrafo *t){
  free(a->s_marcas);
  free(a->as_marcas);
  //free(a);
  free(c->s_marcas);
  free(c->as_marcas);
  //free(c);
  free(g->s_marcas);
  free(g->as_marcas);
  //free(g);
  free(t->s_marcas);
  free(t->as_marcas);
  //free(t);
  return;
}

extern "C" void destroy_grafo_CUDA(int size,vgrafo *a,vgrafo *c,vgrafo *g, vgrafo *t){
  hipFree(a);
  hipFree(c);
  hipFree(g);
  hipFree(t);
  return;
}
