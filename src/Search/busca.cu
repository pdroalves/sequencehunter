#include "hip/hip_runtime.h"
//      busca.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Implementação do algoritmo de busca.
//
//		27/03/2012

#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "../Headers/estruturas.h"
#include "../Headers/cuda_functions.h"
#include "../Headers/log.h"

#define ABSOLUTO(a) a>=0?a:-a

__constant__ short int d_matrix_senso[MAX_SEQ_SIZE];
__constant__ short int d_matrix_antisenso[MAX_SEQ_SIZE];


extern "C" void checkCudaError();

////////////////////////////////////////////////////////////////////////////////////////
//////////////////					Buscador		 				////////////////////
////////////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////////////
///////////////				Metodo de busca com CUDA				////////////////////
////////////////////////////////////////////////////////////////////////////////////////

__global__ void k_buscador(int totalseqs,
										int seqSize_an,
										char **data,
										short int *resultados,
										short int *search_gaps,
										int bloco1,
										int bloco2,
										int blocoV){

  ////////		UM THREAD POR SEQUENCIA
  ////////
  ////////
  ////////
  ////////		seqSize_an: o tamanho da sequencia analisada
  ////////		seqSize_busca: o tamanho da sequencia alvo
  ////////		data: o endereco com todo o buffer carregado
  ////////
  ////////
  ////////
  
  unsigned int seqId;// id da sequencia analisada
  int baseId;// id da base analisada
  short int tipo;// Variavel temporaria para salvar o resultado de uma analise
  short int linha;// Cada thread cuida de uma linha
  short int lsenso;// Guarda o valor da matriz senso para ser comparado com a sequencia alvo
  short int lantisenso;// Guarda o valor da matriz antisenso para ser comparado com a sequencia alvo
  short int alarmS;// Caso a comparacao do valor da matriz senso falhe, essa variavel encerra o loop
  short int alarmAS;// Caso a comparacao do valor da matriz antisenso falhe, essa variavel encerra o loop
  short int fase;// Guarda a posicao analisada
  short int seqSize_bu;// Tamanho da sequencia alvo
  char *seq;// Sequencia sob analise
  seqId = threadIdx.x + blockIdx.x*blockDim.x;
  seqSize_bu = bloco1+bloco2+blocoV;

	if(seqId < totalseqs){
	  tipo = 0;
	  fase = 0;
	  while(fase + seqSize_bu <= seqSize_an && !tipo){
			   seq = data[seqId]+fase;	
			   alarmS = 0;
			   alarmAS = 0;
			   // Quando esse loop for encerrado eu jah saberei se a sequencia eh senso, antisenso ou nada
			   for(baseId=0; 
						(baseId < seqSize_bu) && (!alarmS || !alarmAS); 
										baseId++){	
					linha = 0;
					// Carrega a linha analisada	
					lsenso = d_matrix_senso[baseId];
					lantisenso = d_matrix_antisenso[baseId];	
										
					linha = seq[baseId];
					
					// Verifica se algum alarme deve ser ativado			
					alarmS += (linha-lsenso)*(lsenso-'N');		
					alarmAS += (linha-lantisenso)*(lantisenso-'N');
				}
				
			// Guarda resultados
			if(!alarmS)
				tipo = SENSO;
			else 
				if(!alarmAS) 
					tipo = ANTISENSO;
			
			// Caso nao tenha encontrado nada, tenta pular para a base seguinte
			fase++;   
		}			
		
	   
		if(tipo == SENSO){
				search_gaps[seqId] = fase + bloco1 -1;
		}else  
			if(tipo == ANTISENSO){
				search_gaps[seqId] = fase + bloco2 -1;
			}
								 
		resultados[seqId] = tipo;	 
	}
	return;
}

extern "C" void k_busca(const int loaded,const int seqSize_an,const int seqSize_bu,int bloco1,int bloco2,int blocoV,char **data,short int *resultados,short int *search_gaps,char **founded,hipStream_t stream){
	int num_threads;
	int num_blocks;
	
	if(loaded > MAX_CUDA_THREADS_PER_BLOCK){
		num_threads = MAX_CUDA_THREADS_PER_BLOCK;
		num_blocks = (float)loaded/(float)num_threads + 1;
	}else{
		num_threads = loaded;
		num_blocks = 1;
	}
	
	dim3 dimBlock(num_threads);
	dim3 dimGrid(num_blocks);
	
	k_buscador<<<dimGrid,dimBlock,0>>>(loaded,seqSize_an,data,resultados,search_gaps,bloco1,bloco2,blocoV);
	
	checkCudaError();
	return;
}

////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////////////
///////////////				Metodo de busca sem CUDA				////////////////////
////////////////////////////////////////////////////////////////////////////////////////

/*void failure_function(char *s,short int *table){
	short int pos = 2;
	short int cnd = 0;
	short int seq_size = strlen(s);
	
	table[0] = -1;
	table[1] = 0;
	
	while(pos < seq_size){
		if(s[pos-1] == s[cnd]){
			cnd++;
			table[pos] = cnd;
			pos++;
		}else if(cnd > 0){
			cnd = table[cnd];
		}else{
			table[pos] = 0;
			pos++;
		}
	} 
	
}*/


////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////   	Auxiliar     ///////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////

extern "C" void checkCudaError(){
	char erro[100];
	strcpy(erro,hipGetErrorString(hipGetLastError()));
    if(strcmp(erro,"no error") != 0){
		printf("%s\n",erro);
    }   
}


////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
void getMatrix(short int *matrix,char *str){
	// Matrix já deve vir alocada
	int size_y;
	int i;

	size_y = strlen(str);

	// Preenche matriz
	for(i = 0; i < size_y;i++){
		matrix[i] = str[i];
	}	

	return;
}

 void set_grafo(char *senso,char *antisenso,short int *matrix_senso,short int *matrix_antisenso){
  // As matrizes já devem vir alocadas
  
  getMatrix(matrix_senso,senso);
  getMatrix(matrix_antisenso,antisenso);
  
  return;
}

char* get_antisenso(char *s){
	int i;
	char *antisenso;
	int a_size;
	
	a_size = strlen(s);
	antisenso = (char*)malloc((a_size+1)*sizeof(char));
	strcpy(antisenso,"");
	
	for(i=0;i<a_size;i++){
		switch(s[a_size - i-1]){
			case 'A':
				strcat(antisenso, "T");
			break;
			case 'C':
				strcat(antisenso, "G");
			break;
			case 'T':
				strcat(antisenso, "A");
			break;
			case 'G':
				strcat(antisenso, "C");
			break;
			default:
				strcat(antisenso,"N");
				break;
		}	
	}
	//strcat(antisenso,'\0');
	
	return antisenso;
}

extern "C" void setup_for_cuda(char *seq){
	// Recebe um vetor de caracteres com o padrão a ser procurado
	short int *h_matrix_senso;
	short int *h_matrix_antisenso;
	int size = strlen(seq);
	char *d_senso;
	char *d_antisenso;
	
	h_matrix_senso = (short int*)malloc(size*sizeof(short int));
	h_matrix_antisenso = (short int*)malloc(size*sizeof(short int));
	    
    //Configura grafos direto na memória da GPU
	set_grafo(seq,get_antisenso(seq),h_matrix_senso,h_matrix_antisenso);
	
	// Copia dados
	hipMemcpyToSymbol(HIP_SYMBOL(d_matrix_senso),h_matrix_senso,size*sizeof(short int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_matrix_antisenso),h_matrix_antisenso,size*sizeof(short int),0,hipMemcpyHostToDevice);
	hipMalloc((void**)&d_senso,(size+1)*sizeof(char));
	hipMalloc((void**)&d_antisenso,(size+1)*sizeof(char));
	
	hipMemcpy(d_senso,seq,(size+1)*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_antisenso,get_antisenso(seq),(size+1)*sizeof(char),hipMemcpyHostToDevice);
	
	//printf("Verificando matrizes:...\n");
	//check_matrix<<<1,size,0>>>(d_senso,d_antisenso);
	
	//printString("Grafo de busca configurado.",NULL);
	free(h_matrix_senso);
	free(h_matrix_antisenso);
	
	return;
}
