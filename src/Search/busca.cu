#include "hip/hip_runtime.h"
//      busca.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Implementação do algoritmo de busca.
//
//		27/03/2012

#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "../Headers/estruturas.h"
#include "../Headers/cuda_functions.h"
#include "../Headers/log.h"

#define ABSOLUTO(a) a>=0?a:-a

__constant__ short int d_matrix_senso[MAX_SEQ_SIZE];
__constant__ short int d_matrix_antisenso[MAX_SEQ_SIZE];

extern "C" void cuda_convert_to_graph(char *s,short int s_len,short int *vector){
  int i;
  for(i=0;i<s_len-1;i++)
    vector[i] = s[i]*(2+s[i+1]);
  return;
}

void convert_target_to_graph(char *s,int s_len,short int *vector){
  int i;
  for(i=0;i<s_len-1;i++)
    if(s[i] == 'N' || s[i+1] == 'N')
      vector[i] = -1*s[i]*(2+s[i+1]);
    else
      vector[i] = s[i]*(2+s[i+1]);
  vector[s_len-1] = 0;
  return;
}

__device__ int get_candidate_table(short int start_vertex,short int *vertexes,short int v_size,short int *table){
  int i;
  int j = 0;
  for(i=0;i<v_size;i++){
    if(vertexes[i] == start_vertex){
      table[j] = i;
      j++;
    }
  }
  return j;
}

__device__ int match_check(short int *target,short int target_size,short int *analyse){
  int i;
  for(i=0;i<target_size;i++){
    if(target[i] > 0)
      if(target[i] != analyse[i])
        return 0;
  }
  return 1;
}

extern "C" void checkCudaError();

////////////////////////////////////////////////////////////////////////////////////////
//////////////////					Buscador		 				////////////////////
////////////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////////////
///////////////				Metodo de busca com CUDA				////////////////////
////////////////////////////////////////////////////////////////////////////////////////

__global__ void k_buscador(int totalseqs,
										int seqSize_an,
										short int *vertexes,
										short int *candidates,
										short int *resultados,
										short int *search_gaps,
										int bloco1,
										int bloco2,
										int blocoV){

  ////////		UM THREAD POR SEQUENCIA
  ////////
  ////////
  ////////
  ////////		seqSize_an: o tamanho da sequencia analisada
  ////////		seqSize_busca: o tamanho da sequencia alvo
  ////////
  ////////
  ////////
  
	unsigned int seqId = threadIdx.x + blockIdx.x*blockDim.x;;// id da sequencia analisada
	short int seqSize_bu = bloco1+bloco2+blocoV;;// Tamanho da sequencia alvo
	short tipo;
	short int *this_candidates = &candidates[seqId*seqSize_an];
	short int *this_vertexes = &vertexes[seqId*seqSize_an];
	short int num_sensos_candidates;
	short int num_antisensos_candidates;
	short int candidate_pos_sensos;
	short int candidate_pos_antisensos;
	short int i;

	if(seqId < totalseqs){
		num_sensos_candidates = get_candidate_table(d_matrix_senso[0],this_vertexes,seqSize_an-seqSize_bu+1,this_candidates);
		tipo = 0;
	  	for(i=0;i<num_sensos_candidates && !tipo;i++){
			candidate_pos_sensos = this_candidates[i];
			if(match_check(d_matrix_senso,seqSize_bu,&this_vertexes[candidate_pos_sensos])){
			  search_gaps[seqId] = i + bloco1;
			  tipo = SENSO;  
			}
		}
		if(!tipo){		
			num_antisensos_candidates = get_candidate_table(d_matrix_antisenso[0],this_vertexes,seqSize_an-seqSize_bu+1,this_candidates);
			for(i=0;i<num_antisensos_candidates && !tipo;i++){
			 candidate_pos_antisensos = this_candidates[i];
			  if(match_check(d_matrix_antisenso,seqSize_bu,&this_vertexes[candidate_pos_antisensos])){
			    search_gaps[seqId] = i + bloco2;
			    tipo = ANTISENSO;
			  }
			}
		}		
								 
		resultados[seqId] = tipo;	 
	}
	return;
}

extern "C" void k_busca(const int loaded,const int seqSize_an,const int seqSize_bu,int bloco1,int bloco2,int blocoV,short int *vertexes,short int *candidates,short int *resultados,short int *search_gaps,hipStream_t stream){
	int num_threads;
	int num_blocks;
	
	if(loaded > MAX_CUDA_THREADS_PER_BLOCK){
		num_threads = MAX_CUDA_THREADS_PER_BLOCK;
		num_blocks = (float)loaded/(float)num_threads + 1;
	}else{
		num_threads = loaded;
		num_blocks = 1;
	}
	
	dim3 dimBlock(num_threads);
	dim3 dimGrid(num_blocks);
	
	k_buscador<<<dimGrid,dimBlock,0,stream>>>(loaded,seqSize_an,vertexes,candidates,resultados,search_gaps,bloco1,bloco2,blocoV);
	
	checkCudaError();
	return;
}
////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////   	Auxiliar     ///////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////

extern "C" void checkCudaError(){
	char erro[100];
	strcpy(erro,hipGetErrorString(hipGetLastError()));
    if(strcmp(erro,"no error") != 0){
		printf("%s\n",erro);
    }   
}


////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
void getMatrix(short int *matrix,char *str){
	// Matrix já deve vir alocada
	int size_y;
	int i;

	size_y = strlen(str);

	// Preenche matriz
	for(i = 0; i < size_y;i++){
		matrix[i] = str[i];
	}	

	return;
}

 void set_grafo(char *senso,char *antisenso,short int *matrix_senso,short int *matrix_antisenso){
  // As matrizes já devem vir alocadas
  
  convert_target_to_graph(senso,strlen(senso),matrix_senso);
  convert_target_to_graph(antisenso,strlen(antisenso),matrix_antisenso);
  
  return;
}

char* get_antisenso(char *s){
	int i;
	char *antisenso;
	int a_size;
	
	a_size = strlen(s);
	antisenso = (char*)malloc((a_size+1)*sizeof(char));
	strcpy(antisenso,"");
	
	for(i=0;i<a_size;i++){
		switch(s[a_size - i-1]){
			case 'A':
				strcat(antisenso, "T");
			break;
			case 'C':
				strcat(antisenso, "G");
			break;
			case 'T':
				strcat(antisenso, "A");
			break;
			case 'G':
				strcat(antisenso, "C");
			break;
			default:
				strcat(antisenso,"N");
				break;
		}	
	}
	//strcat(antisenso,'\0');
	
	return antisenso;
}

extern "C" void setup_for_cuda(char *seq){
	// Recebe um vetor de caracteres com o padrão a ser procurado
	short int *h_matrix_senso;
	short int *h_matrix_antisenso;
	int size = strlen(seq);
	char *d_senso;
	char *d_antisenso;
	
	h_matrix_senso = (short int*)malloc(size*sizeof(short int));
	h_matrix_antisenso = (short int*)malloc(size*sizeof(short int));
	    
    //Configura grafos direto na memória da GPU
	set_grafo(seq,get_antisenso(seq),h_matrix_senso,h_matrix_antisenso);
	
	// Copia dados
	hipMemcpyToSymbol(HIP_SYMBOL(d_matrix_senso),h_matrix_senso,size*sizeof(short int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_matrix_antisenso),h_matrix_antisenso,size*sizeof(short int),0,hipMemcpyHostToDevice);
	/*hipMalloc((void**)&d_senso,(size+1)*sizeof(char));
	hipMalloc((void**)&d_antisenso,(size+1)*sizeof(char));
	
	hipMemcpy(d_senso,seq,(size+1)*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_antisenso,get_antisenso(seq),(size+1)*sizeof(char),hipMemcpyHostToDevice);
	*/
	//printf("Verificando matrizes:...\n");
	//check_matrix<<<1,size,0>>>(d_senso,d_antisenso);
	
	//printString("Grafo de busca configurado.",NULL);
	free(h_matrix_senso);
	free(h_matrix_antisenso);
	
	return;
}
