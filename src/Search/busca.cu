#include "hip/hip_runtime.h"
//      busca.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Implementação do algoritmo de busca por grafos.
//
//		27/03/2012

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../Headers/estruturas.h"
#include "../Headers/cuda_functions.h"
#include "../Headers/log.h"

#define ABSOLUTO(a) a>=0?a:-a
short int matrix_senso[MAX_SEQ_SIZE];
short int matrix_antisenso[MAX_SEQ_SIZE];

__constant__ short int d_matrix_senso[MAX_SEQ_SIZE];
__constant__ short int d_matrix_antisenso[MAX_SEQ_SIZE];


extern "C" void checkCudaError();

////////////////////////////////////////////////////////////////////////////////////////
//////////////////					Buscador		 				////////////////////
////////////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////////////
///////////////				Metodo de busca com CUDA				////////////////////
////////////////////////////////////////////////////////////////////////////////////////

__global__ void k_buscador(int totalseqs,
										int seqSize_an,
										char **data,
										short int *resultados,
										short int *search_gaps,
										char **founded,
										int bloco1,
										int bloco2,
										int blocoV){

  ////////		UM THREAD POR SEQUENCIA
  ////////
  ////////
  ////////
  ////////		seqSize_an: o tamanho da sequencia analisada
  ////////		seqSize_busca: o tamanho da sequencia alvo
  ////////		data: o endereco com todo o buffer carregado
  ////////
  ////////
  ////////
  
  unsigned int seqId;// id da sequencia analisada
  int baseId;// id da base analisada
  short int tipo;// Variavel temporaria para salvar o resultado de uma analise
  short int linha;// Cada thread cuida de uma linha
  short int lsenso;// Guarda o valor da matriz senso para ser comparado com a sequencia alvo
  short int lantisenso;// Guarda o valor da matriz antisenso para ser comparado com a sequencia alvo
  short int alarmS;// Caso a comparacao do valor da matriz senso falhe, essa variavel encerra o loop
  short int alarmAS;// Caso a comparacao do valor da matriz antisenso falhe, essa variavel encerra o loop
  short int fase;// Guarda a posicao analisada
  short int seqSize_bu;// Tamanho da sequencia alvo
  short int i;
  char *seq;// Sequencia sob analise
  seqId = threadIdx.x + blockIdx.x*blockDim.x;
  seqSize_bu = bloco1+bloco2+blocoV;

	if(seqId < totalseqs){
	  tipo = 0;
	  fase = 0;
	  while(fase + seqSize_bu <= seqSize_an && !tipo){
			   seq = data[seqId]+fase;	
			   alarmS = 0;
			   alarmAS = 0;
			   // Quando esse loop for encerrado eu jah saberei se a sequencia eh senso, antisenso ou nada
			   for(baseId=0; 
						(baseId < seqSize_bu) && (!alarmS || !alarmAS); 
										baseId++){	
					linha = 0;
					// Carrega a linha analisada	
					lsenso = d_matrix_senso[baseId];
					lantisenso = d_matrix_antisenso[baseId];	
											
					// Conversao de char para inteiro
					switch(seq[baseId]){
						case 'A':
							linha = A;	
						break;
						case 'C':
							linha = C;		
						break;
						case 'G':
							linha = G;	
						break;
						case 'T':
							linha = T;
						break;
						default:
							linha = N;
						break;
					}
					
					// Verifica se algum alarme deve ser ativado			
					alarmS += (linha-lsenso)*(lsenso-N);		
					alarmAS += (linha-lantisenso)*(lantisenso-N);
				}
				
			// Guarda resultados
			if(!alarmS)
				tipo = SENSO;
			else 
				if(!alarmAS) 
					tipo = ANTISENSO;
			
			// Caso nao tenha encontrado nada, tenta pular para a base seguinte
			fase++;   
		}			
		
	   
		if(tipo == SENSO)
			 for(i=0;i<seqSize_an;i++)
					founded[seqId][i] = data[seqId][i];
		else  
			if(tipo == ANTISENSO)
				for(i=0;i<seqSize_an;i++)
					founded[seqId][i] = data[seqId][i];
			
								 
		resultados[seqId] = tipo;	 
		search_gaps[seqId] = fase-1;
	}
	return;
}

extern "C" void k_busca(const int loaded,const int seqSize_an,const int seqSize_bu,int bloco1,int bloco2,int blocoV,char **data,short int *resultados,short int *search_gaps,char **founded,hipStream_t stream){
	int num_threads;
	int num_blocks;
	
	if(loaded > MAX_CUDA_THREADS_PER_BLOCK){
		num_threads = MAX_CUDA_THREADS_PER_BLOCK;
		num_blocks = (float)loaded/(float)num_threads + 1;
	}else{
		num_threads = loaded;
		num_blocks = 1;
	}
	
	dim3 dimBlock(num_threads);
	dim3 dimGrid(num_blocks);
	
	k_buscador<<<dimGrid,dimBlock,0,stream>>>(loaded,seqSize_an,data,resultados,search_gaps,founded,bloco1,bloco2,blocoV);
	
	checkCudaError();
	return;
}

////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////////////
///////////////				Metodo de busca sem CUDA				////////////////////
////////////////////////////////////////////////////////////////////////////////////////

extern "C" __host__ void buscador(const int bloco1,const int bloco2,const int seqSize_bu,Buffer *buffer,int *resultados,int *search_gaps,const int seqId){
  int baseId;// id da base analisada
  short int tipo;
  short int linha;// Cada thread cuida de uma linha
  short int lsenso;
  short int lantisenso;
  short int alarmS;
  short int alarmAS;
  short int fase;
  const short int seqSize_an = strlen(buffer->seq[seqId]);
  const short int blocoZ = seqSize_bu - bloco1 - bloco2 + 1;
  char *seq;  
  int i;
  
	  tipo = 0;
	  fase = 0;
	  while(fase + seqSize_bu <= seqSize_an && !tipo){
			   seq = buffer->seq[seqId]+fase;	
			   alarmS = 0;
			   alarmAS = 0;
			   // Quando esse loop for encerrado eu jah saberei se a sequencia eh senso, antisenso ou nada
			   for(baseId=0; 
						(baseId < seqSize_bu) && (!alarmS || !alarmAS); 
										baseId++){
					// Carrega a linha relativa a base analisada		
					linha = 0;
					lsenso = matrix_senso[baseId];
					lantisenso = matrix_antisenso[baseId];
											
					switch(seq[baseId]){
						case 'A':
							linha = A;	
						break;
						case 'C':
							linha = C;		
						break;
						case 'G':
							linha = G;	
						break;
						case 'T':
							linha = T;
						break;
						default:
							linha = N;
						break;
					}
								
					alarmS += (linha-lsenso)*(lsenso-N);		
					alarmAS += (linha-lantisenso)*(lantisenso-N);	
					
				}
			if(!alarmS)
				tipo = SENSO;
			else 
				if(!alarmAS) 
					tipo = ANTISENSO;
			
			fase++;   
									
		
		resultados[seqId] = tipo;	
		if(tipo == SENSO){
			//printf("%s -> s_match= %d e as_match=%d\n",seq,s_match,as_match);
			for(i=0;i<blocoZ;i++){
				  seq[i] = seq[i];
			}
			search_gaps[seqId] = fase + bloco1 -1;
		}else if(tipo == ANTISENSO){
			//printf("%s -> s_match= %d e as_match=%d\n",seq,s_match,as_match);
			for(i=0;i<blocoZ;i++){
				  seq[i] = seq[i];
			}			 
			search_gaps[seqId] = fase + bloco2 -1;
		}
	
	}
	return;
}
////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////   	Auxiliar     ///////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////

extern "C" void checkCudaError(){
	char erro[100];
	strcpy(erro,hipGetErrorString(hipGetLastError()));
    if(strcmp(erro,"no error") != 0){
		printf("%s\n",erro);
    }   
}

extern "C" void busca(const int bloco1,const int bloco2,const int blocos,Buffer *buffer,int *resultados,int *search_gaps){
	int i;
	int size;
	
	size = buffer->load;
	
	for(i=0; i < size; i++)
		buscador(bloco1,bloco2,blocos,buffer,resultados,search_gaps,i);//Metodo de busca
		
	return;
}
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
int getLine(char c){
	// Recebe uma base e retorna uma linha de binarios
	
	switch(c){
		case 'A':
			return A;
		case 'C':
			return C;
		case 'G':
			return G;
		case 'T':
			return T;
		default:
			return N;
	}
}

void getMatrix(short int *matrix,char *str){
	// Matrix já deve vir alocada
	int size_y;
	int i;

	size_y = strlen(str);

	// Preenche matriz
	for(i = 0; i < size_y;i++){
		matrix[i] = getLine(str[i]);
	}	

	return;
}

 void set_grafo_CUDA(char *senso,char *antisenso,short int *matrix_senso,short int *matrix_antisenso){
  // As matrizes já devem vir alocadas
  
  getMatrix(matrix_senso,senso);
  getMatrix(matrix_antisenso,antisenso);
  
  return;
}

char* get_antisenso(char *s){
	int i;
	char *antisenso;
	int a_size;
	
	a_size = strlen(s);
	antisenso = (char*)malloc((a_size+1)*sizeof(char));
	strcpy(antisenso,"");
	
	for(i=0;i<a_size;i++){
		switch(s[a_size - i-1]){
			case 'A':
				strcat(antisenso, "T");
			break;
			case 'C':
				strcat(antisenso, "G");
			break;
			case 'T':
				strcat(antisenso, "A");
			break;
			case 'G':
				strcat(antisenso, "C");
			break;
			default:
				strcat(antisenso,"N");
			break;
		}	
	}
	//strcat(antisenso,'\0');
	
	return antisenso;
}

__global__ void check_matrix(char *senso,char *antisenso){
	// Verifica se a matriz montada corresponde a sequencia desejada
	// Devem haver N threads e 1 bloco para uma sequencia de tamanho N
	
	__shared__ int alarmS;
	__shared__ int alarmAS;
	char cS;
	char cAS;
	int id = threadIdx.x;
	int e;
	
	cS = senso[threadIdx.x];
	cAS = antisenso[threadIdx.x];
	alarmS = 0;
	alarmAS = 0;
	
    switch(cS){
		case 'A':
			e = A;	
		break;
		case 'C':
			e = C;		
		break;
		case 'G':
			e = G;	
		break;
		case 'T':
			e = T;
		break;
		default:
			e = N;
		break;
	}	
	
	// Confere Senso
	if(d_matrix_senso[id] != e) alarmS = 1;
	
	switch(cAS){
		case 'A':
			e = A;	
		break;
		case 'C':
			e = C;		
		break;
		case 'G':
			e = G;	
		break;
		case 'T':
			e = T;
		break;
		default:
			e = N;
		break;
	}	
	
	// Confere Antisenso	
	if(d_matrix_antisenso[id] != e) alarmAS = 1;	
			
	__syncthreads();
	
	if(threadIdx.x == 0){
		if(alarmS)
			printf("Erro! Matriz senso montada incorretamente.\n");					
		else
			printf("Matriz senso montada corretamente.\n");
			
		if(alarmAS)
			printf("Erro! Matriz antisenso montada incorretamente.\n");
		else
			printf("Matriz antisenso montada corretamente.\n");
	}
		
	return;
}

extern "C" void setup_for_cuda(char *seq){
	// Recebe um vetor de caracteres com o padrão a ser procurado
	short int *h_matrix_senso;
	short int *h_matrix_antisenso;
	int size = strlen(seq);
	char *d_senso;
	char *d_antisenso;
	
	h_matrix_senso = (short int*)malloc(size*sizeof(short int));
	h_matrix_antisenso = (short int*)malloc(size*sizeof(short int));
	    
    //Configura grafos direto na memória da GPU
	set_grafo_CUDA(seq,get_antisenso(seq),h_matrix_senso,h_matrix_antisenso);
	
	// Copia dados
	hipMemcpyToSymbol(HIP_SYMBOL(d_matrix_senso),h_matrix_senso,size*sizeof(short int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_matrix_antisenso),h_matrix_antisenso,size*sizeof(short int),0,hipMemcpyHostToDevice);
	hipMalloc((void**)&d_senso,(size+1)*sizeof(char));
	hipMalloc((void**)&d_antisenso,(size+1)*sizeof(char));
	
	hipMemcpy(d_senso,seq,(size+1)*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_antisenso,get_antisenso(seq),(size+1)*sizeof(char),hipMemcpyHostToDevice);
	
	//printf("Verificando matrizes:...\n");
	//check_matrix<<<1,size,0>>>(d_senso,d_antisenso);
	
	//printString("Grafo de busca configurado.",NULL);
	free(h_matrix_senso);
	free(h_matrix_antisenso);
	
	return;
}


extern "C" void set_grafo_NONCuda(char *senso,char *antisenso,short int *matrix_senso,short int *matrix_antisenso){									
  
  getMatrix(matrix_senso,senso);
  getMatrix(matrix_antisenso,antisenso);
  return;
}

extern "C"  void setup_without_cuda(char *seq){
// Recebe um vetor de caracteres com o padrão a ser procurado
	int size = strlen(seq);
	
    //Configura grafos direto na memória da GPU
	set_grafo_NONCuda(seq,get_antisenso(seq),matrix_senso,matrix_antisenso);
	

	return;
}
