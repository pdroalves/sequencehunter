#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "../Headers/busca.h"
extern "C" int gpuDeviceInit(int devID);
extern "C" inline int _ConvertSMVer2Cores(int major, int minor);
extern "C" int gpuGetMaxGflopsDeviceId();
extern "C" int findCudaDevice();

int MAX(int A,int B){
	return A >= B ? A:B;
 }

extern "C" int check_gpu_mode(){
	
	return gpuDeviceInit(findCudaDevice());
}

	//#######################
	
	// General GPU Device CUDA Initialization
extern "C" int gpuDeviceInit(int devID)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
    {
        fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
        return 0;
    }

    if (devID < 0)
       devID = 0;
        

    hipDeviceProp_t deviceProp;
   hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.major < 1)
    {
        fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA. Revision < 1.0.\n");
        return 0;                                                  
    }

    return deviceCount;
}

extern "C" inline int _ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct {
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] = 
	{ { 0x10,  8 },
	  { 0x11,  8 },
	  { 0x12,  8 },
	  { 0x13,  8 },
	  { 0x20, 32 },
	  { 0x21, 48 },
	  {   -1, -1 } 
	};

	int index = 0;
	while (nGpuArchCoresPerSM[index].SM != -1) {
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor) ) {
			return nGpuArchCoresPerSM[index].Cores;
		}
		index++;
	}
	printf("MapSMtoCores undefined SMversion %d.%d!\n", major, minor);
	return -1;
}

// This function returns the best GPU (with maximum GFLOPS)
extern "C" int gpuGetMaxGflopsDeviceId()
{
    int current_device     = 0, sm_per_multiproc  = 0;
    int max_compute_perf   = 0, max_perf_device   = 0;
    int device_count       = 0, best_SM_arch      = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount( &device_count );
    
    // Find the best major SM Architecture GPU device
    while (current_device < device_count)
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major > 0 && deviceProp.major < 9999)
        {
            best_SM_arch = MAX(best_SM_arch, deviceProp.major);
        }
        current_device++;
    }

    // Find the best CUDA capable GPU device
    current_device = 0;
    while( current_device < device_count )
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major == 9999 && deviceProp.minor == 9999)
        {
            sm_per_multiproc = 1;
        }
        else
        {
            sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
        }
        
        int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
        
    if( compute_perf  > max_compute_perf )
    {
            // If we find GPU with SM major > 2, search only these
            if ( best_SM_arch > 2 )
            {
                // If our device==dest_SM_arch, choose this, or else pass
                if (deviceProp.major == best_SM_arch)
                {
                    max_compute_perf  = compute_perf;
                    max_perf_device   = current_device;
                 }
            }
            else
            {
                max_compute_perf  = compute_perf;
                max_perf_device   = current_device;
             }
        }
        ++current_device;
    }
    return max_perf_device;
}


// Initialization code to find the best CUDA Device
extern "C" int findCudaDevice()
{
    hipDeviceProp_t deviceProp;
    int devID = 0;
    
    // Escolhe o device com maior taxa de Gflops/s
    devID = gpuGetMaxGflopsDeviceId();
    hipSetDevice( devID );
    hipGetDeviceProperties(&deviceProp, devID);
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    
    return devID;
}
// end of CUDA Helper Functions

