#include "hip/hip_runtime.h"
//      cuda_stack.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Recupera as sequências encontradas e salvas na memória da GPU
//
//		31/03/2012

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "estruturas.h"

__global__ void k_recupera(char **data,char **senso,char **antisenso){
	
  ////////
  ////////	
  ////////		Esse kernel deve deve ser executado pela mesma quantidade de threads que processaram o kernel k_busca.
  ////////		Recebe endereços na memória global para que as sequências senso e antisenso sejam salvas.
  ////////		Recebe os endereços originais onde todas as sequências foram salvas.
  ////////
  ////////
  ////////
  ////////
  
  const int posicao = blockIdx.x*blockDim.x + threadIdx.x;
  
  //printf("%s\n",data[posicao]);
  switch(data[posicao][0]){
	  case 'S':
		//Senso
		senso[posicao] = data[posicao];
		break;
	  case 'N':
		//Antisenso
		antisenso[posicao] = data[posicao];
		break;
		default:
		senso[posicao][0] = '\0';
		antisenso[posicao][0] = '\0';
		break;
	}
	
	return;
}
