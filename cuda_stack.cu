#include "hip/hip_runtime.h"
//      cuda_stack.cu
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Recupera as sequências encontradas e salvas na memória da GPU
//
//		31/03/2012

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "estruturas.h"

__global__ void k_recupera(char **data,char **senso,char **antisensos){
	
  ////////
  ////////	
  ////////		Esse kernel deve deve ser executado por dois threads, um para o senso e o outro para o antisenso
  ////////		Recebe endereços na memória global para que as sequências senso e antisenso sejam salvas
  ////////		Recebe os endereços originais onde todas as sequências foram salvas
  ////////
  ////////
  ////////
  ////////
  
  const int posicao = blockIdx.x*blockDim.x + threadIdx.x;
  
  switch(data[posicao][0]){
	  case 'S':
		//Senso
		senso[posicao] = data[posicao][1];
		break;
	  case 'N':
		//Antisenso
		antisenso[posicao] = data[posicao][1];
		break;
	return;
}
