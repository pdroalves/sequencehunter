#include "hip/hip_runtime.h"
//      linkedlist.c
//      
//      Copyright 2012 Pedro Alves <pdroalves@gmail.com>
//      
//		Implementação de estruturas do tipo "lista ligada".
//		Suporta a criação de uma nova lista, adicionar novo elemento
//		ao final e todos os algoritmos de busca necessários.
//
//		27/03/2012

#include<stdio.h>
#include "estruturas.h"
#include "operacoes.h"

#define MIN(a,b) a>=b?b:a

lista_ligada* criar_lista(){
	lista_ligada *l;
	l = (lista_ligada*)malloc(sizeof(lista_ligada));
	l->qsenso = 0;
	l->qasenso = 0;
	l->senso = (char*)calloc('\0',5*sizeof(char));
	strcat(l->senso,"Head");
	return l;
}

lista_ligada* criar_elemento_lista(char *seq){
	lista_ligada *novo;
	int seq_size;
	
	seq_size = strlen(seq);
	novo = (lista_ligada*)malloc(sizeof(lista_ligada));
	novo->qsenso = 1;
	novo->qasenso = 0;
	novo->senso = (char*)calloc('\0',(seq_size+1)*sizeof(char));
	memcpy(novo->senso,seq,seq_size+1);
	novo->prox = NULL;
	return novo;
}

void adicionar_elemento(lista_ligada *lista,lista_ligada *novo){
	//Recebe ultimo elemento da lista ligada
	lista_ligada *tmp;
	
	tmp = lista;
	while(tmp->prox != NULL){
		tmp = tmp->prox;//Procura o ultimo elemento da fila
	}
	tmp->prox = novo;

	return;
}

void remover_elemento(lista_ligada *atual, lista_ligada *anterior){
	anterior->prox = atual->prox;
	free(atual->senso);
	free(atual);
	atual = anterior->prox;
	return;
}

int busca_lista_s(lista_ligada *l, char *seq){
	//Busca por determinada sequência senso nas listas ligadas
	//Retorna 0 se encontrar ou se seq for nulo
	//Retorna 1 se não encontrar e tiver de adiciona-lo no final da lista
	lista_ligada *p;
	int cmp;
	int str_size;
	
	str_size = strlen(seq);
	
	if(seq != NULL){
		if(l->prox != NULL){//Lista vazia
			p = l->prox;	
			cmp = strncmp(seq,p->senso,str_size);
			if(cmp == 0){
					//Encontrou
					p->qsenso++;
					return 0;
			}
			
			while(p != NULL){
				cmp = strncmp(seq,p->senso,str_size);
				if(cmp == 0){
						//Encontrou
						p->qsenso++;
						return 0;
				}
				p = p->prox;
			}
		}
		//Não encontrou e chegou ao fim da lista ligada
		
		adicionar_elemento(l,criar_elemento_lista(seq));
		return 1;
	}
	
	return 0;	
}

int busca_lista_as(lista_ligada *l, char *seq){
	//Busca por determinada sequência antisenso nas listas ligadas
	//Retorna 0 se encontrar
	//Retorna 1 se não enccontrar
	lista_ligada *p;
	int cmp;
	
	if(seq != NULL){
		if(l->prox != NULL){//Lista vazia
			p = l->prox;	
			cmp = strcmp(seq,p->senso);
			if(cmp == 0){
					//Encontrou
					p->qasenso++;
					return 0;
			}
			
			while(p != NULL){
				cmp = strcmp(seq,p->senso);
				if(cmp == 0){
						//Encontrou
						p->qasenso++;
						return 0;
				}
				p = p->prox;
			}
		}
		
		return 1;
	}
	
	return 1;	
}

int limpando_sensos(lista_ligada *l){
	lista_ligada *atual,*anterior;
	int sensos_solitarios = 0;
	
	anterior = l;
	if(anterior != NULL){
		atual = anterior->prox;
		while(atual != NULL){
			if(atual->qsenso != atual->qasenso){
				 printf("%s x%d\n",atual->senso,abs(atual->qsenso - atual->qasenso));
				 //remover_elemento(atual,anterior);
				 sensos_solitarios++; 
			}
			anterior = atual;
			atual = atual->prox;
		}
	}
	
	return sensos_solitarios;
}

void imprimir_sensos(lista_ligada *l){
	
	lista_ligada *p;
	
	p = l->prox;
	
	while(p !=NULL){
		printf("	%s x%d\n",p->senso,MIN(p->qsenso,p->qasenso));
		p = p->prox;
	}
}
